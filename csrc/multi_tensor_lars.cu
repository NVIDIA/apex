#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>

#include "type_shim.h"
#include "compat.h"
#include "multi_tensor_apply.cuh"

#include <assert.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 512
#define ILP 4

/**
 * Perform fused SGD on multiple buffers
 * N: number of tensors
 * tl[0] : gradients
 * tl[1] : weights
 * tl[2] : momentum buffers
 * tl[3] : fp16 weights (if appropriate)
 * wd : weight_decay (scalar)
 * momentum : momentum (scalar)
 * dampening : momentum dampening (scalar)
 * lr : learning rate (scalar)
 * nesterov : enable nesterov (bool)
 * first run : necessary for proper momentum handling & init
 * wd_after_momentum : apply weight decay _after_ momentum instead of before
 **/

template<int N, typename T_grad, typename T_weight>
struct LARSFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<N>& tl,
    float *grad_norms,
    float *param_norms,
    float lr,
    float trust_coefficient,
    float epsilon,
    float weight_decay,
    float momentum,
    float dampening,
    bool nesterov,
    bool first_run,
    bool wd_after_momentum,
    float scale,
    const bool is_skipped) {
    
    // Early exit if we don't need to do anything
    if (*noop_gmem) return;
    	   
    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    n -= chunk_idx * chunk_size;
    //n = min(n, chunk_size);

    T_grad* grad_in = (T_grad*) tl.addresses[0][tensor_loc];
    grad_in += chunk_idx * chunk_size;

    T_weight* weight_in = (T_weight*) tl.addresses[1][tensor_loc];
    weight_in += chunk_idx * chunk_size;

    T_weight* mom_in = (T_weight*)tl.addresses[2][tensor_loc];
    mom_in += chunk_idx*chunk_size;

    at::Half *model_weights_out = nullptr;
    if(N == 4)
    {
      model_weights_out = (at::Half*)tl.addresses[3][tensor_loc];
      model_weights_out += chunk_idx*chunk_size;
    }

    float scaled_lr;
    if (is_skipped) {
      scaled_lr = lr;
    }
    else {
      int tensor_offset = tl.start_tensor_this_launch + tensor_loc;
      float p_norm = param_norms[tensor_offset];
      float trust_ratio = 1.0;
      float g_norm = grad_norms[tensor_offset];
      if (g_norm > 0.0f && p_norm > 0.0f) {
        trust_ratio = trust_coefficient * p_norm / (g_norm + p_norm * weight_decay + epsilon);
      }
      scaled_lr = lr * trust_ratio;
    }

    // Non-divergent exit condition for the __syncthreads
    float incoming_grads[ILP];
    float incoming_weights[ILP];
    float incoming_moms[ILP];
    for(int i_start = 0;
        i_start < n && i_start < chunk_size;
        i_start += blockDim.x*ILP)
    {
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        incoming_grads[ii] = 0;
        incoming_weights[ii] = 0;
        incoming_moms[ii] = 0;
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          incoming_grads[ii] = static_cast<float>(grad_in[i]);
          incoming_weights[ii] = static_cast<float>(weight_in[i]);
          incoming_moms[ii] = static_cast<float>(mom_in[i]);
        }
      }

      // note for clarification to future michael:
      // From a pure memory dependency perspective, there's likely no point unrolling
      // the write loop, since writes just fire off once their LDGs arrive.
      // Put another way, the STGs are dependent on the LDGs, but not on each other.
      // There is still compute ILP benefit from unrolling the loop though.
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          // apply weight decay before momentum
          incoming_grads[ii] += weight_decay * incoming_weights[ii];
          incoming_moms[ii] = incoming_moms[ii] * momentum - scaled_lr * incoming_grads[ii];

          // adjust the weight and write out
          if (nesterov) {
            incoming_weights[ii] += incoming_moms[ii] * momentum - scaled_lr * incoming_grads[ii];
          } else {
            incoming_weights[ii] += incoming_moms[ii];
          }

          weight_in[i] = static_cast<T_weight>(incoming_weights[ii]);
          
          // if necessary, write out an fp16 copy of the weights
          if(N == 4)
            model_weights_out[i] = static_cast<at::Half>(weight_in[i]);

          // also write out the new momentum
          //if(momentum != 0.f)
            mom_in[i] = static_cast<T_weight>(incoming_moms[ii]);
        }
      }
    }
  }
};

void multi_tensor_lars_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor grad_norms,
  at::Tensor param_norms,
  float lr,
  float trust_coefficient,
  float epsilon,
  float weight_decay,
  float momentum,
  float dampening,
  bool nesterov,
  bool first_run,
  bool wd_after_momentum,
  float scale,
  const bool is_skipped)
{
  auto num_tensors = tensor_lists.size();
  auto grad_type = tensor_lists[0][0].scalar_type();
  auto weight_type = tensor_lists[1][0].scalar_type();

  if(num_tensors == 4) {
    for(int i = 0; i < tensor_lists[3].size(); i++) {
        TORCH_CHECK(tensor_lists[3][i].scalar_type() == at::ScalarType::Half,
                 "Additional output tensors should always be fp16.");
    }
  }

  TORCH_CHECK(noop_flag.device() == tensor_lists[0][0].device(), "expected noop flag to be on the same device as tensors");

  // We have 3 possibilities to handle here, in terms of
  // grad_type, param_type, momentum_type, requires_fp16_copy
  // 1. fp16, fp16, fp16, No
  // 2. fp32, fp32, fp32, No
  // 3. fp16, fp32, fp32, Yes
  // 4. fp32, fp32, fp32, Yes // this is the materialize_master_grads=True case
  // 5. bfp16, bfp16, bfp16, No
  // 6. bfp16, fp32, fp32, Yes
  // It's easier to hardcode these possibilities than to use
  // switches etc. to handle the cross-product of cases where
  // we don't want the majority of them.

  // Case 1. fp16, fp16, fp16, No
  if(grad_type == at::ScalarType::Half &&
     weight_type == at::ScalarType::Half &&
     num_tensors == 3)
  {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<3, at::Half, at::Half>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  // Case 2. fp32, fp32, fp32, No
  else if(grad_type == at::ScalarType::Float &&
          weight_type == at::ScalarType::Float &&
          num_tensors == 3)
  {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<3, float, float>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  // Case 3. fp16, fp32, fp32, Yes
  else if(grad_type == at::ScalarType::Half &&
          weight_type == at::ScalarType::Float &&
          num_tensors == 4)
  {
    multi_tensor_apply<4>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<4, at::Half, float>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  // Case 4. fp32, fp32, fp32, Yes
  else if(grad_type == at::ScalarType::Float &&
          weight_type == at::ScalarType::Float &&
          num_tensors == 4)
  {
    multi_tensor_apply<4>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<4, float, float>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  // Case 5. bfp16, bfp16, bfp16, No
  else if(grad_type == at::ScalarType::BFloat16 &&
     weight_type == at::ScalarType::BFloat16 &&
     num_tensors == 3)
  {
    multi_tensor_apply<3>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<3, at::BFloat16, at::BFloat16>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  // Case 6. bfp16, fp32, fp32, Yes
  else if(grad_type == at::ScalarType::BFloat16 &&
          weight_type == at::ScalarType::Float &&
          num_tensors == 4)
  {
    multi_tensor_apply<4>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LARSFunctor<4, at::BFloat16, float>(),
        grad_norms.DATA_PTR<float>(),
        param_norms.DATA_PTR<float>(),
        lr,
        trust_coefficient,
        epsilon,
        weight_decay,
        momentum,
        dampening,
        nesterov,
        first_run,
        wd_after_momentum,
        scale,
        is_skipped);
  }
  else
  {
    AT_ERROR("multi_tensor_lars only supports some combinations of gradient & weight types. Given: ",
             "gradient: ", grad_type, ", weight: ", weight_type, ", num_lists: ", num_tensors);
  }

  AT_CUDA_CHECK(hipGetLastError());
}
