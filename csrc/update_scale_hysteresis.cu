#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/Exceptions.h>
#include <ATen/cuda/HIPContext.h>

__global__ void update_scale_hysteresis_cuda_kernel(float* current_scale,
                                                    int* growth_tracker,
                                                    int* hysteresis_tracker,
                                                    const float* found_inf,
                                                    double growth_factor,
                                                    double backoff_factor,
                                                    int growth_interval,
                                                    int hysteresis)
{
  if (*found_inf > 0) {
    *hysteresis_tracker -= 1;

    // Only reset the growth tracker when hysteresis is larger than zero
    if (*hysteresis_tracker > 0) {
      *growth_tracker = 0;
      return;
    }
  }

  if (*found_inf) {
    *current_scale = (*current_scale)*backoff_factor;
    *growth_tracker = 0;
  } else {
    // Entering this branch means we just carried out a successful step,
    // so growth_tracker is incremented before comparing to growth_interval.
    auto successful = (*growth_tracker) + 1;
    if (successful == growth_interval) {
      auto new_scale = static_cast<float>((*current_scale)*growth_factor);
      // Do not grow the scale past fp32 bounds to inf.
      if (isfinite(new_scale)) {
          *current_scale = new_scale;
      }
      *growth_tracker = 0;
    } else {
      *growth_tracker = successful;
    }
  }

  // Reset the hysteresis tracker if no infs are found
  if (*found_inf <= 0) {
    *hysteresis_tracker = hysteresis;
  }
}

at::Tensor update_scale_hysteresis_cuda(at::Tensor current_scale,
                                        at::Tensor growth_tracker,
                                        at::Tensor hysteresis_tracker,
                                        at::Tensor found_inf,
                                        const double growth_factor,
                                        const double backoff_factor,
                                        const int64_t growth_interval,
                                        const int hysteresis)
{
  update_scale_hysteresis_cuda_kernel<<<1, 1, 0, at::cuda::getCurrentCUDAStream()>>>(
    current_scale.mutable_data_ptr<float>(),
    growth_tracker.mutable_data_ptr<int>(),
    hysteresis_tracker.mutable_data_ptr<int>(),
    found_inf.const_data_ptr<float>(),
    growth_factor,
    backoff_factor,
    growth_interval,
    hysteresis);

  AT_CUDA_CHECK(hipGetLastError());

  return current_scale;
}
