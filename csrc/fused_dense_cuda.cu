#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <torch/torch.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11000
// includes cublaslt
#include <hipblaslt.h>
#endif
// FP64 Wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    double* A,
    int lda,
    double* B,
    int ldb,
    const float* beta,
    double* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_64F,
      lda,
      B,
      HIP_R_64F,
      ldb,
      beta,
      C,
      HIP_R_64F,
      ldc,
      HIP_R_64F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP32 Wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    float* A,
    int lda,
    float* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      HIPBLAS_GEMM_DEFAULT);
}

// FP16 Tensor core wrapper around cublas GEMMEx
hipblasStatus_t gemm_bias(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    at::Half* C,
    int ldc) {
  return hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP);
}


#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600


int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  } 

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}







int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double* A,
    int lda,
    double* B,
    int ldb,
    const float *beta, /* host pointer */
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  return 1;
}

int gemm_bias_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BIAS;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          &heuristicResult.algo,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}



int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* gelu_in,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_GELU_AUX;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_GELU_AUX_BIAS;
  } 

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}


int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double* A,
    int lda,
    double* B,
    int ldb,
    const float *beta, /* host pointer */
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void *gelu_in,
    const void* bias) {
  return 1;
}


int gemm_bias_gelu_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* gelu_in,
    const void* bias) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_GELU_AUX;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bias, sizeof(bias));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_GELU_AUX_BIAS;
  } 

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}



int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  } 

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}







int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double* A,
    int lda,
    double* B,
    int ldb,
    const float *beta, /* host pointer */
    double* C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bgrad) {
  return 1;
}

int gemm_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    bool use_bias,
    const void* bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DEFAULT;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (use_bias) {
    status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
    if (status != HIPBLAS_STATUS_SUCCESS) {
      goto CLEANUP;
    }
      epilogue = HIPBLASLT_EPILOGUE_BGRADB;
  }

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }

  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          &heuristicResult.algo,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}


int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float *beta, /* host pointer */
    at::Half* C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_16F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_16F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_16F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    double *A,
    int lda,
    double *B,
    int ldb,
    const float *beta, /* host pointer */
    double *C,
    int ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
    return 1;
}

int gemm_dgelu_bgradb_lt(
    hipblasLtHandle_t ltHandle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha, /* host pointer */
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta, /* host pointer */
    float *C,
    int64_t ldc,
    void *workspace,
    size_t workspaceSize,
    hipStream_t stream,
    const void *gelu_in,
    const void *bgrad) {
  hipblasStatus_t status = HIPBLAS_STATUS_SUCCESS;

  hipblasLtMatmulDescOpaque_t operationDesc = {};
  hipblasLtMatrixLayoutOpaque_t Adesc = {}, Bdesc = {}, Cdesc = {};
  hipblasLtMatmulPreferenceOpaque_t preference = {};

  int returnedResults                             = 0;
  hipblasLtMatmulHeuristicResult_t heuristicResult = {};
  hipblasLtEpilogue_t epilogue = HIPBLASLT_EPILOGUE_DGELU_BGRAD;

  // Create operation descriptor; see hipblasLtMatmulDescAttributes_t
  // for details about defaults; here we just set the transforms for
  // A and B.
  status = cublasLtMatmulDescInit(&operationDesc, HIPBLAS_COMPUTE_32F, HIP_R_32F);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transa));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_BIAS_POINTER, &bgrad, sizeof(bgrad));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER, &gelu_in, sizeof(gelu_in));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }
  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD, &ldc, sizeof(ldc));

  status = hipblasLtMatmulDescSetAttribute(&operationDesc, HIPBLASLT_MATMUL_DESC_EPILOGUE, &epilogue, sizeof(epilogue));
  if (status != HIPBLAS_STATUS_SUCCESS) {
    goto CLEANUP;
  }

  // Create matrix descriptors. Not setting any extra attributes.
  status = cublasLtMatrixLayoutInit(
    &Adesc, HIP_R_32F, transa == HIPBLAS_OP_N ? m : k, transa == HIPBLAS_OP_N ? k : m, lda);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(
    &Bdesc, HIP_R_32F, transb == HIPBLAS_OP_N ? k : n, transb == HIPBLAS_OP_N ? n : k, ldb);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = cublasLtMatrixLayoutInit(&Cdesc, HIP_R_32F, m, n, ldc);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // Create preference handle; In general, extra attributes can be
  // used here to disable tensor ops or to make sure algo selected
  // will work with badly aligned A, B, C. However, for simplicity
  // here we assume A,B,C are always well aligned (e.g., directly
  // come from hipMalloc)
  status = cublasLtMatmulPreferenceInit(&preference);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;
  status = hipblasLtMatmulPreferenceSetAttribute(
    &preference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, &workspaceSize, sizeof(workspaceSize));
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  // We just need the best available heuristic to try and run matmul.
  // There is no guarantee that this will work. For example, if A is
  // badly aligned, you can request more (e.g. 32) algos and try to
  // run them one by one until something works.
  status = hipblasLtMatmulAlgoGetHeuristic(
    ltHandle, &operationDesc, &Adesc, &Bdesc, &Cdesc, &Cdesc, &preference, 1, &heuristicResult, &returnedResults);
  if (status != HIPBLAS_STATUS_SUCCESS) goto CLEANUP;

  if (returnedResults == 0) {
    status = HIPBLAS_STATUS_NOT_SUPPORTED;
    goto CLEANUP;
  }
  status = hipblasLtMatmul(ltHandle,
                          &operationDesc,
                          alpha,
                          A,
                          &Adesc,
                          B,
                          &Bdesc,
                          beta,
                          C,
                          &Cdesc,
                          C,
                          &Cdesc,
                          //&heuristicResult.algo,
                          NULL,
                          workspace,
                          workspaceSize,
                          stream);

CLEANUP:
  // Descriptors are no longer needed as all GPU work was already
  // enqueued.
  return status == HIPBLAS_STATUS_SUCCESS ? 0 : 1;
}

#endif

template <typename T>
int linear_bias_forward_cuda(at::Tensor input, T *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    const float beta_one       = 1.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight,
    in_features,
    input.data_ptr<T>(),
    in_features,
    &beta_zero, /* host pointer */
    output.data_ptr<T>(),
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(bias.data_ptr<T>()));
#endif
    if (status != 0){
        output.copy_(bias);
        status = gemm_bias(
          handle,
          HIPBLAS_OP_T,
          HIPBLAS_OP_N,
          out_features,
          batch_size,
          in_features,
          &alpha,
          weight,
          in_features,
          input.data_ptr<T>(),
          in_features,
          &beta_one,
          output.data_ptr<T>(),
          out_features);
    }
    return status;
}

    
template <typename T>
int linear_bias_backward_cuda(T *input, T *weight, T *d_output, int in_features, int batch_size, int out_features, T *d_weight, T *d_bias, T *d_input,  void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    in_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    input,
    in_features,
    d_output,
    out_features,
    &beta_zero, /* host pointer */
    d_weight,
    in_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(d_bias));
#endif
    

    if (status != 0){
    
        status = gemm_bias(
          handle,
          HIPBLAS_OP_N,
          HIPBLAS_OP_T,
          in_features,
          out_features,
          batch_size,
          &alpha,
          input,
          in_features,
          d_output,
          out_features,
          &beta_zero,
          d_weight,
          in_features);
    }
    
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      in_features,
      batch_size,
      out_features,
      &alpha,
      weight,
      in_features,
      d_output,
      out_features,
      &beta_zero,
      d_input,
      in_features);
    return status;

}

template <typename T>
int linear_gelu_linear_forward_cuda(T *input, T *weight1, T *bias1, T *weight2, T *bias2, int in_features, int hidden_features, int batch_size, int out_features, T *output1, T *output2, T *gelu_in, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
    status = gemm_bias_gelu_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    hidden_features,
    batch_size,
    in_features,
    &alpha, /* host pointer */
    weight1,
    in_features,
    input,
    in_features,
    &beta_zero, /* host pointer */
    output1,
    hidden_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(bias1));
    status = gemm_bias_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_T,
    HIPBLAS_OP_N,
    out_features,
    batch_size,
    hidden_features,
    &alpha, /* host pointer */
    weight2,
    hidden_features,
    output1,
    hidden_features,
    &beta_zero, /* host pointer */
    output2,
    out_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(bias2));
    return status;
#else 
    return 1;
#endif
}

template <typename T>
int linear_gelu_linear_backward_cuda(T *input, T *gelu_in, T *output1, T *weight1, T *weight2, T *d_output1, T *d_output2, int in_features, int batch_size, int hidden_features, int out_features, T *d_weight1, T *d_weight2, T *d_bias1, T *d_bias2, T *d_input, void *lt_workspace) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    // Get the stream from cublas handle to reuse for biasReLU kernel.
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha          = 1.0;
    const float beta_zero       = 0.0;
    int status = 1;
#if defined(CUBLAS_VERSION) && CUBLAS_VERSION >= 11600
//wgrad for first gemm
    status = gemm_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_T,
    hidden_features,
    out_features,
    batch_size,
    &alpha, /* host pointer */
    output1,
    hidden_features,
    d_output2,
    out_features,
    &beta_zero, /* host pointer */
    d_weight2,
    hidden_features,
    lt_workspace,
    1 << 22,
    stream,
    true,
    static_cast<const void*>(d_bias2));
//dgrad for second GEMM
    status = gemm_dgelu_bgradb_lt(
    (hipblasLtHandle_t)handle,
    HIPBLAS_OP_N,
    HIPBLAS_OP_N,
    hidden_features,
    batch_size,
    out_features,
    &alpha, /* host pointer */
    weight2,
    hidden_features,
    d_output2,
    out_features,
    &beta_zero, /* host pointer */
    d_output1,
    hidden_features,
    lt_workspace,
    1 << 22,
    stream,
    static_cast<const void*>(gelu_in),
    static_cast<const void*>(d_bias1));
//wgrad for the first GEMM
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_T,
      in_features,
      hidden_features,
      batch_size,
      &alpha,
      input,
      in_features,
      d_output1,
      hidden_features,
      &beta_zero,
      d_weight1,
      in_features);

//dgrad for the first GEMM
    status = gemm_bias(
      handle,
      HIPBLAS_OP_N,
      HIPBLAS_OP_N,
      in_features,
      batch_size,
      hidden_features,
      &alpha,
      weight1,
      in_features,
      d_output1,
      hidden_features,
      &beta_zero,
      d_input,
      in_features);
#endif
    return status;

}


template int linear_bias_forward_cuda<at::Half>(at::Tensor input, at::Half *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_forward_cuda<float>(at::Tensor input, float *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_forward_cuda<double>(at::Tensor input, double *weight, at::Tensor bias, int in_features, int batch_size, int out_features, at::Tensor output, void *lt_workspace);

template int linear_bias_backward_cuda<at::Half>(at::Half *input, at::Half *weight, at::Half *d_output, int in_features, int batch_size, int out_features, at::Half *d_weight, at::Half *d_bias, at::Half *d_input,  void *lt_workspace) ;

template int linear_bias_backward_cuda<float>(float *input, float *weight, float *d_output, int in_features, int batch_size, int out_features, float *d_weight, float *d_bias, float *d_input,  void *lt_workspace) ;

template int linear_bias_backward_cuda<double>(double *input, double *weight, double *d_output, int in_features, int batch_size, int out_features, double *d_weight, double *d_bias, double *d_input,  void *lt_workspace) ;


template int linear_gelu_linear_forward_cuda<at::Half>(at::Half *input, at::Half *weight1, at::Half *bias1, at::Half *weight2, at::Half *bias2, int in_features, int hidden_features, int batch_size, int out_features, at::Half *output1, at::Half *output2, at::Half *gelu_in, void *lt_workspace) ;

template int linear_gelu_linear_forward_cuda<float>(float *input, float *weight1, float *bias1, float *weight2, float *bias2, int in_features, int hidden_features, int batch_size, int out_features, float *output1, float *output2, float *gelu_in, void *lt_workspace);

template int linear_gelu_linear_forward_cuda<double>(double *input, double *weight1, double *bias1, double *weight2, double *bias2, int in_features, int hidden_features, int batch_size, int out_features, double *output1, double *output2, double *gelu_in, void *lt_workspace) ;

template int linear_gelu_linear_backward_cuda<at::Half>(at::Half *input, at::Half *gelu_in, at::Half *output1, at::Half *weight1, at::Half *weight2, at::Half *d_output1, at::Half *d_output2, int in_features, int batch_size, int hidden_features, int out_features, at::Half *d_weight1, at::Half *d_weight2, at::Half *d_bias1, at::Half *d_bias2, at::Half *d_input, void *lt_workspace);

template int linear_gelu_linear_backward_cuda<float>(float *input, float *gelu_in, float *output1, float *weight1, float *weight2, float *d_output1, float *d_output2, int in_features, int batch_size, int hidden_features, int out_features, float *d_weight1, float *d_weight2, float *d_bias1, float *d_bias2, float *d_input, void *lt_workspace);

template int linear_gelu_linear_backward_cuda<double>(double *input, double *gelu_in, double *output1, double *weight1, double *weight2, double *d_output1, double *d_output2, int in_features, int batch_size, int hidden_features, int out_features, double *d_weight1, double *d_weight2, double *d_bias1, double *d_bias2, double *d_input, void *lt_workspace);
