#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 1024
#define ILP 4

// Step 1 computes the 'update' value of regular Adam optimizer.
template<typename GRAD_T, typename T, typename UPD_T>
struct LAMBStage1Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<5>& tl,
    const float* per_tensor_decay,
    const float beta1,
    const float beta2,
    const float beta1_correction,
    const float beta2_correction,
    const float epsilon,
    const float clipped_global_grad_norm)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float decay = per_tensor_decay[tensor_num];

    GRAD_T* g = (GRAD_T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    T* v = (T*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    UPD_T* update = (UPD_T*)tl.addresses[4][tensor_loc];
    update += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      GRAD_T r_g[ILP];
      T r_p[ILP];
      T r_m[ILP];
      T r_v[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
          r_v[ii] = v[i];
        } else {
          r_g[ii] = GRAD_T(0);
          r_p[ii] = T(0);
          r_m[ii] = T(0);
          r_v[ii] = T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        T scaled_grad = r_g[ii] / clipped_global_grad_norm;
        r_m[ii] = r_m[ii] * beta1 + (1-beta1) * scaled_grad;
        r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
        T next_m_unbiased = r_m[ii] / beta1_correction;
        T next_v_unbiased = r_v[ii] / beta2_correction;
        T denom = std::sqrt(next_v_unbiased) + epsilon;
        r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          update[i] = (UPD_T)r_p[ii];
          m[i] = r_m[ii];
          v[i] = r_v[ii];
        }
      }
    }
  }
};

void multi_tensor_lamb_stage1_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor per_tensor_decay,
  const int step,
  const float beta1,
  const float beta2,
  const float epsilon,
  at::Tensor global_grad_norm,
  const float max_global_grad_norm)
{
  using namespace at;

  const float* g_grad_norm = global_grad_norm.DATA_PTR<float>();
  float clipped_global_grad_norm = *(g_grad_norm) > max_global_grad_norm ? *(g_grad_norm) / max_global_grad_norm : 1.0f;
  float next_step = float(step+1);
  float beta1_correction = 1.0f - std::pow(beta1, next_step);
  float beta2_correction = 1.0f - std::pow(beta2, next_step);
  DISPATCH_FLOAT_AND_HALF_AND_BFLOAT16(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_1",
    DISPATCH_FLOAT_AND_HALF_AND_BFLOAT16(tensor_lists[1][0].scalar_type(), 1, "lamb_stage_1",
      DISPATCH_FLOAT_AND_HALF_AND_BFLOAT16(tensor_lists[4][0].scalar_type(), 2, "lamb_stage_1",
        multi_tensor_apply<5>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          tensor_lists,
          LAMBStage1Functor<scalar_t_0, scalar_t_1, scalar_t_2>(),
          per_tensor_decay.DATA_PTR<float>(),
          beta1,
          beta2,
          beta1_correction,
          beta2_correction,
          epsilon,
          clipped_global_grad_norm); )))

  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
