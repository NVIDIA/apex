#include "hip/hip_runtime.h"
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

#include "type_shim.h"
#include "compat.h"


__device__ __forceinline__ int lastpow2(int n)
{
  int out = 1 << (31 - __clz(n));
  if(n == out)
    out >>= 1;
  return out;
}

__host__ __forceinline__ int h_next_pow2(unsigned int n) {
    n--;
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return ++n;
}

__host__ __forceinline__ int h_last_pow2(unsigned int n) {
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return n - (n >> 1);
}


#define WARP_SIZE 32

template<typename T>
__device__ __forceinline__ T warp_reduce_sum(T val)
{
  #pragma unroll
  for(int i = WARP_SIZE/2; i > 0; i >>= 1)
    val = val + __shfl_down_sync(0xffffffff, val, i);
  return val;
}

template<typename T>
__device__ __forceinline__ T reduce_block(T *x, T val)
{
  int tid = threadIdx.y*blockDim.x + threadIdx.x;
  int blockSize = blockDim.x * blockDim.y;

  if (blockSize > 32) {
    val = warp_reduce_sum(val);
    if (tid % WARP_SIZE == 0)
      x[tid/WARP_SIZE] = val;

    __syncthreads();

    val = (tid < blockSize / WARP_SIZE? x[tid%WARP_SIZE] : T(0));
  }

  if(tid/WARP_SIZE==0) val = warp_reduce_sum(val);

  return val;
}

#define ELEMENTS_PER_ITER 4 // enables concurrency within each thread to hide latency
#define ELEMENTS_PER_THREAD 16
#define OPTIMAL_TILE_W 32
#define MAX_H_BLOCK 128
#define MAX_BLOCK_SIZE 512

__host__ int div_ru(int x, int y) {
  return h_last_pow2(1 + (x-1)/y);
}

__host__ void flexible_launch_configs(
      const int reduction,
      const int stride,
      dim3 &block,
      dim3 &grid,
      const bool coop_flag = false) {
  int block_x = std::min(h_last_pow2(stride), OPTIMAL_TILE_W);
  int block_y = std::min(h_last_pow2(div_ru(reduction , ELEMENTS_PER_THREAD)),
                         MAX_BLOCK_SIZE / block_x);
  if (block_x * block_y != MAX_BLOCK_SIZE) {
    block_x = std::min(h_last_pow2(stride), MAX_BLOCK_SIZE / block_y);
  }

  int grid_x = div_ru(stride, block_x);
  int grid_y = std::min(div_ru(reduction, block_y * ELEMENTS_PER_THREAD), MAX_H_BLOCK);
  if (coop_flag) {
    // it's not worth having a grid reduction if the reduction dimension is not big enough
    grid_y = grid_y < 8 ? 1 : grid_y;
  }

  block.x = block_x;
  block.y = block_y;
  block.z = 1;
  grid.x = grid_x;
  grid.y = grid_y;
  grid.z = 1;
}

template<typename T, typename C>
__device__ __forceinline__ void welford_merge_element(C& count,
                                                      T& mean,
                                                      T& m2n,
                                                      const C& num_new,
                                                      const T& mean_new,
                                                      const T& m2n_new) {
      T factor = T(1.0) / max(1, (count + num_new));
      T delta0 = mean - mean_new;
      mean = (mean_new * num_new + mean * count) * factor;
      m2n += m2n_new + delta0 * delta0 * num_new * count * factor;
      count += num_new;
}

template<typename T>
__device__ __forceinline__ void warp_reduce_mean_m2n(T &mean, T &m2n, int &num)
{
  #pragma unroll
  for(int i = WARP_SIZE/2; i > 0; i >>= 1) {
    auto num_new = __shfl_down_sync(0xffffffff, num, i);
    auto mean_new = __shfl_down_sync(0xffffffff, mean, i);
    auto m2n_new = __shfl_down_sync(0xffffffff, m2n, i);
    welford_merge_element(num, mean, m2n, num_new, mean_new, m2n_new);
  }
}

template <typename T>
__device__ void welford_reduce_mean_m2n(
      T* __restrict__ x,
      int* __restrict__ count,
      T &mean,
      T &m2n,
      int &num,
      int block_size,
      int thread_id)
{
  int lane = thread_id % WARP_SIZE;
  int wid = thread_id / WARP_SIZE;

  if (block_size > 32) {
    warp_reduce_mean_m2n(mean, m2n, num);
    if (lane == 0) {
      x[wid*2] = mean;
      x[wid*2+1] = m2n;
      count[wid] = num;
    }
    __syncthreads();

    if (wid == 0) {
      mean = (thread_id < block_size / WARP_SIZE)? x[lane*2] : T(0);
      m2n = (thread_id < block_size / WARP_SIZE)? x[lane*2+1] : T(0);
      num = (thread_id < block_size / WARP_SIZE)? count[lane] : int(0);
    }
  }

  if (wid==0) warp_reduce_mean_m2n(mean, m2n, num);

  return;
}

// return spatial size for NC+ Tensors
__host__ int get_tensor_spatial_size(const at::Tensor& input)
{
  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }
  return space_size;
}

// promote accumulation scalar type. promote half to float.
__host__ at::ScalarType promote_scalartype(const at::Tensor& input)
{
  return input.scalar_type() == at::ScalarType::Half ?
           at::ScalarType::Float : input.scalar_type();
}

// return single element size, optional accumulation type promotion.
__host__ size_t get_element_data_size(const at::Tensor& input, bool accumulation = false)
{
  auto scalar_type = accumulation ? promote_scalartype(input) : input.scalar_type();
  return at::elementSize(scalar_type);
}

template<typename T, typename C>
__device__ __forceinline__ void welford_merge_block_vertical(C& count,
                                                             T& mean,
                                                             T& m2n,
                                                             C* shmem_count,
                                                             T* shmem_mean,
                                                             T* shmem_m2n) {
  // write to shared memory
  auto address_base = threadIdx.x + threadIdx.y * blockDim.x;
  shmem_mean[address_base] = mean;
  shmem_m2n[address_base] = m2n;
  shmem_count[address_base] = count;

#pragma unroll
  for (int offset = blockDim.y/2; offset > 0; offset >>= 1) {
    __syncthreads();
    if (threadIdx.y < offset && threadIdx.y + offset < blockDim.y) {
      auto address = address_base + offset * blockDim.x;
      // read shared memory back to register for reduction
      auto num_new = shmem_count[address];
      auto mean_new = shmem_mean[address];
      auto m2n_new = shmem_m2n[address];

      welford_merge_element(count, mean, m2n, num_new, mean_new, m2n_new);

      // last write is not necessary
      shmem_mean[address_base] = mean;
      shmem_m2n[address_base] = m2n;
      shmem_count[address_base] = count;
    }
  }
}

template<typename T>
__device__ __forceinline__ void merge_block_vertical(T& sum_dy,
                                                     T& sum_dy_xmu,
                                                     T* shmem_sum_dy,
                                                     T* shmem_sum_dy_xmu) {
  // write to shared memory
  auto address_base = threadIdx.x + threadIdx.y * blockDim.x;
  shmem_sum_dy[address_base] = sum_dy;
  shmem_sum_dy_xmu[address_base] = sum_dy_xmu;

#pragma unroll
  for (int offset = blockDim.y/2; offset > 0; offset >>= 1) {
    __syncthreads();
    if (threadIdx.y < offset && threadIdx.y + offset < blockDim.y) {
      auto address = address_base + offset * blockDim.x;

      sum_dy += shmem_sum_dy[address];
      sum_dy_xmu += shmem_sum_dy_xmu[address];

      // last write is not necessary
      shmem_sum_dy[address_base] = sum_dy;
      shmem_sum_dy_xmu[address_base] = sum_dy_xmu;
    }
  }
}


// welford kernel calculating mean/biased_variance/unbiased_variance
template <typename scalar_t, typename accscalar_t, typename outscalar_t>
__global__ void welford_kernel(
      const scalar_t* __restrict__ input,
      outscalar_t* __restrict__ out_mean,
      outscalar_t* __restrict__ out_var_biased,
      const int bs,
      const int fs,
      const int ss) {
  int block_size = blockDim.x * blockDim.y;
  int count = 0;
  accscalar_t x_mean = accscalar_t(0);
  accscalar_t m_2_n = accscalar_t(0);

  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  for (int batch_id = threadIdx.y; batch_id < bs; batch_id += blockDim.y) {
    int input_base = blockIdx.x*ss + batch_id*ss*fs;
    // sequential welford
    for (int offset = threadIdx.x; offset < ss ; offset += blockDim.x) {
      count++;
      auto x_n = static_cast<accscalar_t>(input[offset+input_base]);
      auto d = x_n - x_mean;
      x_mean += d / count;
      m_2_n += d * (x_n - x_mean);
    }
  }

  static __shared__ int s_mem[160];
  accscalar_t* s_mem_ac = (accscalar_t*) &s_mem[32];

  welford_reduce_mean_m2n<accscalar_t>(s_mem_ac, s_mem, x_mean, m_2_n, count, block_size, thread_id);

  if (thread_id == 0) {
    out_mean[blockIdx.x] = static_cast<outscalar_t>(x_mean);
    out_var_biased[blockIdx.x] = static_cast<outscalar_t>(m_2_n/count);
  }
}

// elementwise BN kernel
template <typename scalar_t, typename accscalar_t, typename layerscalar_t>
__global__ void batchnorm_forward_kernel(
      const scalar_t* __restrict__ input,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      const layerscalar_t* __restrict__ weight,
      const layerscalar_t* __restrict__ shift,
      scalar_t* __restrict__ out,
      const int ss,
      const int bs) {
  auto m_c = mean[blockIdx.x];
  auto inv_std_c = inv_std[blockIdx.x];
  auto w_c = weight == NULL ? accscalar_t(1.0) : static_cast<accscalar_t>(weight[blockIdx.x]);
  auto s_c = shift == NULL ? accscalar_t(0.0) : static_cast<accscalar_t>(shift[blockIdx.x]);

  for (int batch_offset = blockIdx.y*blockDim.y + threadIdx.y; batch_offset < bs; batch_offset += gridDim.y*blockDim.y) {
    int address_base = blockIdx.x*ss + batch_offset*gridDim.x*ss;
    for (int offset = threadIdx.x + blockIdx.z*blockDim.x; offset < ss ; offset+= gridDim.z*blockDim.x) {
      out[address_base+offset] = static_cast<scalar_t>(w_c * (static_cast<accscalar_t>(input[address_base+offset]) - m_c ) * inv_std_c + s_c);
    }
  }
}

// Backward BN kernel, calculates grad_bias, grad_weight as well as intermediate
// results to calculating grad_input.
// Breaking the grad_input to two step to support sync BN, which requires all
// reduce of the intermediate results across processes.
template <typename scalar_t, typename accscalar_t, typename layerscalar_t>
__global__ void reduce_bn_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ grad_output,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      accscalar_t* __restrict__ sum_dy_o,
      accscalar_t* __restrict__ sum_dy_xmu_o,
      layerscalar_t* __restrict__ grad_weight,
      layerscalar_t* __restrict__ grad_bias,
      const int bs,
      const int fs,
      const int ss) {
  static __shared__ int s_mem[64];
  //int total_item_num = bs * ss;

  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  auto r_mean = mean[blockIdx.x];
  auto factor = inv_std[blockIdx.x];

  // Kahan sum
  accscalar_t sum_dy = 0.0;
  accscalar_t sum_dy_xmu = 0.0;
  accscalar_t sum_dy_c = 0.0;
  accscalar_t sum_dy_xmu_c = 0.0;
  for (int batch_id = threadIdx.y; batch_id < bs; batch_id += blockDim.y) {
    int input_base = blockIdx.x*ss + batch_id*ss*fs;
    for (int offset = threadIdx.x; offset < ss ; offset += blockDim.x) {
      auto e_grad = static_cast<accscalar_t>(grad_output[offset+input_base]);
      auto e_input = static_cast<accscalar_t>(input[offset+input_base]);
      // calculating sum_dy
      auto sum_dy_y = e_grad - sum_dy_c;
      auto sum_dy_t = sum_dy + sum_dy_y;
      sum_dy_c = (sum_dy_t - sum_dy) - sum_dy_y;
      sum_dy = sum_dy_t;

      // calculating sum_dy_xmu
      auto sum_dy_xmu_y = e_grad * (e_input - r_mean) - sum_dy_xmu_c;
      auto sum_dy_xmu_t = sum_dy_xmu + sum_dy_xmu_y;
      sum_dy_xmu_c = (sum_dy_xmu_t - sum_dy_xmu) - sum_dy_xmu_y;
      sum_dy_xmu = sum_dy_xmu_t;
    }
  }

  sum_dy = reduce_block((accscalar_t*)s_mem, sum_dy);
  __syncthreads();
  sum_dy_xmu = reduce_block((accscalar_t*)s_mem, sum_dy_xmu);

  if (thread_id == 0) {
    if (grad_bias != NULL) {
      grad_bias[blockIdx.x] = static_cast<layerscalar_t>(sum_dy);
    }
    if (grad_weight != NULL) {
      grad_weight[blockIdx.x] = static_cast<layerscalar_t>(sum_dy_xmu * factor);
    }
    //mean_dy[blockIdx.x] = sum_dy / total_item_num;
    //mean_dy_xmu[blockIdx.x] = sum_dy_xmu / total_item_num;
    sum_dy_o[blockIdx.x] = sum_dy;
    sum_dy_xmu_o[blockIdx.x] = sum_dy_xmu;
  }
}

// elementwise backward BN kernel
template <typename scalar_t, typename accscalar_t, typename layerscalar_t>
__global__ void batchnorm_backward_kernel(
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ input,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      const layerscalar_t* __restrict__ weight,
      const accscalar_t* __restrict__ sum_dy,
      const accscalar_t* __restrict__ sum_dy_xmu,
      const int* __restrict__ numel,
      scalar_t* __restrict__ grad_input,
      const int64_t world_size,
      const int ss,
      const int bs) {
  int64_t div = 0;
  for (int i = 0; i < world_size; i++) {
    div += numel[i];
  }
  auto m_c = static_cast<accscalar_t>(mean[blockIdx.x]);
  //auto m_dy_c = static_cast<accscalar_t>(mean_dy[blockIdx.x]);
  auto m_dy_c = static_cast<accscalar_t>(sum_dy[blockIdx.x]) / div;
  auto factor_1_c = inv_std[blockIdx.x];
  auto factor_2_c = (weight == NULL ? accscalar_t(1.0) : static_cast<accscalar_t>(weight[blockIdx.x])) * factor_1_c;
  //factor_1_c = factor_1_c * factor_1_c * mean_dy_xmu[blockIdx.x];
  factor_1_c = factor_1_c * factor_1_c * sum_dy_xmu[blockIdx.x] / div;

  for (int batch_offset = blockIdx.y*blockDim.y+threadIdx.y; batch_offset < bs; batch_offset += gridDim.y*blockDim.y) {
    int address_base = blockIdx.x*ss + batch_offset*gridDim.x*ss;
    for (int offset = threadIdx.x + blockIdx.z*blockDim.x; offset < ss ; offset+= gridDim.z*blockDim.x) {
      grad_input[address_base+offset] = (static_cast<accscalar_t>(grad_output[address_base+offset]) - m_dy_c - (static_cast<accscalar_t>(input[address_base+offset]) - m_c) * factor_1_c) * factor_2_c;
    }
  }
}

// welford kernel for c last tensor calculating mean/biased_variance/unbiased_variance
template
   <typename scalar_t,
    typename accscalar_t,
    typename outscalar_t,
    int PARALLEL_LOADS>
__global__ void
welford_kernel_c_last(
      const scalar_t* __restrict__ input,
      outscalar_t* __restrict__ out_mean,
      outscalar_t* __restrict__ out_var_biased,
      volatile accscalar_t* staging_data,
      int* semaphores,
      const int reduction_size,
      const int stride) {
  // hide latency with concurrency
  accscalar_t x_mean[PARALLEL_LOADS];
  accscalar_t m_2_n[PARALLEL_LOADS];
  int count[PARALLEL_LOADS];

#pragma unroll
  for (int i = 0; i < PARALLEL_LOADS; i++) {
    x_mean[i] = accscalar_t(0);
    m_2_n[i] = accscalar_t(0);
    count[i] = accscalar_t(0);
  }
  // tensor dimension (m,c)

  // loop along m dimension
  int inner_loop_stride = blockDim.y * gridDim.y;

  // offset along m dimension
  int m_offset = blockIdx.y * blockDim.y + threadIdx.y;
  int c_offset = blockIdx.x * blockDim.x + threadIdx.x;

  int loop_count = 1 + (reduction_size - 1) / (inner_loop_stride * PARALLEL_LOADS);
  int address_base = m_offset * stride + c_offset;
  int address_increment = inner_loop_stride * stride;

  for (int i = 0; i < loop_count; i++) {
    accscalar_t x_math[PARALLEL_LOADS];
    accscalar_t x_count_inv[PARALLEL_LOADS];
    accscalar_t is_valid[PARALLEL_LOADS];

    // load multiple data in
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      if (c_offset < stride && m_offset < reduction_size) {
        x_math[j] = input[address_base];
        count[j]++;
        x_count_inv[j] = accscalar_t(1) / count[j];
        is_valid[j] = accscalar_t(1);
      } else {
        x_math[j] = accscalar_t(0);
        x_count_inv[j] = accscalar_t(0);
        is_valid[j] = accscalar_t(0);
      }
      m_offset += inner_loop_stride;
      address_base += address_increment;
    }

    // calculate mean/m2n with welford
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      accscalar_t delta0 = x_math[j] - x_mean[j];
      x_mean[j] += delta0 * x_count_inv[j];
      accscalar_t delta1 = x_math[j] - x_mean[j];
      m_2_n[j] += delta0 * delta1 * is_valid[j];
    }
  }

  // thread reduction to accumulate mean/m_2_n/count between PARALLEL_LOADS
#pragma unroll
  for (int j = 1; j < PARALLEL_LOADS; j++) {
    welford_merge_element(count[0], x_mean[0], m_2_n[0], count[j], x_mean[j], m_2_n[j]);
  }

  // release x_mean / m_2_n
  auto mean_th = x_mean[0];
  auto m2_th = m_2_n[0];
  auto count_th = count[0];

  // block-wise reduction with shared memory (since reduction cannot be done within a warp)
  static __shared__ accscalar_t shmem_mean[MAX_BLOCK_SIZE];
  static __shared__ accscalar_t shmem_m2n[MAX_BLOCK_SIZE];
  static __shared__ int shmem_count[MAX_BLOCK_SIZE];

  welford_merge_block_vertical(count_th, mean_th, m2_th, shmem_count, shmem_mean, shmem_m2n);

  // grid reduction if needed (coop launch used at the first place)
  if (gridDim.y > 1) {
    volatile accscalar_t* staging_mean = staging_data;
    volatile accscalar_t* staging_m2n = &staging_data[stride*gridDim.y];
    volatile int* staging_count = reinterpret_cast<volatile int*>(&staging_m2n[stride*gridDim.y]);

    address_base = c_offset + blockIdx.y * stride;
    // write data to staging_data;
    if (threadIdx.y == 0 && c_offset < stride) {
      staging_mean[address_base] = mean_th;
      staging_m2n[address_base] = m2_th;
      staging_count[address_base] = count_th;
    }

    __threadfence();
    __syncthreads(); // ensuring writes to staging_ is visible to all blocks

    __shared__ bool is_last_block_done;
    // mark block done
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      int old = atomicAdd(&semaphores[blockIdx.x], 1);
      is_last_block_done = (old == (gridDim.y-1));
    }

    __syncthreads();

    // check that all data is now available in global memory
    if (is_last_block_done) {
      count_th = 0;
      mean_th = accscalar_t(0.0);
      m2_th = accscalar_t(0.0);

      for (int y = threadIdx.y; y < gridDim.y; y += blockDim.y) {
        address_base = c_offset + y * stride;
        int num_new = c_offset < stride ? staging_count[address_base] : 0;
        accscalar_t mean_new = c_offset < stride ? staging_mean[address_base] : accscalar_t(0.0);
        accscalar_t m2n_new = c_offset < stride ? staging_m2n[address_base] : accscalar_t(0.0);

        welford_merge_element(count_th, mean_th, m2_th, num_new, mean_new, m2n_new);
      }

      welford_merge_block_vertical(count_th, mean_th, m2_th, shmem_count, shmem_mean, shmem_m2n);
      if (threadIdx.y == 0 && c_offset < stride) {
        out_mean[c_offset] = static_cast<outscalar_t>(mean_th);
        out_var_biased[c_offset] = static_cast<outscalar_t>(m2_th / count_th);
      }
    }
  } else {
    if (blockIdx.y == 0 && threadIdx.y == 0 && c_offset < stride) {
      out_mean[c_offset] = static_cast<outscalar_t>(mean_th);
      out_var_biased[c_offset] = static_cast<outscalar_t>(m2_th / count_th);
    }
  }
}

// parallel welford kernel to further reduce mean / biased_var
// into mean / unbiased_var / inv_std across multiple processes.
template <typename scalar_t>
__global__ void welford_kernel_parallel(
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var_biased,
      const int* __restrict__ numel,
      scalar_t* __restrict__ out_mean,
      scalar_t* __restrict__ out_var,
      scalar_t* __restrict__ inv_std,
      const int world_size,
      const int feature_size,
      const float eps) {

  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < feature_size; i += gridDim.x * blockDim.x) {
    // load data;
    int address = i;
    scalar_t x_mean = 0;
    scalar_t m_2_n = 0;
    int count = 0;
    for (int j = 0; j < world_size; j++) {
      welford_merge_element(count, x_mean, m_2_n, numel[j], mean[address], var_biased[address]*numel[j]);
      address += feature_size;
    }
    out_mean[i] = x_mean;
    out_var[i] = m_2_n/ (count - 1);
    inv_std[i] = scalar_t(1) / sqrt(m_2_n/count + eps);
  }
}

// elementwise BN kernel
template <
    typename scalar_t,
    typename accscalar_t,
    typename layerscalar_t,
    int PARALLEL_LOADS>
__global__ void batchnorm_forward_c_last_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ z,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      const layerscalar_t* __restrict__ weight,
      const layerscalar_t* __restrict__ shift,
      scalar_t* __restrict__ out,
      const int reduction_size,
      const int stride,
      const bool fuse_relu) {
  // tensor dimension (m,c)
  // loop along m dimension
  int inner_loop_stride = blockDim.y * gridDim.y;

  // offset along m dimension
  int m_offset = blockIdx.y * blockDim.y + threadIdx.y;
  int c_offset = blockIdx.x * blockDim.x + threadIdx.x;

  auto m_c = mean[c_offset];
  auto inv_std_c = static_cast<accscalar_t>(inv_std[c_offset]);
  auto w_c = weight == NULL ? accscalar_t(1.0) : static_cast<accscalar_t>(weight[c_offset]);
  auto s_c = shift == NULL ? accscalar_t(0.0) : static_cast<accscalar_t>(shift[c_offset]);

  int loop_count = 1 + (reduction_size - 1) / (inner_loop_stride * PARALLEL_LOADS);
  int address_base = m_offset * stride + c_offset;
  int address_increment = inner_loop_stride * stride;

  for (int i = 0; i < loop_count; i++) {
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      if (c_offset < stride && m_offset < reduction_size) {
        auto tmp = w_c * (static_cast<accscalar_t>(input[address_base]) - m_c ) * inv_std_c + s_c;
        if (z != NULL) {
          tmp += z[address_base];
        }
        out[address_base] = (fuse_relu && tmp <= accscalar_t(0.0) ? scalar_t(0.0) : static_cast<scalar_t>(tmp));
      }
      m_offset += inner_loop_stride;
      address_base += address_increment;
    }
  }
}

// elementwise BN kernel
template <
    typename scalar_t,
    typename accscalar_t,
    typename layerscalar_t,
    int PARALLEL_LOADS>
__global__ void relu_backward_c_last_kernel(
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ z,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      const layerscalar_t* __restrict__ weight,
      const layerscalar_t* __restrict__ shift,
      scalar_t* __restrict__ out,
      const int reduction_size,
      const int stride) {
  // tensor dimension (m,c)
  // loop along m dimension
  int inner_loop_stride = blockDim.y * gridDim.y;

  // offset along m dimension
  int m_offset = blockIdx.y * blockDim.y + threadIdx.y;
  int c_offset = blockIdx.x * blockDim.x + threadIdx.x;

  auto m_c = mean[c_offset];
  auto inv_std_c = static_cast<accscalar_t>(inv_std[c_offset]);
  auto w_c = weight == NULL ? accscalar_t(1.0) : static_cast<accscalar_t>(weight[c_offset]);
  auto s_c = shift == NULL ? accscalar_t(0.0) : static_cast<accscalar_t>(shift[c_offset]);

  int loop_count = 1 + (reduction_size - 1) / (inner_loop_stride * PARALLEL_LOADS);
  int address_base = m_offset * stride + c_offset;
  int address_increment = inner_loop_stride * stride;

  for (int i = 0; i < loop_count; i++) {
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      if (c_offset < stride && m_offset < reduction_size) {
        auto tmp = w_c * (static_cast<accscalar_t>(input[address_base]) - m_c ) * inv_std_c + s_c;
        if (z != NULL) {
          tmp += z[address_base];
        }
        out[address_base] = (tmp <= accscalar_t(0.0) ? scalar_t(0.0) : grad_output[address_base]);
      }
      m_offset += inner_loop_stride;
      address_base += address_increment;
    }
  }
}

// batchnorm backward kernel for c last tensor
template
   <typename scalar_t,
    typename accscalar_t,
    typename layerscalar_t,
    int PARALLEL_LOADS>
__global__ void reduce_bn_c_last_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ grad_output,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      accscalar_t* __restrict__ sum_dy_o,
      accscalar_t* __restrict__ sum_dy_xmu_o,
      layerscalar_t* __restrict__ grad_weight,
      layerscalar_t* __restrict__ grad_bias,
      volatile accscalar_t* staging_data,
      int* semaphores,
      const int reduction_size,
      const int stride) {

  // hide latency with concurrency
  accscalar_t sum_dy[PARALLEL_LOADS];
  accscalar_t sum_dy_xmu[PARALLEL_LOADS];

#pragma unroll
  for (int i = 0; i < PARALLEL_LOADS; i++) {
    sum_dy[i] = accscalar_t(0);
    sum_dy_xmu[i] = accscalar_t(0);
  }
  // tensor dimension (m,c)

  // loop along m dimension
  int inner_loop_stride = blockDim.y * gridDim.y;

  // offset along m dimension
  int m_offset = blockIdx.y * blockDim.y + threadIdx.y;
  int c_offset = blockIdx.x * blockDim.x + threadIdx.x;

  int loop_count = 1 + (reduction_size - 1) / (inner_loop_stride * PARALLEL_LOADS);
  int address_base = m_offset * stride + c_offset;
  int address_increment = inner_loop_stride * stride;

  auto r_mean = mean[c_offset];
  auto factor = inv_std[c_offset];

  for (int i = 0; i < loop_count; i++) {
    accscalar_t x_input[PARALLEL_LOADS];
    accscalar_t x_grad_output[PARALLEL_LOADS];

    // load multiple data in
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      if (c_offset < stride && m_offset < reduction_size) {
        x_input[j] = input[address_base];
        x_grad_output[j] = grad_output[address_base];
      } else {
        x_input[j] = accscalar_t(0);
        x_grad_output[j] = accscalar_t(0);
      }
      m_offset += inner_loop_stride;
      address_base += address_increment;
    }

    // calculate sum_dy / sum_dy_xmu
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      sum_dy[j] += x_grad_output[j];
      sum_dy_xmu[j] += x_grad_output[j] * (x_input[j] - r_mean);
    }
  }

  // thread reduction to accumulate sum_dy / sum_dy_xmu between PARALLEL_LOADS
#pragma unroll
  for (int j = 1; j < PARALLEL_LOADS; j++) {
    sum_dy[0] += sum_dy[j];
    sum_dy_xmu[0] += sum_dy_xmu[j];
  }

  // release array of registers
  auto sum_dy_th = sum_dy[0];
  auto sum_dy_xmu_th = sum_dy_xmu[0];

  // block-wise reduction with shared memory (since reduction cannot be done within a warp)
  static __shared__ accscalar_t shmem_sum_dy[MAX_BLOCK_SIZE];
  static __shared__ accscalar_t shmem_sum_dy_xmu[MAX_BLOCK_SIZE];

  merge_block_vertical(sum_dy_th, sum_dy_xmu_th, shmem_sum_dy, shmem_sum_dy_xmu);

  // grid reduction if needed (coop launch used at the first place)
  if (gridDim.y > 1) {
    volatile accscalar_t* staging_sum_dy = staging_data;
    volatile accscalar_t* staging_sum_dy_xmu = &staging_data[stride*gridDim.y];

    address_base = c_offset + blockIdx.y * stride;
    // write data to staging_data;
    if (threadIdx.y == 0 && c_offset < stride) {
      staging_sum_dy[address_base] = sum_dy_th;
      staging_sum_dy_xmu[address_base] = sum_dy_xmu_th;
    }

    __threadfence();
    __syncthreads(); // ensuring writes to staging_ is visible to all blocks

    __shared__ bool is_last_block_done;
    // mark block done
    if (threadIdx.x == 0 && threadIdx.y == 0) {
      int old = atomicAdd(&semaphores[blockIdx.x], 1);
      is_last_block_done = (old == (gridDim.y-1));
    }

    __syncthreads();

    // check that all data is now available in global memory
    if (is_last_block_done) {
      sum_dy_th = accscalar_t(0.0);
      sum_dy_xmu_th = accscalar_t(0.0);

      for (int y = threadIdx.y; y < gridDim.y; y += blockDim.y) {
        address_base = c_offset + y * stride;
        sum_dy_th += (c_offset < stride ? staging_sum_dy[address_base] : accscalar_t(0.0));
        sum_dy_xmu_th += (c_offset < stride ? staging_sum_dy_xmu[address_base] : accscalar_t(0.0));
      }

      merge_block_vertical(sum_dy_th, sum_dy_xmu_th, shmem_sum_dy, shmem_sum_dy_xmu);
      if (threadIdx.y == 0 && c_offset < stride) {
        if (grad_bias != NULL) {
          grad_bias[c_offset] = static_cast<layerscalar_t>(sum_dy_th);
        }
        if (grad_weight != NULL) {
          grad_weight[c_offset] = static_cast<layerscalar_t>(sum_dy_xmu_th * factor);
        }
        //mean_dy[c_offset] = sum_dy_th / reduction_size;
        //mean_dy_xmu[c_offset] = sum_dy_xmu_th / reduction_size;
        sum_dy_o[c_offset] = sum_dy_th;
        sum_dy_xmu_o[c_offset] = sum_dy_xmu_th;
      }
    }
  } else {
    if (blockIdx.y == 0 && threadIdx.y == 0 && c_offset < stride) {
      if (grad_bias != NULL) {
        grad_bias[c_offset] = static_cast<layerscalar_t>(sum_dy_th);
      }
      if (grad_weight != NULL) {
        grad_weight[c_offset] = static_cast<layerscalar_t>(sum_dy_xmu_th * factor);
      }
      //mean_dy[c_offset] = sum_dy_th / reduction_size;
      //mean_dy_xmu[c_offset] = sum_dy_xmu_th / reduction_size;
      sum_dy_o[c_offset] = sum_dy_th;
      sum_dy_xmu_o[c_offset] = sum_dy_xmu_th;
    }
  }
}

// elementwise BN kernel
template <
    typename scalar_t,
    typename accscalar_t,
    typename layerscalar_t,
    int PARALLEL_LOADS>
__global__ void batchnorm_backward_c_last_kernel(
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ input,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ inv_std,
      const layerscalar_t* __restrict__ weight,
      const accscalar_t* __restrict__ sum_dy,
      const accscalar_t* __restrict__ sum_dy_xmu,
      const int* __restrict__ numel,
      scalar_t* __restrict__ grad_input,
      const int64_t world_size,
      const int reduction_size,
      const int stride) {
  int64_t div = 0;
  for (int i = 0; i < world_size; i++) {
    div += numel[i];
  }
  // tensor dimension (m,c)
  // loop along m dimension
  int inner_loop_stride = blockDim.y * gridDim.y;

  // offset along m dimension
  int m_offset = blockIdx.y * blockDim.y + threadIdx.y;
  int c_offset = blockIdx.x * blockDim.x + threadIdx.x;

  auto m_c = mean[c_offset];
  auto m_dy_c = sum_dy[c_offset] / div;
  auto factor_1_c = inv_std[c_offset];
  auto factor_2_c = (weight == NULL? accscalar_t(1.0) : static_cast<accscalar_t>(weight[c_offset])) * factor_1_c;
  factor_1_c = factor_1_c * factor_1_c * sum_dy_xmu[c_offset] / div;

  int loop_count = 1 + (reduction_size - 1) / (inner_loop_stride * PARALLEL_LOADS);
  int address_base = m_offset * stride + c_offset;
  int address_increment = inner_loop_stride * stride;

  for (int i = 0; i < loop_count; i++) {
#pragma unroll
    for (int j = 0; j < PARALLEL_LOADS; j++) {
      if (c_offset < stride && m_offset < reduction_size) {
        grad_input[address_base] = static_cast<scalar_t>(
            (static_cast<accscalar_t>(grad_output[address_base]) - m_dy_c -
            (static_cast<accscalar_t>(input[address_base]) - m_c) * factor_1_c)
            * factor_2_c);
      }
      m_offset += inner_loop_stride;
      address_base += address_increment;
    }
  }
}

std::vector<at::Tensor> welford_mean_var_CUDA(const at::Tensor input) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  auto space_size = get_tensor_spatial_size(input);
  auto scalar_type = promote_scalartype(input);

  at::Tensor out_var_biased = at::empty({feature_size}, input.options().dtype(scalar_type));
  at::Tensor out_mean = at::empty({feature_size}, input.options().dtype(scalar_type));

  int block_y = min(h_last_pow2(batch_size), int(MAX_BLOCK_SIZE / 32));
  int block_x = max(1, min(MAX_BLOCK_SIZE / block_y, h_last_pow2(space_size)));
  const dim3 block(block_x, block_y);
  const dim3 grid(feature_size);

  auto stream = at::cuda::getCurrentCUDAStream();

  {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "welford_mean_var_kernel",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      welford_kernel<scalar_t_0, accscalar_t, accscalar_t><<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          out_mean.DATA_PTR<accscalar_t>(),
          out_var_biased.DATA_PTR<accscalar_t>(),
          batch_size,
          feature_size,
          space_size);
    );
  }

  return {out_mean, out_var_biased};
}

at::Tensor batchnorm_forward_CUDA(
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight,
    const at::optional<at::Tensor> shift) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);
  at::Tensor out = at::empty_like(input);

  auto space_size = get_tensor_spatial_size(input);

  int block_x = max(32, min(MAX_BLOCK_SIZE, h_last_pow2(space_size)/4));
  int block_y = max(1, min(MAX_BLOCK_SIZE/block_x, h_last_pow2(batch_size)/4));
  const dim3 block(block_x, block_y);
  int grid_z = max(1, min(65535, h_last_pow2(space_size)/4/block_x));
  int batch_group_size = max(1, min(65535, h_last_pow2(batch_size)/block_y));
  const dim3 grid(feature_size, batch_group_size, grid_z);
  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() &&
      weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_forward_kernel<scalar_t_0, accscalar_t, accscalar_t><<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<accscalar_t>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<accscalar_t>() : NULL,
          out.DATA_PTR<scalar_t_0>(),
          space_size,
          batch_size);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_forward_kernel<scalar_t_0, accscalar_t, scalar_t_0><<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<scalar_t_0>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<scalar_t_0>() : NULL,
          out.DATA_PTR<scalar_t_0>(),
          space_size,
          batch_size);
    );
  }
  return out;
}

std::vector<at::Tensor> reduce_bn_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight)
{
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  auto scalar_type = promote_scalartype(input);

  at::Tensor sum_dy = at::empty({feature_size}, mean.options());
  at::Tensor sum_dy_xmu = at::empty({feature_size}, mean.options());

  at::Tensor grad_weight;
  at::Tensor grad_bias;
  if (weight.has_value()) {
    grad_weight = at::empty({feature_size}, weight.value().options());
    grad_bias = at::empty({feature_size}, weight.value().options());
  } else {
    grad_weight = at::empty({0}, mean.options());
    grad_bias = at::empty({0}, mean.options());
  }

  auto space_size = get_tensor_spatial_size(input);

  int block_y = min(h_last_pow2(batch_size), int(MAX_BLOCK_SIZE/ 32));
  int block_x = max(1, min(MAX_BLOCK_SIZE/ block_y, h_last_pow2(space_size)));
  const dim3 block(block_x, block_y);
  const dim3 grid(feature_size);
  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() &&
      weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward_reduce",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      reduce_bn_kernel<scalar_t_0, accscalar_t, accscalar_t><<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          grad_output.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          weight.has_value() ? grad_weight.DATA_PTR<accscalar_t>() : NULL,
          weight.has_value() ? grad_bias.DATA_PTR<accscalar_t>() : NULL,
          batch_size,
          feature_size,
          space_size);
    );
  } else {
    if (weight.has_value()) {
        TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
            "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward_reduce",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      reduce_bn_kernel<scalar_t_0, accscalar_t, scalar_t_0><<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          grad_output.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          weight.has_value() ? grad_weight.DATA_PTR<scalar_t_0>() : NULL,
          weight.has_value() ? grad_bias.DATA_PTR<scalar_t_0>() : NULL,
          batch_size,
          feature_size,
          space_size);
    );
  }

  return {sum_dy, sum_dy_xmu, grad_weight, grad_bias};
}

at::Tensor batchnorm_backward_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight,
    const at::Tensor sum_dy,
    const at::Tensor sum_dy_xmu,
    const at::Tensor count) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  at::Tensor grad_input = at::empty_like(input);

  auto space_size = get_tensor_spatial_size(input);

  int block_x = max(32, min(MAX_BLOCK_SIZE, h_last_pow2(space_size)/4));
  int block_y = max(1, min(MAX_BLOCK_SIZE/block_x, h_last_pow2(batch_size)/4));
  const dim3 block(block_x, block_y);
  int grid_z = max(1, min(65535, h_last_pow2(space_size)/4/block_x));
  int batch_group_size = max(1, min(65535, h_last_pow2(batch_size)/block_y));
  const dim3 grid(feature_size, batch_group_size, grid_z);

  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() &&
      weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_backward_kernel<scalar_t_0, accscalar_t, accscalar_t><<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<accscalar_t>() : NULL,
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          count.DATA_PTR<int>(),
          grad_input.DATA_PTR<scalar_t_0>(),
          count.numel(),
          space_size,
          batch_size);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_backward_kernel<scalar_t_0, accscalar_t, scalar_t_0><<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<scalar_t_0>() : NULL,
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          count.DATA_PTR<int>(),
          grad_input.DATA_PTR<scalar_t_0>(),
          count.numel(),
          space_size,
          batch_size);
    );
  }

  return grad_input;
}

std::vector<at::Tensor> welford_parallel_CUDA(const at::Tensor mean_feature_nodes,
                                              const at::Tensor var_biased,
                                              const at::Tensor numel,
                                              const float eps) {
  const auto world_size = mean_feature_nodes.size(0);
  const auto feature_size = mean_feature_nodes.size(1);

  at::Tensor out_var = at::empty({feature_size}, var_biased.options());
  at::Tensor inv_std = at::empty_like(out_var);
  at::Tensor out_mean = at::empty_like(out_var);

  at::Tensor mean_feature_nodes_ = mean_feature_nodes.contiguous();
  at::Tensor var_biased_ = var_biased.contiguous();
  at::Tensor numel_ = numel.contiguous();

  // TODO(jie): tile this for memory coalescing!
  const int block = std::min(h_last_pow2(feature_size), MAX_BLOCK_SIZE);
  const int grid = std::max<int>(1, feature_size / block);

  auto stream = at::cuda::getCurrentCUDAStream();

  {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(mean_feature_nodes.scalar_type(), 0, "welford_parallel_kernel",
      welford_kernel_parallel<scalar_t_0><<<grid, block, 0, stream>>>(
          mean_feature_nodes_.DATA_PTR<scalar_t_0>(),
          var_biased_.DATA_PTR<scalar_t_0>(),
          numel_.DATA_PTR<int>(),
          out_mean.DATA_PTR<scalar_t_0>(),
          out_var.DATA_PTR<scalar_t_0>(),
          inv_std.DATA_PTR<scalar_t_0>(),
          world_size,
          feature_size,
          eps);
    );
  }

  return {out_mean, out_var, inv_std};
}

std::vector<at::Tensor> welford_mean_var_c_last_CUDA(const at::Tensor input) {
  const auto stride = input.size(input.ndimension()-1);
  const auto reduction_size = input.numel() / stride;

  auto scalar_type = promote_scalartype(input);
  auto option = input.options().dtype(scalar_type);

  at::Tensor out_var_biased = at::empty({stride}, option);
  at::Tensor out_mean = at::empty({stride}, option);

  dim3 block;
  dim3 grid;
  flexible_launch_configs(reduction_size, stride, block, grid, true);

  at::Tensor staging_data;
  at::Tensor semaphores;
  if (grid.y > 1) {
    staging_data = at::empty({4*stride*grid.y}, option);
    semaphores = at::zeros({grid.x}, input.options().dtype(at::kInt));
  }

  auto stream = at::cuda::getCurrentCUDAStream();

  {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "welford_mean_var_c_last",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      accscalar_t* staging_data_ptr = grid.y > 1 ? staging_data.DATA_PTR<accscalar_t>() : nullptr;
      int* semaphores_ptr = grid.y > 1 ? semaphores.DATA_PTR<int>() : nullptr;
      welford_kernel_c_last<scalar_t_0, accscalar_t, accscalar_t, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          out_mean.DATA_PTR<accscalar_t>(),
          out_var_biased.DATA_PTR<accscalar_t>(),
          staging_data_ptr,
          semaphores_ptr,
          reduction_size,
          stride);
    );
  }

  return {out_mean, out_var_biased};
}

at::Tensor batchnorm_forward_c_last_CUDA(
    const at::Tensor input,
    const at::optional<at::Tensor> z,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight,
    const at::optional<at::Tensor> shift,
    const bool fuse_relu) {
  const auto stride = input.size(input.ndimension()-1);
  const auto reduction_size = input.numel() / stride;

  at::Tensor out = at::empty_like(input);

  dim3 block;
  dim3 grid;
  flexible_launch_configs(reduction_size, stride, block, grid);

  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() && weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_forward_c_last_kernel<scalar_t_0, accscalar_t, accscalar_t, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          z.has_value() ? z.value().DATA_PTR<scalar_t_0>() : NULL,
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<accscalar_t>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<accscalar_t>(): NULL,
          out.DATA_PTR<scalar_t_0>(),
          reduction_size,
          stride,
          fuse_relu);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_forward_c_last_kernel<scalar_t_0, accscalar_t, scalar_t_0, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          z.has_value() ? z.value().DATA_PTR<scalar_t_0>() : NULL,
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<scalar_t_0>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<scalar_t_0>(): NULL,
          out.DATA_PTR<scalar_t_0>(),
          reduction_size,
          stride,
          fuse_relu);
    );
  }
  return out;
}

std::vector<at::Tensor> reduce_bn_c_last_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight) {
  const auto stride = input.size(input.ndimension()-1);
  const auto reduction_size = input.numel() / stride;

  at::Tensor sumn_dy = at::empty({stride}, mean.options());
  at::Tensor sum_dy_xmu = at::empty({stride}, mean.options());

  at::Tensor grad_weight;
  at::Tensor grad_bias;
  if (weight.has_value()) {
    grad_weight = at::empty({stride}, weight.value().options());
    grad_bias = at::empty({stride}, weight.value().options());
  } else {
    // because I cannot return an uninitialized at::Tensor
    grad_weight = at::empty({0}, mean.options());
    grad_bias = at::empty({0}, mean.options());
  }

  dim3 block;
  dim3 grid;
  flexible_launch_configs(reduction_size, stride, block, grid, true);

  at::Tensor staging_data;
  at::Tensor semaphores;
  if (grid.y > 1) {
    staging_data = at::empty({2*stride*grid.y}, mean.options());
    semaphores = at::zeros({grid.x}, input.options().dtype(at::kInt));
  }
  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value()
      && weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward_reduce",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      accscalar_t* staging_data_ptr = grid.y > 1 ? staging_data.DATA_PTR<accscalar_t>() : nullptr;
      int* semaphores_ptr = grid.y > 1 ? semaphores.DATA_PTR<int>() : nullptr;
      reduce_bn_c_last_kernel<scalar_t_0, accscalar_t, accscalar_t, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          grad_output.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          sumn_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          weight.has_value() ? grad_weight.DATA_PTR<accscalar_t>() : NULL,
          weight.has_value() ?grad_bias.DATA_PTR<accscalar_t>() : NULL,
          staging_data_ptr,
          semaphores_ptr,
          reduction_size,
          stride);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_backward_reduce",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      accscalar_t* staging_data_ptr = grid.y > 1 ? staging_data.DATA_PTR<accscalar_t>() : nullptr;
      int* semaphores_ptr = grid.y > 1 ? semaphores.DATA_PTR<int>() : nullptr;
      reduce_bn_c_last_kernel<scalar_t_0, accscalar_t, scalar_t_0, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          input.DATA_PTR<scalar_t_0>(),
          grad_output.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          sumn_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          weight.has_value() ? grad_weight.DATA_PTR<scalar_t_0>() : NULL,
          weight.has_value() ?grad_bias.DATA_PTR<scalar_t_0>() : NULL,
          staging_data_ptr,
          semaphores_ptr,
          reduction_size,
          stride);
    );
  }

  return {sumn_dy, sum_dy_xmu, grad_weight, grad_bias};
}

at::Tensor batchnorm_backward_c_last_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight,
    const at::Tensor sum_dy,
    const at::Tensor sum_dy_xmu,
    const at::Tensor count) {
  const auto stride = input.size(input.ndimension()-1);
  const auto reduction_size = input.numel() / stride;

  at::Tensor grad_input = at::empty_like(input);

  dim3 block;
  dim3 grid;
  flexible_launch_configs(reduction_size, stride, block, grid);

  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() && weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_backward_c_last_kernel<scalar_t_0, accscalar_t, accscalar_t, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<accscalar_t>() : NULL,
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          count.DATA_PTR<int>(),
          grad_input.DATA_PTR<scalar_t_0>(),
          count.numel(),
          reduction_size,
          stride);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      batchnorm_backward_c_last_kernel<scalar_t_0, accscalar_t, scalar_t_0, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<scalar_t_0>() : NULL,
          sum_dy.DATA_PTR<accscalar_t>(),
          sum_dy_xmu.DATA_PTR<accscalar_t>(),
          count.DATA_PTR<int>(),
          grad_input.DATA_PTR<scalar_t_0>(),
          count.numel(),
          reduction_size,
          stride);
    );
  }
 
  return grad_input;
}

at::Tensor relu_backward_c_last_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::optional<at::Tensor> z,
    const at::Tensor mean,
    const at::Tensor inv_std,
    const at::optional<at::Tensor> weight,
    const at::optional<at::Tensor> shift) {

  const auto stride = input.size(input.ndimension()-1);
  const auto reduction_size = input.numel() / stride;

  at::Tensor out = at::empty_like(input);

  dim3 block;
  dim3 grid;
  flexible_launch_configs(reduction_size, stride, block, grid);

  auto stream = at::cuda::getCurrentCUDAStream();

  if (input.scalar_type() == at::ScalarType::Half
      && weight.has_value() && weight.value().scalar_type() == at::ScalarType::Float) {
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      relu_backward_c_last_kernel<scalar_t_0, accscalar_t, accscalar_t, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          z.has_value() ? z.value().DATA_PTR<scalar_t_0>() : NULL,
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<accscalar_t>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<accscalar_t>(): NULL,
          out.DATA_PTR<scalar_t_0>(),
          reduction_size,
          stride);
    );
  } else {
    if (weight.has_value()) {
      TORCH_CHECK(input.scalar_type() == weight.value().scalar_type(),
          "input.scalar_type() is not supported with weight.scalar_type()");
    }
    using namespace at;
    DISPATCH_FLOAT_AND_HALF(input.scalar_type(), 0, "batchnorm_forward",
      using accscalar_t = at::acc_type<scalar_t_0, true>;
      relu_backward_c_last_kernel<scalar_t_0, accscalar_t, scalar_t_0, ELEMENTS_PER_ITER>
          <<<grid, block, 0, stream>>>(
          grad_output.DATA_PTR<scalar_t_0>(),
          input.DATA_PTR<scalar_t_0>(),
          z.has_value() ? z.value().DATA_PTR<scalar_t_0>() : NULL,
          mean.DATA_PTR<accscalar_t>(),
          inv_std.DATA_PTR<accscalar_t>(),
          weight.has_value() ? weight.value().DATA_PTR<scalar_t_0>() : NULL,
          shift.has_value() ? shift.value().DATA_PTR<scalar_t_0>(): NULL,
          out.DATA_PTR<scalar_t_0>(),
          reduction_size,
          stride);
    );
  }
  return out;
}
