#include "hip/hip_runtime.h"
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <vector>


__device__ __forceinline__ int lastpow2(int n)
{
  int out = 1 << (31 - __clz(n));
  if(n == out) 
    out >>= 1;
  return out;
}

template <typename scalar_t, typename accscalar_t>
__global__ void welford_kernel(
      const scalar_t* __restrict__ input,
      scalar_t* __restrict__ out_mean,
      scalar_t* __restrict__ out_var,
      scalar_t* __restrict__ out_var_biased,
      const int bs,
      const int fs,
      const int ss) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x * blockDim.y;

  float *mean_l = (float*) s_mem;
  float *m2n_l = (float*) &(s_mem[block_size]);
  int *num_item_l = (int*) &(s_mem[block_size*2]);

  int count = 0;
  float x_mean = 0;
  float m_2_n = 0;
  int input_base = blockIdx.x*ss + threadIdx.y*ss*fs;
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  // sequential welford
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    count++;
    auto x_n = static_cast<accscalar_t>(input[offset+input_base]);
    auto x_mean_new = x_mean + (x_n - x_mean) / count;
    m_2_n = m_2_n + (x_n - x_mean_new) * (x_n - x_mean);
    x_mean = x_mean_new;
  }

  // allow idle thread to write to shared memory
  mean_l[thread_id] = x_mean;
  m2n_l[thread_id] = m_2_n;
  num_item_l[thread_id] = count;
  __syncthreads();
  
  // parallel reduce with interleaved threads
  // TODO(jie): unroll this?
  // TODO(jie): maybe I should pad the blockDim.y to power of 2?
  for (int offset = lastpow2(block_size); offset > 0; offset>>=1) {
    // excluding idle threads, because /0!
    if (thread_id < offset && thread_id + offset < block_size && threadIdx.x < ss) {
      auto count = num_item_l[thread_id];
      auto val = mean_l[thread_id];
      auto count2 = num_item_l[thread_id+offset];
      auto val2 = mean_l[thread_id+offset];

      mean_l[thread_id] = (val * count + val2 * count2) / (count + count2);
      val = val - val2;
      m2n_l[thread_id] += m2n_l[thread_id + offset] + val*val*count*count2/(count+count2);
      num_item_l[thread_id] = count + count2;
    }
    __syncthreads();
  }

  if (thread_id == 0) {
    out_mean[blockIdx.x] = static_cast<scalar_t>(mean_l[0]);
    out_var[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/(num_item_l[0]-1));
    out_var_biased[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/num_item_l[0]);
  }
}

template <typename scalar_t, typename accscalar_t>
__global__ void batchnorm_forward_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var,
      const scalar_t* __restrict__ weight,
      const scalar_t* __restrict__ shift,
      scalar_t* __restrict__ out,
      const int ss,
      const float eps) {
  int address_base = blockIdx.x*ss + blockIdx.y*gridDim.x*ss;

  auto m_c = static_cast<accscalar_t>(mean[blockIdx.x]);
  auto var_c = sqrt(static_cast<accscalar_t>(var[blockIdx.x]) + eps);
  auto w_c = static_cast<accscalar_t>(weight[blockIdx.x]);
  auto s_c = static_cast<accscalar_t>(shift[blockIdx.x]);

  // sequential welford
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    out[address_base+offset] = static_cast<scalar_t>((static_cast<accscalar_t>(input[address_base+offset]) - m_c ) / var_c * w_c + s_c);
  }
}

template <typename scalar_t, typename accscalar_t>
__global__ void reduce_bn_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var,
      accscalar_t* __restrict__ mean_dy,
      accscalar_t* __restrict__ mean_dy_xmu,
      scalar_t* __restrict__ grad_weight,
      scalar_t* __restrict__ grad_bias,
      const int bs,
      const int fs,
      const int ss,
      const float eps) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x * blockDim.y;

  float *sum_dy_l = (float*) s_mem;
  float *sum_dy_xmu_l = (float*) &(s_mem[block_size]);
  int total_item_num = bs * ss;

  float s_dy = 0.0;
  float s_dy_xmu = 0.0;
  int input_base = blockIdx.x*ss + threadIdx.y*ss*fs;
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  auto r_mean = static_cast<accscalar_t>(mean[blockIdx.x]);
  auto factor = 1.0 / sqrt(static_cast<accscalar_t>(var[blockIdx.x]) + eps);

  // sequential welford
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    auto e_grad = static_cast<accscalar_t>(grad_output[offset+input_base]);
    auto e_input = static_cast<accscalar_t>(input[offset+input_base]);
    s_dy += e_grad;
    s_dy_xmu += e_grad * (e_input - r_mean);
  }

  sum_dy_l[thread_id] = s_dy;
  sum_dy_xmu_l[thread_id] = s_dy_xmu;
  __syncthreads();
  
  // parallel reduce with interleaved threads
  // TODO(jie): unroll this?
  // TODO(jie): maybe I should pad the blockDim.y to power of 2?
  for (int offset = lastpow2(block_size); offset > 0; offset>>=1) {
    if (thread_id < offset && thread_id + offset < block_size) {
      sum_dy_l[thread_id] += sum_dy_l[thread_id+offset];
      sum_dy_xmu_l[thread_id] += sum_dy_xmu_l[thread_id+offset];
    }
    __syncthreads();
  }

  if (thread_id == 0) {
    grad_bias[blockIdx.x] = static_cast<scalar_t>(sum_dy_l[0]);
    grad_weight[blockIdx.x] = static_cast<scalar_t>(sum_dy_xmu_l[0] * factor);
    mean_dy[blockIdx.x] = sum_dy_l[0] / total_item_num;
    mean_dy_xmu[blockIdx.x] = sum_dy_xmu_l[0] / total_item_num;
  }
}

template <typename scalar_t, typename accscalar_t>
__global__ void batchnorm_backward_kernel(
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var,
      const scalar_t* __restrict__ weight,
      const scalar_t* __restrict__ mean_dy,
      const scalar_t* __restrict__ mean_dy_xmu,
      scalar_t* __restrict__ grad_input,
      const int ss,
      const float eps) {
  int address_base = blockIdx.x*ss + blockIdx.y*gridDim.x*ss;

  auto m_c = static_cast<accscalar_t>(mean[blockIdx.x]);
  auto m_dy_c = static_cast<accscalar_t>(mean_dy[blockIdx.x]);
  auto factor_1_c = static_cast<accscalar_t>(var[blockIdx.x]) + eps;
  auto factor_2_c = static_cast<accscalar_t>(weight[blockIdx.x]) / sqrt(factor_1_c);
  factor_1_c /= static_cast<accscalar_t>(mean_dy_xmu[blockIdx.x]);

  // sequential welford
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    grad_input[address_base+offset] = (static_cast<accscalar_t>(grad_output[address_base+offset]) - m_dy_c - (static_cast<accscalar_t>(input[address_base+offset]) - m_c) / factor_1_c) * factor_2_c;
  }
}

std::vector<at::Tensor> welford_mean_var_CUDA(const at::Tensor input) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }

  at::Tensor out_var = at::empty_like(input).resize_({feature_size});
  at::Tensor out_var_biased = at::empty_like(out_var);
  at::Tensor out_mean = at::empty_like(out_var);

  int block_x = 16;
  const dim3 block(block_x, batch_size);
  const dim3 grid(feature_size);
  // save current mean, var, num_elements;
  int smem_size = batch_size * block_x * 3 * sizeof(int);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "welford_mean_var_kernel", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    welford_kernel<scalar_t, accscalar_t><<<grid, block, smem_size>>>(
        input.data<scalar_t>(),
        out_mean.data<scalar_t>(),
        out_var.data<scalar_t>(),
        out_var_biased.data<scalar_t>(),
        batch_size,
        feature_size,
        space_size);
  }));

  return {out_mean, out_var, out_var_biased};
}

at::Tensor batchnorm_forward_CUDA(
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const at::Tensor weight,
    const at::Tensor shift,
    const float eps) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);
  at::Tensor out = at::empty_like(input);

  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }

  const dim3 block(512);
  // TODO(jie): should I do 1 block per feature?
  const dim3 grid(feature_size, batch_size);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_forward", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    batchnorm_forward_kernel<scalar_t, accscalar_t><<<grid, block>>>(
        input.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        shift.data<scalar_t>(),
        out.data<scalar_t>(),
        space_size,
        eps);
  }));
  
  return out;
}

std::vector<at::Tensor> reduce_bn_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const float eps) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  at::Tensor mean_dy = at::empty_like(mean);
  at::Tensor mean_dy_xmu = at::empty_like(mean);
  at::Tensor grad_weight = at::empty_like(mean);
  at::Tensor grad_bias = at::empty_like(mean);

  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }

  int block_x = 16;
  const dim3 block(block_x, batch_size);
  const dim3 grid(feature_size);
  // shared memory used for reduce;
  int smem_size = batch_size * block_x * 2 * sizeof(int);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_forward", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    reduce_bn_kernel<scalar_t, accscalar_t><<<grid, block, smem_size>>>(
        input.data<scalar_t>(),
        grad_output.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        mean_dy.data<accscalar_t>(),
        mean_dy_xmu.data<accscalar_t>(),
        grad_weight.data<scalar_t>(),
        grad_bias.data<scalar_t>(),
        batch_size,
        feature_size,
        space_size,
        eps);
  }));
  
  return {mean_dy, mean_dy_xmu, grad_weight, grad_bias};
}

at::Tensor batchnorm_backward_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const at::Tensor weight,
    const at::Tensor shift,
    const at::Tensor mean_dy,
    const at::Tensor mean_dy_xmu,
    const float eps) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  at::Tensor grad_input = at::empty_like(input);

  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }

  const dim3 block(512);
  // TODO(jie): should I do 1 block per feature?
  const dim3 grid(feature_size, batch_size);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_forward", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    batchnorm_backward_kernel<scalar_t, accscalar_t><<<grid, block>>>(
        grad_output.data<scalar_t>(),
        input.data<scalar_t>(),
        mean.data<scalar_t>(),
        var.data<scalar_t>(),
        weight.data<scalar_t>(),
        mean_dy.data<scalar_t>(),
        mean_dy_xmu.data<scalar_t>(),
        grad_input.data<scalar_t>(),
        space_size,
        eps);
  }));
  
  return grad_input;
}

template <typename scalar_t, typename accscalar_t>
__global__ void welford_kernel_parallel(
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var_biased,
      scalar_t* __restrict__ out_mean,
      scalar_t* __restrict__ out_var,
      scalar_t* __restrict__ out_var_biased,
      const int ns,
      const int fs,
      const int numel) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x;

  float *mean_l = (float*) s_mem;
  float *m2n_l = (float*) &(s_mem[block_size]);
  int *num_item_l = (int*) &(s_mem[block_size*2]);

  int input_base = blockIdx.x*ns + threadIdx.x;
  int thread_id = threadIdx.x;

  // load data; 
  mean_l[thread_id] = static_cast<accscalar_t>(mean[input_base]);
  m2n_l[thread_id] = static_cast<accscalar_t>(var_biased[input_base]) * numel;
  num_item_l[thread_id] = numel;

  __syncthreads();
  
  // parallel reduce with interleaved threads
  // TODO(jie): unroll this?
  for (int offset = lastpow2(block_size); offset > 0; offset>>=1) {
    if (thread_id < offset && thread_id + offset < block_size) {
      auto count = num_item_l[thread_id];
      auto val = mean_l[thread_id];
      auto count2 = num_item_l[thread_id+offset];
      auto val2 = mean_l[thread_id+offset];

      mean_l[thread_id] = (val * count + val2 * count2) / (count + count2);
      val = val - val2;
      m2n_l[thread_id] += m2n_l[thread_id + offset] + val*val*count*count2/(count+count2);
      num_item_l[thread_id] = count + count2;
    }
    __syncthreads();
  }

  if (thread_id == 0) {
    out_mean[blockIdx.x] = static_cast<scalar_t>(mean_l[0]);
    out_var[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/(num_item_l[0]-1));
    out_var_biased[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/num_item_l[0]);
  }
}
  
std::vector<at::Tensor> welford_parallel_CUDA(const at::Tensor mean_feature_nodes, const at::Tensor var_biased, int numel) {
  const auto feature_size = mean_feature_nodes.size(0);
  const auto node_size = mean_feature_nodes.size(1);

  // TODO(jie): how to properly construct empty tensor with shape?
  at::Tensor out_var = at::empty_like(var_biased).resize_({feature_size});
  at::Tensor out_var_biased = at::empty_like(out_var);
  at::Tensor out_mean = at::empty_like(out_var);

  // TODO(jie): 
  const dim3 block(node_size);
  const dim3 grid(feature_size);
  // save current mean, var, num_elements;
  int smem_size = node_size * 3 * sizeof(int);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(mean_feature_nodes.type(), "welford_parallel_kernel", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    welford_kernel_parallel<scalar_t, accscalar_t><<<grid, block, smem_size>>>(
        mean_feature_nodes.data<scalar_t>(),
        var_biased.data<scalar_t>(),
        out_mean.data<scalar_t>(),
        out_var.data<scalar_t>(),
        out_var_biased.data<scalar_t>(),
        node_size,
        feature_size,
        numel);
  }));

  return {out_mean, out_var, out_var_biased};
}
