#include "hip/hip_runtime.h"
#include <iostream>
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>


#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


#include <vector>


__device__ __forceinline__ int lastpow2(int n)
{
  int out = 1 << (31 - __clz(n));
  if(n == out) 
    out >>= 1;
  return out;
}

// parallel reduce with interleaved threads
// TODO(jie): unroll this?
template <typename scalar_t>
__device__ void welford_reduce_mean_m2n(
      scalar_t* __restrict__ mean_l,
      scalar_t* __restrict__ m2n_l,
      int* __restrict__ num_item_l,
      int block_size,
      int thread_id)
{
  for (int offset = lastpow2(block_size); offset > 0; offset>>=1) {
    if (thread_id < offset && thread_id + offset < block_size) {
      auto count = num_item_l[thread_id];
      auto val = mean_l[thread_id];
      auto count2 = num_item_l[thread_id+offset];
      auto val2 = mean_l[thread_id+offset];

      mean_l[thread_id] = (val * count + val2 * count2) / (count + count2);
      val = val - val2;
      m2n_l[thread_id] += m2n_l[thread_id + offset] + val*val*count*count2/(count+count2);
      num_item_l[thread_id] = count + count2;
    }
    __syncthreads();
  }
}

// return spatial size for NC+ Tensors
__host__ int get_tensor_spatial_size(const at::Tensor& input)
{
  auto space_size = input.size(2);
  for (int i = 3; i < input.ndimension(); i++) {
    space_size *= input.size(i);
  }
  return space_size;
}

// promote accumulation scalar type. promote half to float.
__host__ at::ScalarType promote_scalartype(const at::Tensor& input)
{
  return input.type().scalarType() == at::ScalarType::Half ?
           at::ScalarType::Float : input.type().scalarType();
}

// return single element size, optional accumulation type promotion.
__host__ size_t get_element_data_size(const at::Tensor& input, bool accumulation = false)
{
  auto scalar_type = accumulation ? promote_scalartype(input) : input.type().scalarType();
  return at::elementSize(scalar_type);
}


// welford kernel calculating mean/biased_variance/unbiased_variance
template <typename scalar_t, typename accscalar_t, typename outscalar_t>
__global__ void welford_kernel(
      const scalar_t* __restrict__ input,
      outscalar_t* __restrict__ out_mean,
      outscalar_t* __restrict__ out_var,
      outscalar_t* __restrict__ out_var_biased,
      const int bs,
      const int fs,
      const int ss) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x * blockDim.y;

  accscalar_t* mean_l = (accscalar_t*) s_mem;
  accscalar_t* m2n_l = &(mean_l[block_size]);
  int *num_item_l = (int*) &(m2n_l[block_size]);

  int count = 0;
  float x_mean = 0;
  float m_2_n = 0;
  int input_base = blockIdx.x*ss + threadIdx.y*ss*fs;
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  // sequential welford
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    count++;
    auto x_n = static_cast<accscalar_t>(input[offset+input_base]);
    auto x_mean_new = x_mean + (x_n - x_mean) / count;
    m_2_n = m_2_n + (x_n - x_mean_new) * (x_n - x_mean);
    x_mean = x_mean_new;
  }

  // allow idle thread to write to shared memory
  mean_l[thread_id] = x_mean;
  m2n_l[thread_id] = m_2_n;
  num_item_l[thread_id] = count;
  __syncthreads();
  
  // parallel reduce with interleaved threads
  // TODO(jie): unroll this?
  // TODO(jie): maybe I should pad the blockDim.y to power of 2?
  welford_reduce_mean_m2n<accscalar_t>(mean_l, m2n_l, num_item_l, block_size, thread_id);

  if (thread_id == 0) {
    out_mean[blockIdx.x] = static_cast<outscalar_t>(mean_l[0]);
    out_var[blockIdx.x] = static_cast<outscalar_t>(m2n_l[0]/(num_item_l[0]-1));
    out_var_biased[blockIdx.x] = static_cast<outscalar_t>(m2n_l[0]/num_item_l[0]);
  }
}

// elementwise BN kernel
template <typename scalar_t, typename accscalar_t>
__global__ void batchnorm_forward_kernel(
      const scalar_t* __restrict__ input,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ var,
      const scalar_t* __restrict__ weight,
      const scalar_t* __restrict__ shift,
      scalar_t* __restrict__ out,
      const int ss,
      const float eps) {
  int address_base = blockIdx.x*ss + blockIdx.y*gridDim.x*ss;

  auto m_c = mean[blockIdx.x];
  auto inv_std_c = static_cast<accscalar_t>(rsqrt(var[blockIdx.x] + eps));
  auto w_c = static_cast<accscalar_t>(weight[blockIdx.x]);
  auto s_c = static_cast<accscalar_t>(shift[blockIdx.x]);

  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    out[address_base+offset] = static_cast<scalar_t>(w_c * (static_cast<accscalar_t>(input[address_base+offset]) - m_c ) * inv_std_c + s_c);
  }
}

// Backward BN kernel, calculates grad_bias, grad_weight as well as intermediate
// results to calculating grad_input.
// Breaking the grad_input to two step to support sync BN, which requires all
// reduce of the intermediate results across processes.
template <typename scalar_t, typename accscalar_t>
__global__ void reduce_bn_kernel(
      const scalar_t* __restrict__ input,
      const scalar_t* __restrict__ grad_output,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ var,
      accscalar_t* __restrict__ mean_dy,
      accscalar_t* __restrict__ mean_dy_xmu,
      scalar_t* __restrict__ grad_weight,
      scalar_t* __restrict__ grad_bias,
      const int bs,
      const int fs,
      const int ss,
      const float eps) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x * blockDim.y;

  accscalar_t* sum_dy_l = (accscalar_t*) s_mem;
  accscalar_t* sum_dy_xmu_l = &(sum_dy_l[block_size]);
  int total_item_num = bs * ss;


  int input_base = blockIdx.x*ss + threadIdx.y*ss*fs;
  int thread_id = threadIdx.y*blockDim.x + threadIdx.x;

  auto r_mean = mean[blockIdx.x];
  auto factor = accscalar_t(1.0) / (accscalar_t)sqrt(var[blockIdx.x] + eps);

  // Kahan sum
  accscalar_t sum_dy = 0.0;
  accscalar_t sum_dy_xmu = 0.0;
  accscalar_t sum_dy_c = 0.0;
  accscalar_t sum_dy_xmu_c = 0.0;
  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    auto e_grad = static_cast<accscalar_t>(grad_output[offset+input_base]);
    auto e_input = static_cast<accscalar_t>(input[offset+input_base]);
    // calculating sum_dy
    auto sum_dy_y = e_grad - sum_dy_c;
    auto sum_dy_t = sum_dy + sum_dy_y;
    sum_dy_c = (sum_dy_t - sum_dy) - sum_dy_y;
    sum_dy = sum_dy_t;

    // calculating sum_dy_xmu
    auto sum_dy_xmu_y = e_grad * (e_input - r_mean) - sum_dy_xmu_c;
    auto sum_dy_xmu_t = sum_dy_xmu + sum_dy_xmu_y;
    sum_dy_xmu_c = (sum_dy_xmu_t - sum_dy_xmu) - sum_dy_xmu_y;
    sum_dy_xmu = sum_dy_xmu_t;
  }

  sum_dy_l[thread_id] = sum_dy;
  sum_dy_xmu_l[thread_id] = sum_dy_xmu;
  __syncthreads();
  
  // parallel reduce with interleaved threads
  // TODO(jie): unroll this?
  // TODO(jie): maybe I should pad the blockDim.y to power of 2?
  for (int offset = lastpow2(block_size); offset > 0; offset>>=1) {
    if (thread_id < offset && thread_id + offset < block_size) {
      sum_dy_l[thread_id] = sum_dy_l[thread_id] + sum_dy_l[thread_id+offset];
      sum_dy_xmu_l[thread_id] = sum_dy_xmu_l[thread_id] + sum_dy_xmu_l[thread_id+offset];
    }
    __syncthreads();
  }

  if (thread_id == 0) {
    grad_bias[blockIdx.x] = static_cast<scalar_t>(sum_dy_l[0]);
    grad_weight[blockIdx.x] = static_cast<scalar_t>(sum_dy_xmu_l[0] * factor);
    mean_dy[blockIdx.x] = sum_dy_l[0] / total_item_num;
    mean_dy_xmu[blockIdx.x] = sum_dy_xmu_l[0] / total_item_num;
  }
}

// elementwise backward BN kernel
template <typename scalar_t, typename accscalar_t>
__global__ void batchnorm_backward_kernel(
      const scalar_t* __restrict__ grad_output,
      const scalar_t* __restrict__ input,
      const accscalar_t* __restrict__ mean,
      const accscalar_t* __restrict__ var,
      const scalar_t* __restrict__ weight,
      const accscalar_t* __restrict__ mean_dy,
      const accscalar_t* __restrict__ mean_dy_xmu,
      scalar_t* __restrict__ grad_input,
      const int ss,
      const float eps) {
  int address_base = blockIdx.x*ss + blockIdx.y*gridDim.x*ss;

  auto m_c = static_cast<accscalar_t>(mean[blockIdx.x]);
  auto m_dy_c = static_cast<accscalar_t>(mean_dy[blockIdx.x]);
  auto factor_1_c = static_cast<accscalar_t>(var[blockIdx.x]) + eps;
  auto factor_2_c = static_cast<accscalar_t>(weight[blockIdx.x]) / sqrt(factor_1_c);
  factor_1_c /= static_cast<accscalar_t>(mean_dy_xmu[blockIdx.x]);

  for (int offset = threadIdx.x; offset < ss ; offset+= blockDim.x) {
    grad_input[address_base+offset] = (static_cast<accscalar_t>(grad_output[address_base+offset]) - m_dy_c - (static_cast<accscalar_t>(input[address_base+offset]) - m_c) / factor_1_c) * factor_2_c;
  }
}

// parallel welford kernel to further reduce mean / biased_var / unbiased_var
// across multiple processes.
template <typename scalar_t, typename accscalar_t>
__global__ void welford_kernel_parallel(
      const scalar_t* __restrict__ mean,
      const scalar_t* __restrict__ var_biased,
      scalar_t* __restrict__ out_mean,
      scalar_t* __restrict__ out_var,
      scalar_t* __restrict__ out_var_biased,
      const int ns,
      const int fs,
      const int numel) {
  extern __shared__ int s_mem[];
  int block_size = blockDim.x;

  accscalar_t* mean_l = (accscalar_t*) s_mem;
  accscalar_t* m2n_l = &(mean_l[block_size]);
  int *num_item_l = (int*) &(m2n_l[block_size]);

  int input_base = blockIdx.x*ns + threadIdx.x;
  int thread_id = threadIdx.x;

  // load data; 
  mean_l[thread_id] = static_cast<accscalar_t>(mean[input_base]);
  m2n_l[thread_id] = static_cast<accscalar_t>(var_biased[input_base]) * numel;
  num_item_l[thread_id] = numel;

  __syncthreads();

  welford_reduce_mean_m2n<accscalar_t>(mean_l, m2n_l, num_item_l, block_size, thread_id);

  if (thread_id == 0) {
    out_mean[blockIdx.x] = static_cast<scalar_t>(mean_l[0]);
    out_var[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/(num_item_l[0]-1));
    out_var_biased[blockIdx.x] = static_cast<scalar_t>(m2n_l[0]/num_item_l[0]);
  }
}
  

std::vector<at::Tensor> welford_mean_var_CUDA(const at::Tensor input) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  auto space_size = get_tensor_spatial_size(input);
  auto scalar_type = promote_scalartype(input);

  at::TensorOptions t_op(input);
  t_op.dtype(scalar_type);
  at::Tensor out_var = at::native::empty({feature_size}, t_op);
  at::Tensor out_var_biased = at::native::empty({feature_size}, t_op);
  at::Tensor out_mean = at::native::empty({feature_size}, t_op);

  int block_x = 16;
  const dim3 block(block_x, batch_size);
  const dim3 grid(feature_size);
  
  // shared memory used for reduce on mean, var, num_elements;
  int smem_size = batch_size * block_x * (sizeof(int) + 2 * get_element_data_size(input, true));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "welford_mean_var_kernel", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    welford_kernel<scalar_t, accscalar_t, accscalar_t><<<grid, block, smem_size>>>(
        input.data<scalar_t>(),
        out_mean.data<accscalar_t>(),
        out_var.data<accscalar_t>(),
        out_var_biased.data<accscalar_t>(),
        batch_size,
        feature_size,
        space_size);
  }));

  return {out_mean, out_var, out_var_biased};
}

at::Tensor batchnorm_forward_CUDA(
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const at::Tensor weight,
    const at::Tensor shift,
    const float eps) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);
  at::Tensor out = at::empty_like(input);

  auto space_size = get_tensor_spatial_size(input);

  const dim3 block(512);
  // TODO(jie): should I do 1 block per feature?
  const dim3 grid(feature_size, batch_size);
  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_forward", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    batchnorm_forward_kernel<scalar_t, accscalar_t><<<grid, block>>>(
        input.data<scalar_t>(),
        mean.data<accscalar_t>(),
        var.data<accscalar_t>(),
        weight.data<scalar_t>(),
        shift.data<scalar_t>(),
        out.data<scalar_t>(),
        space_size,
        eps);
  }));
  
  return out;
}

std::vector<at::Tensor> reduce_bn_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const float eps)
{
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  auto scalar_type = promote_scalartype(input);

  at::TensorOptions t_op(mean);
  t_op.dtype(scalar_type);
  at::Tensor mean_dy = at::native::empty({feature_size}, t_op);
  at::Tensor mean_dy_xmu = at::native::empty({feature_size}, t_op);
  at::TensorOptions grad_t_op(mean);
  grad_t_op.dtype(input.dtype());
  at::Tensor grad_weight = at::native::empty({feature_size}, grad_t_op);
  at::Tensor grad_bias = at::native::empty({feature_size}, grad_t_op);

  auto space_size = get_tensor_spatial_size(input);

  int block_x = 16;
  const dim3 block(block_x, batch_size);
  const dim3 grid(feature_size);
  // shared memory used for reduce on sum_dy, sum_dy_xmu;
  int smem_size = batch_size * block_x * 2 * get_element_data_size(input, true);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_backward_reduce", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    reduce_bn_kernel<scalar_t, accscalar_t><<<grid, block, smem_size>>>(
        input.data<scalar_t>(),
        grad_output.data<scalar_t>(),
        mean.data<accscalar_t>(),
        var.data<accscalar_t>(),
        mean_dy.data<accscalar_t>(),
        mean_dy_xmu.data<accscalar_t>(),
        grad_weight.data<scalar_t>(),
        grad_bias.data<scalar_t>(),
        batch_size,
        feature_size,
        space_size,
        eps);
  }));
  
  return {mean_dy, mean_dy_xmu, grad_weight, grad_bias};
}

at::Tensor batchnorm_backward_CUDA(
    const at::Tensor grad_output,
    const at::Tensor input,
    const at::Tensor mean,
    const at::Tensor var,
    const at::Tensor weight,
    const at::Tensor mean_dy,
    const at::Tensor mean_dy_xmu,
    const float eps) {
  const auto batch_size = input.size(0);
  const auto feature_size = input.size(1);

  at::Tensor grad_input = at::empty_like(input);

  auto space_size = get_tensor_spatial_size(input);

  const dim3 block(512);
  // TODO(jie): should I do 1 block per feature?
  const dim3 grid(feature_size, batch_size);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.type(), "batchnorm_backward", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    batchnorm_backward_kernel<scalar_t, accscalar_t><<<grid, block>>>(
        grad_output.data<scalar_t>(),
        input.data<scalar_t>(),
        mean.data<accscalar_t>(),
        var.data<accscalar_t>(),
        weight.data<scalar_t>(),
        mean_dy.data<accscalar_t>(),
        mean_dy_xmu.data<accscalar_t>(),
        grad_input.data<scalar_t>(),
        space_size,
        eps);
  }));
  
  return grad_input;
}

std::vector<at::Tensor> welford_parallel_CUDA(const at::Tensor mean_feature_nodes, const at::Tensor var_biased, int numel) {
  const auto feature_size = mean_feature_nodes.size(0);
  const auto node_size = mean_feature_nodes.size(1);

  at::TensorOptions t_op(var_biased);
  at::Tensor out_var = at::native::empty({feature_size}, t_op);
  at::Tensor out_var_biased = at::empty_like(out_var);
  at::Tensor out_mean = at::empty_like(out_var);

  // TODO(jie): 
  const dim3 block(node_size);
  const dim3 grid(feature_size);
  // shared memory used for reduce on mean, var, num_elements;
  int smem_size = node_size * (sizeof(int) + 2 * get_element_data_size(mean_feature_nodes, true));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(mean_feature_nodes.type(), "welford_parallel_kernel", ([&] {
    using accscalar_t = at::acc_type<scalar_t, true>;
    welford_kernel_parallel<scalar_t, accscalar_t><<<grid, block, smem_size>>>(
        mean_feature_nodes.data<scalar_t>(),
        var_biased.data<scalar_t>(),
        out_mean.data<scalar_t>(),
        out_var.data<scalar_t>(),
        out_var_biased.data<scalar_t>(),
        node_size,
        feature_size,
        numel);
  }));

  return {out_mean, out_var, out_var_biased};
}
