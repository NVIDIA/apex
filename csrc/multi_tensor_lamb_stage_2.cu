#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

using MATH_T = float;

// Step 2 reads in 'update' value and per-tensor param_norm and update_norm.
// It computes new parameter value.
template<typename T, typename UPD_T>
struct LAMBStage2Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<2>& tl,
    const float* per_tensor_param_norm,
    const float* per_tensor_update_norm,
    const float learning_rate,
    const float decay,
    bool use_nvlamb)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    MATH_T ratio = learning_rate;
    // nvlamb: apply adaptive learning rate to all parameters
    // otherwise, only apply to those with non-zero weight decay
    if (use_nvlamb || (decay != 0.0))
    {
      float param_norm = per_tensor_param_norm[tensor_num];
      float update_norm = per_tensor_update_norm[tensor_num];
      ratio = (update_norm != 0.0f && param_norm != 0.0f) ? learning_rate * (param_norm / update_norm) : learning_rate;
    }

    T* p = (T*)tl.addresses[0][tensor_loc];
    p += chunk_idx*chunk_size;

    UPD_T* update = (UPD_T*)tl.addresses[1][tensor_loc];
    update += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      T r_p[ILP];
      UPD_T r_update[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_p[ii] = p[i];
          r_update[ii] = update[i];
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        r_p[ii] = r_p[ii] - (ratio*(T)r_update[ii]);
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          p[i] = r_p[ii];
        }
      }
    }
  }
};

void multi_tensor_lamb_stage2_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor per_tensor_param_norm,
  at::Tensor per_tensor_update_norm,
  const float lr,
  const float weight_decay,
  at::optional<bool> use_nvlamb_python)
{
  bool use_nvlamb = use_nvlamb_python.has_value() ? use_nvlamb_python.value() : false;

  using namespace at;

  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "lamb_stage_2",
    DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 1, "lamb_stage_2",
      multi_tensor_apply<2>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        LAMBStage2Functor<scalar_t_0, scalar_t_1>(),
        per_tensor_param_norm.DATA_PTR<float>(),
        per_tensor_update_norm.DATA_PTR<float>(),
        lr,
	weight_decay,
	use_nvlamb); ))

  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
