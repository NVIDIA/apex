#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "multi_tensor_apply.cuh"
#include "type_shim.h"

#define BLOCK_SIZE 1024
#define ILP 4

typedef enum {
  ADAGRAD_MODE_0 = 0, // L2 regularization mode.
  ADAGRAD_MODE_1 = 1, // AdamW-style weight decay.

} adagradMode_t;

using MATH_T = float;

template <typename T> struct AdagradFunctor {
  __device__ __forceinline__ void
  operator()(int chunk_size, volatile int *noop_gmem, TensorListMetadata<3> &tl,
             const float epsilon, const float lr, adagradMode_t mode,
             const float weight_decay) {
    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    T *g = (T *)tl.addresses[0][tensor_loc];
    g += chunk_idx * chunk_size;

    T *p = (T *)tl.addresses[1][tensor_loc];
    p += chunk_idx * chunk_size;

    T *h = (T *)tl.addresses[2][tensor_loc];
    h += chunk_idx * chunk_size;

    n -= chunk_idx * chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for (int i_start = 0; i_start < n && i_start < chunk_size;
         i_start += blockDim.x * ILP) {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_h[ILP];
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_h[ii] = h[i];
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_h[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        if (mode == ADAGRAD_MODE_0) { // L2
          r_g[ii] = r_g[ii] + weight_decay * r_p[ii];
          r_h[ii] = r_h[ii] + r_g[ii] * r_g[ii];
          r_p[ii] = r_p[ii] - lr * (r_g[ii] / (sqrtf(r_h[ii]) + epsilon));
        } else { // AdamW-style
          r_h[ii] = r_h[ii] + r_g[ii] * r_g[ii];
          r_p[ii] = r_p[ii] - lr * (r_g[ii] / (sqrtf(r_h[ii]) + epsilon) + weight_decay * r_p[ii]);
        }
      }
#pragma unroll
      for (int ii = 0; ii < ILP; ii++) {
        int i = i_start + threadIdx.x + ii * blockDim.x;
        if (i < n && i < chunk_size) {
          p[i] = r_p[ii];
          h[i] = r_h[ii];
        }
      }
    }
  }
};

void multi_tensor_adagrad_cuda(
    int chunk_size, at::Tensor noop_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, const float lr,
    const float epsilon, const int mode, const float weight_decay) {
  using namespace at;

  // Assume single type across p,g,h now
  DISPATCH_DOUBLE_FLOAT_AND_HALF(
      tensor_lists[0][0].scalar_type(), 0, "adagrad",
      multi_tensor_apply<3>(BLOCK_SIZE, chunk_size, noop_flag, tensor_lists,
                            AdagradFunctor<scalar_t_0>(), epsilon, lr,
                            (adagradMode_t)mode, weight_decay);)

  AT_CUDA_CHECK(hipGetLastError());
}
