#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 1024
#define ILP 4

typedef enum{
  MOMENT_MODE_0   =0, // Novograd paper mode, momentum caculation with denom then decay inside
  MOMENT_MODE_1   =1  // Decoupled weight decay mode
} momentMode_t;

void multi_tensor_norm_out_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor out,
  const float alpha,
  const float beta,
  const int norm_type);

using MATH_T = float;

template<typename T>
struct NovoGradFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    const float beta1,
    const float beta2,
    const float beta3,
    const float beta1_correction,
    const float beta2_correction,
    const float epsilon,
    const float lr,
    momentMode_t m_mode,
    const float decay,
    const float* per_tensor_grad_norm)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float grad_norm = per_tensor_grad_norm[tensor_num];

    T* g = (T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // see note in multi_tensor_scale_kernel.cu
    for(int i_start = 0;
            i_start < n && i_start < chunk_size;
            i_start += blockDim.x*ILP)
    {
      MATH_T r_g[ILP];
      MATH_T r_p[ILP];
      MATH_T r_m[ILP];
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          r_g[ii] = g[i];
          r_p[ii] = p[i];
          r_m[ii] = m[i];
        } else {
          r_g[ii] = MATH_T(0);
          r_p[ii] = MATH_T(0);
          r_m[ii] = MATH_T(0);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        if (m_mode == MOMENT_MODE_0) {
          MATH_T next_v_unbiased = grad_norm / beta2_correction;
          MATH_T denom = next_v_unbiased + epsilon;
          r_g[ii] = (r_g[ii] / denom) + (decay * r_p[ii]);
          r_m[ii] = beta1 * r_m[ii] + beta3 * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          r_p[ii] = r_p[ii] - (lr * next_m_unbiased);
        }
        else {
          r_m[ii] = beta1 * r_m[ii] + beta3 * r_g[ii];
          MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
          MATH_T next_v_unbiased = grad_norm / beta2_correction;
          MATH_T denom = next_v_unbiased + epsilon;
          MATH_T update = (next_m_unbiased / denom) + (decay * r_p[ii]);
          r_p[ii] = r_p[ii] - (lr * update);
        }
      }
#pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          p[i] = r_p[ii];
          m[i] = r_m[ii];
        }
      }
    }
  }
};

void multi_tensor_novograd_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor grad_norms,
  const float lr,
  const float beta1,
  const float beta2,
  const float epsilon,
  const int step,
  const int bias_correction,
  const float weight_decay,
  const int grad_averaging,
  const int moment_mode,
  const int norm_type)
{
  using namespace at;

  // Handle bias correction mode
  float bias_correction1 = 1.0f, bias_correction2 = 1.0f;
  if (bias_correction == 1) {
    bias_correction1 = 1 - std::pow(beta1, step);
    bias_correction2 = std::sqrt(1 - std::pow(beta2, step));
  }

  // Handle grad averaging mode
  float beta3 = 1;
  if (grad_averaging == 1) beta3 = 1 - beta1;

  std::vector<std::vector<at::Tensor>> grad_list(tensor_lists.begin(), tensor_lists.begin()+1);

  // Compute and update grad norm
  // Here use a per tensor norm, and blend new norm(n) and old norm(gn) by
  // L-2: gn = sqrt(a * gn^2 + b * n^2)
  // L-inf: gn = a * gn + b * n
  multi_tensor_norm_out_cuda(chunk_size, noop_flag, grad_list, grad_norms, beta2, (1.0f - beta2), norm_type);

  // Assume single type across p,g,m1,m2 now
  DISPATCH_DOUBLE_FLOAT_AND_HALF_AND_BFLOAT16(
    tensor_lists[0][0].scalar_type(), 0, "novograd",
    multi_tensor_apply<3>(
      BLOCK_SIZE,
      chunk_size,
      noop_flag,
      tensor_lists,
      NovoGradFunctor<scalar_t_0>(),
      beta1,
      beta2,
      beta3, // 1-beta1 or 1 depends on averaging mode
      bias_correction1,
      bias_correction2,
      epsilon,
      lr,
      (momentMode_t) moment_mode,
      weight_decay,
      grad_norms.DATA_PTR<float>()); )

  AT_CUDA_CHECK(hipGetLastError());

}
