#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>
// Stringstream is a big hammer, but I want to rely on operator<< for dtype.
#include <sstream>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template<typename in_t, typename out_t>
struct ScaleFunctor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<2>& tl,
    float scale)
  {
    // I'd like this kernel to propagate infs/nans.
    // if(*noop_gmem == 1)
    //   return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    in_t* in = (in_t*)tl.addresses[0][tensor_loc];
    in += chunk_idx*chunk_size;
   
    out_t* out = (out_t*)tl.addresses[1][tensor_loc];
    out += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // Non-divergent exit condition for __syncthreads, not necessary here
    float incoming_vals[ILP];
    for(int i_start = 0;
        i_start < n && i_start < chunk_size;
        i_start += blockDim.x*ILP)
    {
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        incoming_vals[ii] = 0;
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
          incoming_vals[ii] = static_cast<float>(in[i]);
      }

      // note for clarification to future michael:
      // From a pure memory dependency perspective, there's likely no point unrolling
      // the write loop, since writes just fire off once their LDGs arrive.
      // Put another way, the STGs are dependent on the LDGs, but not on each other.
      // There is still compute ILP benefit from unrolling the loop though.
      #pragma unroll
      for(int ii = 0; ii < ILP; ii++)
      {
        int i = i_start + threadIdx.x + ii*blockDim.x;
        if(i < n && i < chunk_size)
        {
          out[i] = static_cast<out_t>(incoming_vals[ii]*scale);
          if(!isfinite(incoming_vals[ii]))
            *noop_gmem = 1; // Blindly fire off a write.  These will race but that's ok.
        }
      }
    }
  }
};

void multi_tensor_scale_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  float scale)
{
  using namespace at;
  // The output (downscaled) type is always float.
  // If build times suffer, think about where to put this dispatch,
  // and what logic should be moved out of multi_tensor_apply.

  DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 0, "multi_tensor_scale_cuda",
    DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 1, "multi_tensor_scale_cuda",
      multi_tensor_apply<2>(
        BLOCK_SIZE,
        chunk_size,
        noop_flag,
        tensor_lists,
        ScaleFunctor<scalar_t_0, scalar_t_1>(),
        scale); ))
  AT_CUDA_CHECK(hipGetLastError());

  // AT_CUDA_CHECK(hipDeviceSynchronize());
}
