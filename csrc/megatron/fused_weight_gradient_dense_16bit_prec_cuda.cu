#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "type_shim.h"


// BF16 inputs and BF16 accumulation
void gemmex_wrapper_fp16(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    at::BFloat16* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_16BF,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// FP16 inputs and FP16 accumulation
void gemmex_wrapper_fp16(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    at::Half* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_16F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

template <typename T>
void wgrad_gemm_accum_fp16_cuda(T *input, T *d_output, T *d_weight, int in_dim, int hidden_dim, int out_dim) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha = 1.0;
    const float beta  = 1.0;

    gemmex_wrapper_fp16(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,
        out_dim,
        hidden_dim,
        &alpha,
        input,
        in_dim,
        d_output,
        out_dim,
        &beta,
        d_weight,
        in_dim);
} 

template void wgrad_gemm_accum_fp16_cuda<at::Half>(at::Half *input, at::Half *d_output, at::Half *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp16_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, at::BFloat16 *d_weight, int in_dim, int hidden_dim, int out_dim);

void wgrad_gemm_accum_fp16_cuda_stub(
  at::Tensor &input,
  at::Tensor &d_output,
  at::Tensor &d_weight
) {
    at::Tensor input_2d, d_output_2d;
    // input tensor: collapse to the first dim
    auto in_sizes = input.sizes();
    if (input.dim() > 2) {
        input_2d = input.view({-1, in_sizes[in_sizes.size() - 1]});
    } else {
        input_2d = input;
    }
    // d_output tensor: collapse to the first dim
    auto d_out_sizes = d_output.sizes();
    if (d_output.dim() > 2) {
        d_output_2d = d_output.view({-1, d_out_sizes[d_out_sizes.size() - 1]});
    } else {
        d_output_2d = d_output;
    }

    const int hidden_dim = input_2d.size(0);
    const int in_dim = input_2d.size(1);
    const int out_dim = d_weight.size(0);

    DISPATCH_HALF_AND_BFLOAT(input_2d.scalar_type(), "wgrad_gemm_accum_fp16",
        wgrad_gemm_accum_fp16_cuda<scalar_t>(
            input_2d.data_ptr<scalar_t>(),
            d_output_2d.data_ptr<scalar_t>(),
            d_weight.data_ptr<scalar_t>(),
            in_dim,
            hidden_dim,
            out_dim);
    );
}
