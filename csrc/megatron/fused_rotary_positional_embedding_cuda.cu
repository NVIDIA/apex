#include "hip/hip_runtime.h"
/* coding=utf-8
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>

#include "fused_rotary_positional_embedding.h"
#include "type_shim.h"

namespace fused_rope {

torch::Tensor fwd_cuda(const torch::Tensor &input, const torch::Tensor &freqs,
                       const bool transpose_output) {
  // input sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = input.size(0);
  const int b = input.size(1);
  const int h = input.size(2);
  const int d = input.size(3);
  // input strides
  const int stride_s = input.stride(0);
  const int stride_b = input.stride(1);
  const int stride_h = input.stride(2);
  const int stride_d = input.stride(3);
  // freqs' shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = freqs.size(3);

  // output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor output;
  if (transpose_output) {
    output = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    output = torch::empty({s, b, h, d}, act_options);
  }
  // output strides
  const int o_stride_s = output.stride(0);
  const int o_stride_b = output.stride(1);
  const int o_stride_h = output.stride(2);
  const int o_stride_d = output.stride(3);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      input.scalar_type(), 0, "dispatch_fused_rope_forward",
      dispatch_fused_rope_forward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d, input.data_ptr<scalar_t_0>(),
          freqs.data_ptr<float>(), output.data_ptr<scalar_t_0>()););
  return output;
}

torch::Tensor bwd_cuda(const torch::Tensor &output_grads,
                       const torch::Tensor &freqs,
                       const bool transpose_output) {
  // output_grads sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = output_grads.size(0);
  const int b = output_grads.size(1);
  const int h = output_grads.size(2);
  const int d = output_grads.size(3);
  // output_grads strides
  const int stride_s = output_grads.stride(0);
  const int stride_b = output_grads.stride(1);
  const int stride_h = output_grads.stride(2);
  const int stride_d = output_grads.stride(3);
  // freqs' shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = freqs.size(3);

  auto act_options = output_grads.options().requires_grad(false);
  torch::Tensor input_grads;
  if (transpose_output) {
    input_grads = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    input_grads = torch::empty({s, b, h, d}, act_options);
  }
  const int o_stride_s = input_grads.stride(0);
  const int o_stride_b = input_grads.stride(1);
  const int o_stride_h = input_grads.stride(2);
  const int o_stride_d = input_grads.stride(3);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      output_grads.scalar_type(), 0, "dispatch_fused_rope_backward",
      dispatch_fused_rope_backward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d,
          output_grads.data_ptr<scalar_t_0>(), freqs.data_ptr<float>(),
          input_grads.data_ptr<scalar_t_0>()););
  return input_grads;
}

#define DISPATCH_FUSED_ROPE_TYPES(TYPE1, TYPE2, NAME, ...)                     \
  switch (TYPE1) {                                                             \
  case at::ScalarType::Float: {                                                \
    using scalar_t_0 = float;                                                  \
    switch (TYPE2) {                                                           \
    case at::ScalarType::Float: {                                              \
      using scalar_t_1 = float;                                                \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      TORCH_CHECK(false, #NAME, " not supported for '", toString(TYPE1),       \
                  "' with '", toString(TYPE2), "'");                           \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  case at::ScalarType::Half: {                                                 \
    using scalar_t_0 = at::Half;                                               \
    switch (TYPE2) {                                                           \
    case at::ScalarType::Float: {                                              \
      using scalar_t_1 = float;                                                \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case at::ScalarType::Half: {                                               \
      using scalar_t_1 = at::Half;                                             \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      TORCH_CHECK(false, #NAME, " not supported for '", toString(TYPE1),       \
                  "' with '", toString(TYPE2), "'");                           \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  case at::ScalarType::BFloat16: {                                             \
    using scalar_t_0 = at::BFloat16;                                           \
    switch (TYPE2) {                                                           \
    case at::ScalarType::Float: {                                              \
      using scalar_t_1 = float;                                                \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    case at::ScalarType::BFloat16: {                                           \
      using scalar_t_1 = at::BFloat16;                                         \
      __VA_ARGS__;                                                             \
      break;                                                                   \
    }                                                                          \
    default:                                                                   \
      TORCH_CHECK(false, #NAME, " not supported for '", toString(TYPE1),       \
                  "' with '", toString(TYPE2), "'");                           \
    }                                                                          \
    break;                                                                     \
  }                                                                            \
  default:                                                                     \
    TORCH_CHECK(false, #NAME, " not supported for '", toString(TYPE1),         \
                "' with '", toString(TYPE2), "'");                             \
  }

torch::Tensor fwd_cached_cuda(const torch::Tensor &input,
                              const torch::Tensor &cos,
                              const torch::Tensor &sin,
                              const bool transpose_output) {
  // input sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = input.size(0);
  const int b = input.size(1);
  const int h = input.size(2);
  const int d = input.size(3);
  // input strides
  const int stride_s = input.stride(0);
  const int stride_b = input.stride(1);
  const int stride_h = input.stride(2);
  const int stride_d = input.stride(3);
  // cos/sin's shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = cos.size(3);

  // output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor output;
  if (transpose_output) {
    output = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    output = torch::empty({s, b, h, d}, act_options);
  }
  // output strides
  const int o_stride_s = output.stride(0);
  const int o_stride_b = output.stride(1);
  const int o_stride_h = output.stride(2);
  const int o_stride_d = output.stride(3);

  DISPATCH_FUSED_ROPE_TYPES(
      input.scalar_type(), cos.scalar_type(),
      "dispatch_fused_rope_cached_forward",
      dispatch_fused_rope_cached_forward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d, input.data_ptr<scalar_t_0>(),
          cos.data_ptr<scalar_t_1>(), sin.data_ptr<scalar_t_1>(),
          output.data_ptr<scalar_t_0>()););
  return output;
}

torch::Tensor bwd_cached_cuda(const torch::Tensor &output_grads,
                              const torch::Tensor &cos,
                              const torch::Tensor &sin,
                              const bool transpose_output) {
  // output_grads sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = output_grads.size(0);
  const int b = output_grads.size(1);
  const int h = output_grads.size(2);
  const int d = output_grads.size(3);
  // output_grads strides
  const int stride_s = output_grads.stride(0);
  const int stride_b = output_grads.stride(1);
  const int stride_h = output_grads.stride(2);
  const int stride_d = output_grads.stride(3);
  // cos/sin's shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = cos.size(3);

  auto act_options = output_grads.options().requires_grad(false);
  torch::Tensor input_grads;
  if (transpose_output) {
    input_grads = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    input_grads = torch::empty({s, b, h, d}, act_options);
  }
  const int o_stride_s = input_grads.stride(0);
  const int o_stride_b = input_grads.stride(1);
  const int o_stride_h = input_grads.stride(2);
  const int o_stride_d = input_grads.stride(3);

  DISPATCH_FUSED_ROPE_TYPES(
      output_grads.scalar_type(), cos.scalar_type(),
      "dispatch_fused_rope_cached_backward",
      dispatch_fused_rope_cached_backward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d,
          output_grads.data_ptr<scalar_t_0>(), cos.data_ptr<scalar_t_1>(),
          sin.data_ptr<scalar_t_1>(), input_grads.data_ptr<scalar_t_0>()););
  return input_grads;
}

torch::Tensor fwd_thd_cuda(const torch::Tensor &input,
                           const torch::Tensor &cu_seqlens,
                           const torch::Tensor &freqs) {
  // input sizes: (t, h, d)
  // t: cumulative sum of sequence lengths
  // h: head num
  // d: dim of each head
  const int t = input.size(0);
  const int h = input.size(1);
  const int d = input.size(2);
  // input strides
  const int stride_t = input.stride(0);
  const int stride_h = input.stride(1);
  const int stride_d = input.stride(2);
  // batch size
  const int b = cu_seqlens.size(0) - 1;
  // freqs' shape is (max_s, 1, 1, d2)
  const int max_s = freqs.size(0);
  const int d2 = freqs.size(3);

  // output
  auto act_options = input.options().requires_grad(false);
  auto output = torch::empty({t, h, d}, act_options);
  // output strides
  const int o_stride_t = output.stride(0);
  const int o_stride_h = output.stride(1);
  const int o_stride_d = output.stride(2);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      input.scalar_type(), 0, "dispatch_fused_rope_thd_forward",
      dispatch_fused_rope_thd_forward(
          max_s, b, h, d, d2, stride_t, stride_h, stride_d, o_stride_t,
          o_stride_h, o_stride_d, input.data_ptr<scalar_t_0>(),
          cu_seqlens.data_ptr<int>(), freqs.data_ptr<float>(),
          output.data_ptr<scalar_t_0>()););
  return output;
}

torch::Tensor bwd_thd_cuda(const torch::Tensor &output_grads,
                           const torch::Tensor &cu_seqlens,
                           const torch::Tensor &freqs) {
  // output_grads sizes: (t, h, d)
  // t: cumulative sum of sequence lengths
  // h: head num
  // d: dim of each head
  const int t = output_grads.size(0);
  const int h = output_grads.size(1);
  const int d = output_grads.size(2);
  // output_grads strides
  const int stride_t = output_grads.stride(0);
  const int stride_h = output_grads.stride(1);
  const int stride_d = output_grads.stride(2);
  // batch size
  const int b = cu_seqlens.size(0) - 1;
  // freqs' shape is (max_s, 1, 1, d2)
  const int max_s = freqs.size(0);
  const int d2 = freqs.size(3);

  auto act_options = output_grads.options().requires_grad(false);
  auto input_grads = torch::empty({t, h, d}, act_options);
  const int o_stride_t = input_grads.stride(0);
  const int o_stride_h = input_grads.stride(1);
  const int o_stride_d = input_grads.stride(2);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      output_grads.scalar_type(), 0, "dispatch_fused_rope_thd_backward",
      dispatch_fused_rope_thd_backward(
          max_s, b, h, d, d2, stride_t, stride_h, stride_d, o_stride_t,
          o_stride_h, o_stride_d, output_grads.data_ptr<scalar_t_0>(),
          cu_seqlens.data_ptr<int>(), freqs.data_ptr<float>(),
          input_grads.data_ptr<scalar_t_0>()););
  return input_grads;
}

} // end namespace fused_rope
