#include "hip/hip_runtime.h"
/* coding=utf-8
 * Copyright (c) 2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>

#include "fused_rotary_positional_embedding.h"
#include "type_shim.h"

namespace fused_rope {

torch::Tensor fwd_cuda(const torch::Tensor &input, const torch::Tensor &cos,
                       const torch::Tensor &sin, const bool transpose_output) {
  // input sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = input.size(0);
  const int b = input.size(1);
  const int h = input.size(2);
  const int d = input.size(3);
  // input strides
  const int stride_s = input.stride(0);
  const int stride_b = input.stride(1);
  const int stride_h = input.stride(2);
  const int stride_d = input.stride(3);
  // cos/sin's shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = cos.size(3);

  // output
  auto act_options = input.options().requires_grad(false);
  torch::Tensor output;
  if (transpose_output) {
    output = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    output = torch::empty({s, b, h, d}, act_options);
  }
  // output strides
  const int o_stride_s = output.stride(0);
  const int o_stride_b = output.stride(1);
  const int o_stride_h = output.stride(2);
  const int o_stride_d = output.stride(3);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      input.scalar_type(), 0, "dispatch_fused_rope_forward",
      dispatch_fused_rope_forward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d, input.data_ptr<scalar_t_0>(),
          cos.data_ptr<scalar_t_0>(), sin.data_ptr<scalar_t_0>(),
          output.data_ptr<scalar_t_0>()););
  return output;
}

torch::Tensor bwd_cuda(const torch::Tensor &output_grads,
                       const torch::Tensor &cos, const torch::Tensor &sin,
                       const bool transpose_output) {
  // output_grads sizes: (s, b, h, d)
  // s: sequence length
  // b: batch size
  // h: head num
  // d: dim of each head
  const int s = output_grads.size(0);
  const int b = output_grads.size(1);
  const int h = output_grads.size(2);
  const int d = output_grads.size(3);
  // output_grads strides
  const int stride_s = output_grads.stride(0);
  const int stride_b = output_grads.stride(1);
  const int stride_h = output_grads.stride(2);
  const int stride_d = output_grads.stride(3);
  // cos/sin's shape is always (s, 1, 1, d2), so the strides are same under
  // different memory formats
  const int d2 = cos.size(3);

  auto act_options = output_grads.options().requires_grad(false);
  torch::Tensor input_grads;
  if (transpose_output) {
    input_grads = torch::empty({b, s, h, d}, act_options).transpose(0, 1);
  } else {
    input_grads = torch::empty({s, b, h, d}, act_options);
  }
  const int o_stride_s = input_grads.stride(0);
  const int o_stride_b = input_grads.stride(1);
  const int o_stride_h = input_grads.stride(2);
  const int o_stride_d = input_grads.stride(3);

  DISPATCH_FLOAT_HALF_AND_BFLOAT(
      output_grads.scalar_type(), 0, "dispatch_fused_rope_backward",
      dispatch_fused_rope_backward(
          s, b, h, d, d2, stride_s, stride_b, stride_h, stride_d, o_stride_s,
          o_stride_b, o_stride_h, o_stride_d,
          output_grads.data_ptr<scalar_t_0>(), cos.data_ptr<scalar_t_0>(),
          sin.data_ptr<scalar_t_0>(), input_grads.data_ptr<scalar_t_0>());)
  return input_grads;
}
}  // end namespace fused_rope
