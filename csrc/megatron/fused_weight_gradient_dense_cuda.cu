#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <cstring>

#include <torch/extension.h>
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

/* Includes, cuda */
#include <hipblas.h>
#include <hip/hip_runtime.h>

#include "type_shim.h"


// BF16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::BFloat16* A,
    int lda,
    at::BFloat16* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16BF,
      lda,
      B,
      HIP_R_16BF,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// FP16 Tensor core wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float* alpha,
    at::Half* A,
    int lda,
    at::Half* B,
    int ldb,
    const float* beta,
    float* C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_16F,
      lda,
      B,
      HIP_R_16F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

// FP32 wrapper around cublas GEMMEx
void gemmex_wrapper(
    hipblasHandle_t handle,
    hipblasOperation_t transa,
    hipblasOperation_t transb,
    int m,
    int n,
    int k,
    const float *alpha,
    float *A,
    int lda,
    float *B,
    int ldb,
    const float *beta,
    float *C,
    int ldc) {
  TORCH_CUDABLAS_CHECK(hipblasGemmEx(
      handle,
      transa,
      transb,
      m,
      n,
      k,
      alpha,
      A,
      HIP_R_32F,
      lda,
      B,
      HIP_R_32F,
      ldb,
      beta,
      C,
      HIP_R_32F,
      ldc,
      HIP_R_32F,
      CUBLAS_GEMM_DEFAULT_TENSOR_OP));
}

template <typename T>
void wgrad_gemm_accum_fp32_cuda(T *input, T *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim) {
    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
    hipStream_t stream;
    hipblasGetStream(handle, &stream);
    const float alpha = 1.0;
    const float beta  = 1.0;

    gemmex_wrapper(
        handle,
        HIPBLAS_OP_N,
        HIPBLAS_OP_T,
        in_dim,
        out_dim,
        hidden_dim,
        &alpha,
        input,
        in_dim,
        d_output,
        out_dim,
        &beta,
        d_weight,
        in_dim);
}

template void wgrad_gemm_accum_fp32_cuda<at::Half>(at::Half *input, at::Half *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<at::BFloat16>(at::BFloat16 *input, at::BFloat16 *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);
template void wgrad_gemm_accum_fp32_cuda<float>(float *input, float *d_output, float *d_weight, int in_dim, int hidden_dim, int out_dim);


void wgrad_gemm_accum_fp32_cuda_stub(
  at::Tensor &input,
  at::Tensor &d_output,
  at::Tensor &d_weight
) {
    at::Tensor input_2d, d_output_2d;
    // input tensor: collapse to the first dim
    auto in_sizes = input.sizes();
    if (input.dim() > 2) {
        input_2d = input.view({-1, in_sizes[in_sizes.size() - 1]});
    } else {
        input_2d = input;
    }
    // d_output tensor: collapse to the first dim
    auto d_out_sizes = d_output.sizes();
    if (d_output.dim() > 2) {
        d_output_2d = d_output.view({-1, d_out_sizes[d_out_sizes.size() - 1]});
    } else {
        d_output_2d = d_output;
    }

    const int hidden_dim = input_2d.size(0);
    const int in_dim = input_2d.size(1);
    const int out_dim = d_weight.size(0);

    DISPATCH_FLOAT_HALF_AND_BFLOAT(input_2d.scalar_type(), 0, "wgrad_gemm_accum_fp32",
        wgrad_gemm_accum_fp32_cuda<scalar_t_0>(
            input_2d.data_ptr<scalar_t_0>(),
            d_output_2d.data_ptr<scalar_t_0>(),
            d_weight.data_ptr<float>(),
            in_dim,
            hidden_dim,
            out_dim);
    );
}
