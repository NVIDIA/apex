#include "hip/hip_runtime.h"
#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>
#include "ATen/TensorUtils.h"
#include "ATen/Type.h"
#include "ATen/AccumulateType.h"
#include <THC/THCGeneral.h>

#include "type_shim.h"

typedef enum{
    ADAM_MODE_0   =0, // eps under square root
    ADAM_MODE_1   =1  // eps outside square root
} adamMode_t;

template <typename T, typename GRAD_T>
__global__ void adam_cuda_kernel(
        T* __restrict__ p,
        GRAD_T* __restrict__ p_copy, // For mixed precision training, pass NULL if not needed
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
        //Assuming 2D grids and 2D blocks
        const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
        const int threadsPerBlock = blockDim.x * blockDim.y;
        const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
        const int i = (blockId * threadsPerBlock + threadIdInBlock);
        const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

        for (int j = i; j < tsize; j+=totThreads) {
                T scaled_grad = g[j]/grad_scale;
                m[j] = b1*m[j] + (1-b1)*scaled_grad;
                v[j] = b2*v[j] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(v[j] + eps);
                else // Mode 1
                    denom = sqrtf(v[j]) + eps;
                float update = (m[j]/denom) + (decay*p[j]);
                p[j] = p[j] - (step_size*update);
                if (p_copy != NULL) p_copy[j] = (GRAD_T) p[j];
        }
}

void fused_adam_cuda(
        at::Tensor & p,
        at::Tensor & p_copy,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//        using namespace at;

        //Get tensor size
        int tsize = p.numel();
        //Determine #threads and #blocks
        const int threadsPerBlock = 512;
        const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
        AT_ASSERTM(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
        //Constants
        float step_size = 0;
        if (bias_correction == 1) {
            const float bias_correction1 = 1 - std::pow(beta1, step);
            const float bias_correction2 = 1 - std::pow(beta2, step);
            step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
        }
        else {
            step_size = lr;
        }
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (g.scalar_type() == at::ScalarType::Half) {
//all other values should be fp32 for half gradients
            AT_ASSERTM(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dispatch is done on the gradient type
            using namespace at; // prevents "toString is undefined" errors
            DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel", 
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                adam_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.data<accscalar_t>(),
                        p_copy.numel() ? p_copy.data<scalar_t_0>() : NULL,
                        m.data<accscalar_t>(),
                        v.data<accscalar_t>(),
                        g.data<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
                )
      } else {
            using namespace at;
            DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                adam_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.data<scalar_t_0>(),
                        NULL, //don't output p_copy for fp32, it's wasted write
                        m.data<scalar_t_0>(),
                        v.data<scalar_t_0>(),
                        g.data<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
            );
      }
      THCudaCheck(hipGetLastError());

}
