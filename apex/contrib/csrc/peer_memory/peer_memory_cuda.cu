#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <list>
#include <cstdio>
#include <cassert>
#include <hip/hip_runtime_api.h>
#include <hip/hip_cooperative_groups.h>
#include "nccl.h"
namespace cg = cooperative_groups;

#define CUDACHECK(cmd) do {                         \
  hipError_t err = cmd;                            \
  if( err != hipSuccess ) {                        \
    char hostname[1024];                            \
    gethostname(hostname, 1024);                    \
    printf("%s: CUDA failure %s:%d '%s'\n",         \
         hostname,                                  \
        __FILE__,__LINE__,hipGetErrorString(err)); \
  }                                                 \
} while(0)

namespace {

/* Basic deleter function for from_blob function.
void deleter(void* ptr)
{
    printf("deleter(ptr=%p)\n",ptr);
    hipFree(ptr);
}
*/

template<class T>
at::Tensor blob_view(T* raw_ptr, std::vector<int64_t> shape, const at::TensorOptions& options, bool channels_last)
{
    size_t size = 1;
    std::vector<int64_t> strides(shape.size());
    if (channels_last) {
        assert(shape.size() == 4);
        strides[0] = shape[1]*shape[2]*shape[3];
        strides[1] = 1;
        strides[2] = shape[1]*shape[3];
        strides[3] = shape[1];
    } else {
        int idx = strides.size();
        for (auto it = shape.rbegin();  it != shape.rend();  ++it)
        {
	    strides[--idx] = size;
	    size *= *it;
        }
    }
    size *= sizeof(T);
    // TODO: Implement dynamic reuse of pooled peer memory.
    // We provide no deleter function because all peer memory allocations are static in this implementation.
    return torch::from_blob((void*)raw_ptr, shape, strides, 0L, options);
}

void tensor_shape(at::Tensor t, bool explicit_nhwc, int& N, int& C, int& H, int& W)
{
    if (t.dim() == 3) {
	N = 1;
        if (explicit_nhwc) {
            C = t.size(2);
            H = t.size(0);
            W = t.size(1);
        } else {
	    C = t.size(0);
    	    H = t.size(1);
    	    W = t.size(2);
        }
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            N = t.size(0);
            C = t.size(3);
            H = t.size(1);
            W = t.size(2);
        } else {
            N = t.size(0);
            C = t.size(1);
            H = t.size(2);
            W = t.size(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

void tensor_strides(at::Tensor t, bool explicit_nhwc, int& stride_N, int& stride_C, int& stride_H, int& stride_W)
{
    if (t.dim() == 3) {
        if (explicit_nhwc) {
            stride_C = t.stride(2);
            stride_H = t.stride(0);
            stride_W = t.stride(1);
        } else {
	    stride_C = t.stride(0);
    	    stride_H = t.stride(1);
    	    stride_W = t.stride(2);
        }
        stride_N = t.size(0)*t.size(1)*t.size(2);
    } else if (t.dim() == 4) {
        if (explicit_nhwc) {
            stride_N = t.stride(0);
            stride_C = t.stride(3);
            stride_H = t.stride(1);
            stride_W = t.stride(2);
        } else {
            stride_N = t.stride(0);
            stride_C = t.stride(1);
            stride_H = t.stride(2);
            stride_W = t.stride(3);
        }
    } else {
        printf("%s;%d - t.dim() must be either 3 or 4 (was %d)\n",__FILE__,__LINE__,t.dim());
        assert(t.dim() == 3 || t.dim() == 4);
    }
}

template<class T, bool is_HWC>
__device__ void strided_copy_kernel(
	T* dst, const int dst_stride_C, const int dst_stride_H, const int dst_stride_W, 
	const T* src, const int src_stride_C, const int src_stride_H, const int src_stride_W, 
	const int NC, const int NH, const int NW
	)
{
    size_t tot_num_threads = gridDim.x * blockDim.x;
    size_t thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    const size_t count = NC*NH*NW;
    for (size_t i = thread_id;  i < count;  i += tot_num_threads)
    {
	size_t c,h,w;
	if (is_HWC) {
	    c = i % NC;
	    w = i / NC;
	    h = w / NW;
	    w = w % NW;
	}
	else {
	    w = i % NW;
	    h = i / NW;
	    c = h / NH;
            h = h % NH;
	}
	size_t dst_off = c*dst_stride_C + h*dst_stride_H + w*dst_stride_W;
	size_t src_off = c*src_stride_C + h*src_stride_H + w*src_stride_W;
	dst[dst_off] = src[src_off];
    }
}

__device__ void checked_signal(
	volatile int* signal1_flag, volatile int* signal2_flag,
	const int v1, const int v2, const int v3, const int v4
	)
{
    cg::this_grid().sync();
    bool is_main_thread = (blockIdx.x == 0 && threadIdx.x == 0) ? true : false;
    if (is_main_thread) {
	// flush all writes to global memory
	__threadfence_system();
	// wait for top or bottom neighbor to clear signal
	register int r1, r2, r3, r4;
	bool top_zeroed=false, btm_zeroed=false, top_done=false, btm_done=false;
	do {
	    do {
		if (!top_zeroed) {
		    asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" : "=r"(r1), "=r"(r2), "=r"(r3), "=r"(r4) : "l"(signal1_flag) : "memory");
		    if (r1 != v1 || r2 != v2 || r3 != v3 || r4 != v4) top_zeroed = true;
		}
		if (!btm_zeroed) {
		    asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" : "=r"(r1), "=r"(r2), "=r"(r3), "=r"(r4) : "l"(signal2_flag) : "memory");
		    if (r1 != v1 || r2 != v2 || r3 != v3 || r4 != v4) btm_zeroed = true;
		}
	    } while((top_zeroed == top_done) && (btm_zeroed == btm_done));
	    if (!top_done && top_zeroed) {
		// signal to top neighbor my output is ready
		asm volatile("st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};" :: "l"(signal1_flag), "r"(v1), "r"(v2), "r"(v3), "r"(v4) : "memory");
		top_done = true;
	    }
	    if (!btm_done && btm_zeroed) {
		// signal to bottom neighbor my output is ready
		asm volatile("st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};" :: "l"(signal2_flag), "r"(v1), "r"(v2), "r"(v3), "r"(v4) : "memory");
		btm_done = true;
	    }
	} while (!top_done || !btm_done);
    }
}

__device__ void wait_for(
	volatile int* wait_flag,
	const int v1, const int v2, const int v3, const int v4
	)
{
    bool is_main_thread = (blockIdx.x == 0 && threadIdx.x == 0) ? true : false;
    if (is_main_thread) {
    	register int r1, r2, r3, r4;
	// wait for senders to signal their output is read
	do {
	    asm volatile("ld.volatile.global.v4.u32 {%0,%1,%2,%3}, [%4];" : "=r"(r1), "=r"(r2), "=r"(r3), "=r"(r4) : "l"(wait_flag) : "memory");
	} while (r1 != v1 || r2 != v2 || r3 != v3 || r4 != v4);
    }
    cg::this_grid().sync();  // all threads wait for main
}


__device__ void clear_flag(
	volatile int* wait_flag
	)
{
    cg::this_grid().sync();  // wait for all threads in kernel to finish
    bool is_main_thread = (blockIdx.x == 0 && threadIdx.x == 0) ? true : false;
    if (is_main_thread) {
	register int r1, r2, r3, r4;
	r1 = 0;  r2 = 0;  r3 = 0;  r4 = 0;
	asm volatile("st.volatile.global.v4.u32 [%0], {%1,%2,%3,%4};" :: "l"(wait_flag), "r"(r1), "r"(r2), "r"(r3), "r"(r4) : "memory");
    }
}

template<class T, bool is_HWC>
#if __CUDA_ARCH__ >= 700
__launch_bounds__(128, 16)
#endif
__global__ void push_pull_halos_1d_kernel(
        // top halo,
        const T* toh, int toh_stride_C, int toh_stride_H, int toh_stride_W,     // top output halo
        T* tox, int tox_stride_C, int tox_stride_H, int tox_stride_W,           // top output tx buffer
        T* tix, int tix_stride_C, int tix_stride_H, int tix_stride_W,           // top input tx buffer
        T* tih, int tih_stride_C, int tih_stride_H, int tih_stride_W,           // top input halo
        // btm halo
        const T* boh, int boh_stride_C, int boh_stride_H, int boh_stride_W,     // btm output halo
        T* box, int box_stride_C, int box_stride_H, int box_stride_W,           // btm output tx buffer
        T* bix, int bix_stride_C, int bix_stride_H, int bix_stride_W,           // btm input tx buffer
        T* bih, int bih_stride_C, int bih_stride_H, int bih_stride_W,           // btm input halo
        // dimensions
        int NC, int NH, int NW,
        // signals
        int* signal1_flag,
        int* signal2_flag,
        int* wait1_flag,
        int* wait2_flag
        )
{
    // push top output halo to transfer buffer
    strided_copy_kernel<T,is_HWC>(tox, tox_stride_C, tox_stride_H, tox_stride_W, toh, toh_stride_C, toh_stride_H, toh_stride_W, NC, NH, NW);
    // push btm output halo to transfer buffer
    strided_copy_kernel<T,is_HWC>(box, box_stride_C, box_stride_H, box_stride_W, boh, boh_stride_C, boh_stride_H, boh_stride_W, NC, NH, NW);
    // signal to top and btm neigbhbors that output halos are ready to be read
    // the choice of values for v1-v4 is arbitrary and does not matter, as long as all ranks use the same values
    checked_signal(signal1_flag, signal2_flag, -987751720, 840868300, -225529332, 281513358);
    // pull top halo from transfer buffer in peer memory to input
    wait_for(wait1_flag, -987751720, 840868300, -225529332, 281513358);
    strided_copy_kernel<T,is_HWC>(tih, tih_stride_C, tih_stride_H, tih_stride_W, tix, tix_stride_C, tix_stride_H, tix_stride_W, NC, NH, NW);
    clear_flag(wait1_flag);
    // pull btm halo from transfer buffer in peer memory to input
    wait_for(wait2_flag, -987751720, 840868300, -225529332, 281513358);
    strided_copy_kernel<T,is_HWC>(bih, bih_stride_C, bih_stride_H, bih_stride_W, bix, bix_stride_C, bix_stride_H, bix_stride_W, NC, NH, NW);
    clear_flag(wait2_flag);
}

__global__ void delay_kernel(int delay_nanoseconds, int* counter)
{
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // waste time while doing something compiler can't predict, thus preventing it from optimizing away this code.
        int new_counter = 0;
        double elapsed = 0;
        clock_t start = clock();
        do {
            clock_t now = clock();
            elapsed = (double)(now - start)*1e9 / CLOCKS_PER_SEC;
            ++new_counter;
        } while (elapsed < (double)delay_nanoseconds);
        *counter = new_counter;
    }
}

}

namespace apex { namespace contrib { namespace peer_memory {

int64_t allocate_raw(int64_t size)
{
    float* ptr = 0L;
    hipMalloc(&ptr, size);
    hipMemset(ptr, 0, size);
    return (int64_t)ptr;
}

void free_raw(int64_t raw)
{
    hipFree((void*)raw);
}

void zero(int64_t raw, int64_t size)
{
    hipMemset((void*)raw, 0, size);
}

at::Tensor get_raw_ipc_address(int64_t raw)
{
    hipIpcMemHandle_t mem_handle;
    CUDACHECK( hipIpcGetMemHandle(&mem_handle, (void*)raw) );
    const int n = sizeof(hipIpcMemHandle_t);
    auto address_tensor = torch::empty({n}, torch::dtype(torch::kUInt8));
    auto address_tensor_p = address_tensor.data_ptr<uint8_t>();
    memcpy(address_tensor_p, (uint8_t*)&mem_handle, n);
    return address_tensor;
}

std::vector<int64_t> get_raw_peers(at::Tensor ipc_addresses, int peer_rank, int64_t raw)
{
    int peer_group_size = ipc_addresses.size(0);
    std::vector<int64_t> results(peer_group_size);
    for (int i = 0;  i < peer_group_size;  ++i) {
        if (i != peer_rank) {
            hipIpcMemHandle_t mem_handle;
            memcpy(&mem_handle, ipc_addresses.index({i}).data_ptr<uint8_t>(), sizeof(hipIpcMemHandle_t));
            void* p = 0L;
            CUDACHECK( hipIpcOpenMemHandle((void**)&p, mem_handle, hipIpcMemLazyEnablePeerAccess) );
            results[i] = (int64_t)p;
        } else {
            results[i] = (int64_t)raw;
        }
    }
    return results;
}

at::Tensor blob_view_half(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<at::Half>((at::Half*)raw, shape, torch::dtype(torch::kFloat16).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_float(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<float>((float*)raw, shape, torch::dtype(torch::kFloat32).device(torch::kCUDA), channels_last);
}

at::Tensor blob_view_int(int64_t raw, std::vector<int64_t> shape, bool channels_last)
{
    return blob_view<int>((int*)raw, shape, torch::dtype(torch::kInt32).device(torch::kCUDA), channels_last);
}

void push_pull_halos_1d(
	bool diagnostics,
        bool explicit_nhwc,
        int numSM,                      // number of SMs to use
        at::Tensor top_out_halo,        // top output halo in sender device memory
        at::Tensor top_out_tx,          // top output transfer buffer in sender peer pool memory
	at::Tensor top_inp_tx,		// top input transfer buffer in top neighbor peer pool memory
        at::Tensor top_inp_halo,        // top input halo in receiver device memory
        at::Tensor btm_out_halo,        // btm output halo in sender device memory
        at::Tensor btm_out_tx,          // btm output transfer buffer in sender peer pool memory
	at::Tensor btm_inp_tx,		// btm input transfer buffer in btm neighbor peer pool memory
        at::Tensor btm_inp_halo,        // btm input halo in receiver device memory
        at::Tensor top_signal,          // top input signal in receiver device memory
        at::Tensor btm_signal,          // btm input signal in receiver device memory
        at::Tensor waits                // top and btm signals for this rank
        )
{
    // basic checks of inputs
    TORCH_CHECK(top_out_halo.is_cuda());
    TORCH_CHECK(top_out_tx.is_cuda());
    TORCH_CHECK(top_inp_tx.is_cuda());
    TORCH_CHECK(top_inp_halo.is_cuda());
    TORCH_CHECK(btm_out_halo.is_cuda());
    TORCH_CHECK(btm_out_tx.is_cuda());
    TORCH_CHECK(btm_inp_tx.is_cuda());
    TORCH_CHECK(btm_inp_halo.is_cuda());
    TORCH_CHECK(top_signal.is_cuda());
    TORCH_CHECK(btm_signal.is_cuda());
    TORCH_CHECK(waits.is_cuda());

    // shapes and strides
    int toh_N, toh_C, toh_H, toh_W;
    tensor_shape(top_out_halo, explicit_nhwc, toh_N, toh_C, toh_H, toh_W);
    int tox_N, tox_C, tox_H, tox_W;
    tensor_shape(top_out_tx, explicit_nhwc, tox_N, tox_C, tox_H, tox_W);
    int tix_N, tix_C, tix_H, tix_W;
    tensor_shape(top_inp_tx, explicit_nhwc, tix_N, tix_C, tix_H, tix_W);
    int tih_N, tih_C, tih_H, tih_W;
    tensor_shape(top_inp_halo, explicit_nhwc, tih_N, tih_C, tih_H, tih_W);
    TORCH_CHECK(
            (toh_N == tox_N && tox_N == tix_N && tix_N == tih_N) &&
            (toh_C == tox_C && tox_C == tix_C && tix_C == tih_C) &&
            (toh_H == tox_H && tox_H == tix_H && tix_H == tih_H) &&
            (toh_W == tox_W && tox_W == tix_W && tix_W == tih_W));
    int boh_N, boh_C, boh_H, boh_W;
    tensor_shape(btm_out_halo, explicit_nhwc, boh_N, boh_C, boh_H, boh_W);
    int box_N, box_C, box_H, box_W;
    tensor_shape(btm_out_tx, explicit_nhwc, box_N, box_C, box_H, box_W);
    int bix_N, bix_C, bix_H, bix_W;
    tensor_shape(btm_inp_tx, explicit_nhwc, bix_N, bix_C, bix_H, bix_W);
    int bih_N, bih_C, bih_H, bih_W;
    tensor_shape(btm_inp_halo, explicit_nhwc, bih_N, bih_C, bih_H, bih_W);
    TORCH_CHECK(
            (boh_N == box_N && box_N == bix_N && bix_N == bih_N) &&
            (boh_C == box_C && box_C == bix_C && bix_C == bih_C) &&
            (boh_H == box_H && box_H == bix_H && bix_H == bih_H) &&
            (boh_W == box_W && box_W == bix_W && bix_W == bih_W));
    TORCH_CHECK(
	    (toh_N == boh_N) &&
	    (toh_C == boh_C) &&
	    (toh_H == boh_H) &&
	    (toh_W == boh_W));
    int NC=toh_C, NH=toh_H, NW=toh_W;
    if (diagnostics) printf("NC=%d, NH=%d, NW=%d\n",NC,NH,NW);

    int toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W;
    tensor_strides(top_out_halo, explicit_nhwc, toh_stride_N, toh_stride_C, toh_stride_H, toh_stride_W);
    int tox_stride_N, tox_stride_C, tox_stride_H, tox_stride_W;
    tensor_strides(top_out_tx, explicit_nhwc, tox_stride_N, tox_stride_C, tox_stride_H, tox_stride_W);
    int tix_stride_N, tix_stride_C, tix_stride_H, tix_stride_W;
    tensor_strides(top_inp_tx, explicit_nhwc, tix_stride_N, tix_stride_C, tix_stride_H, tix_stride_W);
    int tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W;
    tensor_strides(top_inp_halo, explicit_nhwc, tih_stride_N, tih_stride_C, tih_stride_H, tih_stride_W);
    int boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W;
    tensor_strides(btm_out_halo, explicit_nhwc, boh_stride_N, boh_stride_C, boh_stride_H, boh_stride_W);
    int box_stride_N, box_stride_C, box_stride_H, box_stride_W;
    tensor_strides(btm_out_tx, explicit_nhwc, box_stride_N, box_stride_C, box_stride_H, box_stride_W);
    int bix_stride_N, bix_stride_C, bix_stride_H, bix_stride_W;
    tensor_strides(btm_inp_tx, explicit_nhwc, bix_stride_N, bix_stride_C, bix_stride_H, bix_stride_W);
    int bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W;
    tensor_strides(btm_inp_halo, explicit_nhwc, bih_stride_N, bih_stride_C, bih_stride_H, bih_stride_W);

    // determine if nhwc
    auto is_nhwc = (toh_stride_C == 1) ? true : false;
    if (diagnostics) printf("is_nhwc = %s\n",is_nhwc?"true":"false");

    // figure out launch parameters
    int device;
    hipGetDevice(&device);
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, device);
    assert(numSM > 0 && numSM <= prop.multiProcessorCount);
    auto current_stream = at::cuda::getCurrentCUDAStream();
    const int numThreads = 128;
    dim3 block(numThreads,1,1);
    AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, top_out_halo.scalar_type(), "push_pull_halos_1d_kernel", [&]{
	    if (diagnostics) printf("size(scalar_t) = %ld\n",sizeof(scalar_t));
            scalar_t* toh_p = top_out_halo.data_ptr<scalar_t>();
            scalar_t* tox_p = top_out_tx.data_ptr<scalar_t>();
            scalar_t* tix_p = top_inp_tx.data_ptr<scalar_t>();
            scalar_t* tih_p = top_inp_halo.data_ptr<scalar_t>();
            scalar_t* boh_p = btm_out_halo.data_ptr<scalar_t>();
            scalar_t* box_p = btm_out_tx.data_ptr<scalar_t>();
            scalar_t* bix_p = btm_inp_tx.data_ptr<scalar_t>();
            scalar_t* bih_p = btm_inp_halo.data_ptr<scalar_t>();
	    if (diagnostics) printf("waypoint1\n");
	    int* top_signal_p = top_signal.data_ptr<int>() + 4;
	    int* btm_signal_p = btm_signal.data_ptr<int>();
	    int* top_wait_p = waits.data_ptr<int>();
	    int* btm_wait_p = waits.data_ptr<int>() + 4;
	    if (diagnostics) printf("waypoint2\n");

            // do int4 vector loads if channel count permits
            int elem_size_in_bytes = toh_C * sizeof(scalar_t);
            int elem_size_in_int4 = (elem_size_in_bytes / 16);
	    if (diagnostics) printf("elem_size_in_bytes = %d, elem_size_in_int4 = %d\n",elem_size_in_bytes,elem_size_in_int4);
            if (is_nhwc && elem_size_in_int4*16 == elem_size_in_bytes) {
                // can do int4 transfers
	        int divisor = toh_C / elem_size_in_int4;
		if (diagnostics) printf("CAN DO INT4 :: divisor = %d\n",divisor);
		toh_stride_N /= divisor;   toh_stride_H /= divisor;    toh_stride_W /= divisor;
		tox_stride_N /= divisor;   tox_stride_H /= divisor;    tox_stride_W /= divisor;
		tix_stride_N /= divisor;   tix_stride_H /= divisor;    tix_stride_W /= divisor;
		tih_stride_N /= divisor;   tih_stride_H /= divisor;    tih_stride_W /= divisor;
		boh_stride_N /= divisor;   boh_stride_H /= divisor;    boh_stride_W /= divisor;
		box_stride_N /= divisor;   box_stride_H /= divisor;    box_stride_W /= divisor;
		bix_stride_N /= divisor;   bix_stride_H /= divisor;    bix_stride_W /= divisor;
		bih_stride_N /= divisor;   bih_stride_H /= divisor;    bih_stride_W /= divisor;
		NC /= divisor;
		if (diagnostics) {
                    printf("divisor=%d\n",divisor);
                    printf("toh_stride :: N=%d, C=%d, H=%d, W=%d\n",toh_stride_N,toh_stride_C,toh_stride_H,toh_stride_W);
                    printf("tox_stride :: N=%d, C=%d, H=%d, W=%d\n",tox_stride_N,tox_stride_C,tox_stride_H,tox_stride_W);
                    printf("tix_stride :: N=%d, C=%d, H=%d, W=%d\n",tix_stride_N,tix_stride_C,tix_stride_H,tix_stride_W);
                    printf("tih_stride :: N=%d, C=%d, H=%d, W=%d\n",tih_stride_N,tih_stride_C,tih_stride_H,tih_stride_W);
                    printf("boh_stride :: N=%d, C=%d, H=%d, W=%d\n",boh_stride_N,boh_stride_C,boh_stride_H,boh_stride_W);
                    printf("box_stride :: N=%d, C=%d, H=%d, W=%d\n",box_stride_N,box_stride_C,box_stride_H,box_stride_W);
                    printf("bix_stride :: N=%d, C=%d, H=%d, W=%d\n",bix_stride_N,bix_stride_C,bix_stride_H,bix_stride_W);
                    printf("bih_stride :: N=%d, C=%d, H=%d, W=%d\n",bih_stride_N,bih_stride_C,bih_stride_H,bih_stride_W);
                    printf("NC=%d, NH=%d, NW=%d\n",NC,NH,NW);
                }
		void *kernelArgs[] = {
		    (int4**)&toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
		    (int4**)&tox_p, &tox_stride_C, &tox_stride_H, &tox_stride_W,
		    (int4**)&tix_p, &tix_stride_C, &tix_stride_H, &tix_stride_W,
		    (int4**)&tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
		    (int4**)&boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
		    (int4**)&box_p, &box_stride_C, &box_stride_H, &box_stride_W,
		    (int4**)&bix_p, &bix_stride_C, &bix_stride_H, &bix_stride_W,
		    (int4**)&bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
		    &NC, &NH, &NW,
		    &top_signal_p, &btm_signal_p, &top_wait_p, &btm_wait_p
		};
            	int numBlocksPerSm;
	        hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<int4,true>, numThreads, 0);
	        dim3 grid(numSM*numBlocksPerSm,1,1);
	        hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<int4,true>, grid, block, kernelArgs, 0, current_stream);
            } else {
                // cannot do int4 transfers
		if (diagnostics) printf("CAN NOT DO INT4\n");
		void *kernelArgs[] = {
		    &toh_p, &toh_stride_C, &toh_stride_H, &toh_stride_W,
		    &tox_p, &tox_stride_C, &tox_stride_H, &tox_stride_W,
		    &tix_p, &tix_stride_C, &tix_stride_H, &tix_stride_W,
		    &tih_p, &tih_stride_C, &tih_stride_H, &tih_stride_W,
		    &boh_p, &boh_stride_C, &boh_stride_H, &boh_stride_W,
		    &box_p, &box_stride_C, &box_stride_H, &box_stride_W,
		    &bix_p, &bix_stride_C, &bix_stride_H, &bix_stride_W,
		    &bih_p, &bih_stride_C, &bih_stride_H, &bih_stride_W,
		    &NC, &NH, &NW,
		    &top_signal_p, &btm_signal_p, &top_wait_p, &btm_wait_p
		};
                int numBlocksPerSm;
                if (is_nhwc) {
	            hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,true>, numThreads, 0);
	            dim3 grid(numSM*numBlocksPerSm,1,1);
	            hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,true>, grid, block, kernelArgs, 0, current_stream);
                } else {
	            hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocksPerSm, push_pull_halos_1d_kernel<scalar_t,false>, numThreads, 0);
	            dim3 grid(numSM*numBlocksPerSm,1,1);
	            hipLaunchCooperativeKernel((void*)push_pull_halos_1d_kernel<scalar_t,false>, grid, block, kernelArgs, 0, current_stream);
                }
	    }
        } );
}

} } }

