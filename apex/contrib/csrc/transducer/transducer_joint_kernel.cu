#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#include <torch/extension.h>
#include <ATen/AccumulateType.h>

#ifdef OLD_GENERATOR_PATH
#include <ATen/CUDAGeneratorImpl.h>
#else
#include <ATen/cuda/CUDAGeneratorImpl.h>
#endif

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAGraphsUtils.cuh>
#include <c10/macros/Macros.h>

#include "philox.cuh"

// Warp reduce kernels to reduce N groups of data into N numbers, where N = warpSize / width.
// width should be a power of 2 and should be less than warpSize.
template <typename scalar_t>
__device__ __forceinline__ scalar_t warpReduce(scalar_t x, int width=C10_WARP_SIZE){
    for (unsigned offset = width/2; offset > 0; offset /= 2){
        x += __shfl_down_sync(0xffffffff, x, offset, width);   
    }
    return x;
}

inline int largestPowerOfTwo(int x){
    int y = 1;
    while (y <= x)
        y <<= 1;
    return y >> 1;
}

/*
Figure out vectorization type for masks.
Similar to how PyTorch figures out acc_t here:
aten/src/ATen/AccumulateType.h 
*/
template <int V>
struct MaskVecType { };

template <> struct MaskVecType<1> { using type = uint8_t; };
template <> struct MaskVecType<2> { using type = uint16_t; };
template <> struct MaskVecType<4> { using type = uint32_t; };

template<int V>
using mvec_type = typename MaskVecType<V>::type;

// Helper class to calculate pointer offset that can be shared by different flavors of kernels.
// For fwd, batch offset and stride are different for packing and non-packing mode.
struct OffsetCalFwd{
    __device__ __forceinline__ OffsetCalFwd(
        int64_t batch, 
        const int64_t *batchOffset, 
        int64_t maxFLen, 
        int64_t maxGLen, 
        int64_t gLen,
        int64_t hiddenSize,
        bool packOutput) :
        batch(batch),
        batchOffset(batchOffset),
        maxFLen(maxFLen),
        maxGLen(maxGLen),
        gLen(gLen),
        hiddenSize(hiddenSize),
        packOutput(packOutput)
        {}
    
    int64_t batch;
    const int64_t *batchOffset;
    int64_t maxFLen;
    int64_t maxGLen;
    int64_t gLen;
    int64_t hiddenSize;
    bool packOutput;

    __device__ __forceinline__ int64_t getBatchOffset(){
        return packOutput ? ((batch==0) ? 0 : batchOffset[batch-1])*hiddenSize 
                            : batch*maxFLen*maxGLen*hiddenSize;
    }

    __device__ __forceinline__ int64_t getStrideF(){
        return packOutput ? gLen*hiddenSize : maxGLen*hiddenSize;
    }

    
};

// Helper class to calculate pointer offset that can be shared by different flavors of kernels
// For bwd, batch offset and stride are different for packing and non-packing mode.
// The reducion is done for two input tensors. Therefore, generating two sets of offsets
// according to bwdFasterDim can lead to a unified implementation in the actual kernel.
struct OffsetCalBwd{
    __device__ __forceinline__ OffsetCalBwd(
        int64_t batch, 
        const int64_t *batchOffset, 
        const int *fLen, 
        const int *gLen,
        int64_t maxFLen, 
        int64_t maxGLen, 
        int64_t hiddenSize,
        bool packOutput,
        bool bwdFasterDim) :
        batch(batch),
        batchOffset(batchOffset),
        maxFLen(maxFLen),
        maxGLen(maxGLen),
        fLen(fLen),
        gLen(gLen),
        hiddenSize(hiddenSize),
        packOutput(packOutput),
        bwdFasterDim(bwdFasterDim)
        {}

    int64_t batch;
    const int64_t *batchOffset;
    const int *fLen;
    const int *gLen;
    int64_t maxFLen;
    int64_t maxGLen;
    int64_t hiddenSize;
    bool packOutput;
    bool bwdFasterDim;  // whether doing bwd on the faster moving dimension

    __device__ __forceinline__ int64_t getBatchOffset(){
        return packOutput ? ((batch==0) ? 0 : batchOffset[batch-1])*hiddenSize 
                            : batch*maxFLen*maxGLen*hiddenSize;
    }

    __device__ __forceinline__ int64_t getMaxXLen(){
        return bwdFasterDim ? maxGLen : maxFLen;
    }

    __device__ __forceinline__ auto getMyXLen() -> decltype(gLen[batch]){
        return bwdFasterDim ? gLen[batch] : fLen[batch];
    }

    __device__ __forceinline__ auto getMyYLen() -> decltype(gLen[batch]){
        return bwdFasterDim ? fLen[batch] : gLen[batch];
    }
    
    __device__ __forceinline__ int64_t getStrideX(){
        return bwdFasterDim ? hiddenSize : ((packOutput ? gLen[batch] : maxGLen) * hiddenSize);
    }

    __device__ __forceinline__ int64_t getStrideY(){
        return bwdFasterDim ? ((packOutput ? gLen[batch] : maxGLen) * hiddenSize) : hiddenSize;
    }
};


// Vanila transducer joint forward kernel
// Detail of this joint function can be found in: 
// [1] Sequence Transduction with Recurrent Neural Networks.

// f is a tensor of shape [batch, T, H]
// g is a tensor of shape [batch, U, H]
// the transducer joint does
// sum = f.unsqueeze(dim=2) + g.unsqueeze(dim=1)
// The resultant tensor is of shape [batch, T, U, H]
// Each thread block is working on one "batch" of data in the output tensor, [batch, t, u, :]

// This joint function can optionally pack the output where the output tensor with a shape of
// [B, T, U, H] is packed into [B_packed, H].
// Don't-care region (t > fLen) or (u > gLen) is removed.
// To enable packing, the starting offset for each batch need to be specified with batchOffset.
template <typename scalar_t, class OffsetCal>
__global__ void transducer_joint_forward(
    const scalar_t *f,
    const scalar_t *g,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    bool packOutput,
    scalar_t *sum) {


    const int batch = blockIdx.z;
    const int t = blockIdx.y;
    const int u = blockIdx.x;
    const auto myFLen = fLen[batch];
    const auto myGLen = gLen[batch];

    OffsetCal offsetCal(batch, batchOffset, maxFLen, maxGLen, myGLen, hiddenSize, packOutput);
    const auto myBatchOffset = offsetCal.getBatchOffset();
    const auto strideF = offsetCal.getStrideF();
    scalar_t const *myF = f + batch*maxFLen*hiddenSize + t*hiddenSize;
    scalar_t const *myG = g + batch*maxGLen*hiddenSize + u*hiddenSize;
    scalar_t *mySum = sum + myBatchOffset + t*strideF + u * hiddenSize;

    if (t < myFLen and u < myGLen){
        #pragma unroll
        for (int h = threadIdx.x; h < hiddenSize; h += blockDim.x){
            if (h < hiddenSize){
                mySum[h] = myF[h] + myG[h];
            }
        }
    }
    else if (packOutput == false and t < maxFLen and u < maxGLen){
        // Need to write finite data to don't-care region because we instantiate the result tensor
        // with torch::empty for performance reasons. Even though it is don't-care region, the 
        // contents need to be finite, otherwise could lead to NaN in WGRAD.
        // In packing mode, this write is no longer necessary as we remove the don't-care region
        // from the output.
        // Picking -1 (over 0) here for ease of testing.
        #pragma unroll
        for (int h = threadIdx.x; h < hiddenSize; h += blockDim.x){
            if (h < hiddenSize){
                mySum[h] = -1;
            }
        }    
    }
}

/*
Tiled version of the joint forward kernel
Detail of this joint function can be found in: 
[1] Sequence Transduction with Recurrent Neural Networks.

f is a tensor of shape [batch, T, H]
g is a tensor of shape [batch, U, H]
the transducer joint does
sum = f.unsqueeze(dim=2) + g.unsqueeze(dim=1)
The resultant tensor is of shape [batch, T, U, H]
Each thread is working on a tile of the shape of tileF x tileG in the result tensor.
The input for the tile is first loaded in the register and is reused tileG and tileF times. 

This joint function can optionally pack the output where the output tensor with a shape of
[B, T, U, H] is packed into [B_packed, H].
Don't-care region (t > fLen) or (u > gLen) is removed.
To enable packing, the starting offset for each batch need to be specified with batchOffset.

Optionally this joint function performs ReLU and/or dropout on the joint output, which is 
controlled by arguments relu and dropout, respectively. philoxArgs is argument used for generating
pseudorandom number. When at least one of operations in ReLU and dropout is activated, the joint
function is a masked operation, which is controlled by the template argument masked. In this case, 
masks are saved to backward.
*/
template <typename scalar_t, int tileF, int tileG, int U, class OffsetCal, bool masked>
__global__ void transducer_joint_tiled_forward(
    const scalar_t *f,
    const scalar_t *g,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    int64_t hiddenPerBlock,
    bool packOutput,
    bool relu, 
    bool dropout,
    float p,
    at::PhiloxCudaState philoxArgs,
    scalar_t *sum,
    uint8_t *mask) {

    static_assert(U == 4, "U has to be 4, as random numbers are generated in batch of 4");

    const int batch = blockIdx.z;
    const int t = blockIdx.y * tileF;
    const int hiddenBlock = (hiddenSize + hiddenPerBlock - 1) / hiddenPerBlock;
    const int u = blockIdx.x / hiddenBlock * tileG;
    const int hOffset = (blockIdx.x % hiddenBlock) * hiddenPerBlock;
    const int h = threadIdx.x;
    const auto myFLen = fLen[batch];
    const auto myGLen = gLen[batch];

    OffsetCal offsetCal(batch, batchOffset, maxFLen, maxGLen, myGLen, hiddenSize, packOutput);
    const auto myBatchOffset = offsetCal.getBatchOffset();
    const auto strideF = offsetCal.getStrideF();

    scalar_t const *myF = f + batch*maxFLen*hiddenSize + t*hiddenSize + hOffset;
    scalar_t const *myG = g + batch*maxGLen*hiddenSize + u*hiddenSize + hOffset;
    scalar_t *mySum = sum + myBatchOffset + t*strideF + u*hiddenSize + hOffset;
    uint8_t *myMask = mask + myBatchOffset + t*strideF + u*hiddenSize + hOffset;

    // The following code is only needed for dropout. We try to bypass them as much as possible.
    auto seeds = masked ? at::cuda::philox::unpack(philoxArgs) 
                            : std::make_tuple(static_cast<uint64_t>(0), static_cast<uint64_t>(0));
    uint64_t tid = masked ? (static_cast<uint64_t>(blockIdx.z)*gridDim.y*gridDim.x + 
                        blockIdx.y*gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x
                            : 0;
    Philox ph(std::get<0>(seeds), tid, std::get<1>(seeds)); 
    scalar_t scale = masked ? ((p == 0) ? 0 : 1 / p) : 0;  
    bool dropoutMask[U];

    if (t < myFLen and u < myGLen and hOffset+h < hiddenSize){    
        // register buffers for tiled input reuse
        scalar_t fBuffer[tileF], gBuffer[tileG];    
        for (int i = 0; i < tileF; ++i){
            if (t + i < myFLen)
                fBuffer[i] = myF[i*hiddenSize + h];
        }
        for (int j = 0; j < tileG; ++j){
            if (u + j < myGLen)
                gBuffer[j] = myG[j*hiddenSize + h];
        }
        #pragma unroll
        for (int i = 0; i < tileF; ++i){
            if (t + i < myFLen){
                #pragma unroll
                for (int j = 0; j < tileG; ++j){
                    int idx = i*tileG + j;
                    if (masked and dropout and idx % U == 0){
                        // For performance, generate 4 random numbers in one shot
                        // auto rand4 = hiprand_uniform4(&state);
                        auto rand4 = uniform4(ph());
                        dropoutMask[0] = rand4.x < p;
                        dropoutMask[1] = rand4.y < p;
                        dropoutMask[2] = rand4.z < p;
                        dropoutMask[3] = rand4.w < p;
                    }

                    if (u + j < myGLen){
                        scalar_t out = fBuffer[i] + gBuffer[j];
                        if (masked){
                            // Apply ReLU here when relu is True
                            bool localMask = relu ? (out>0) : 1;
                            localMask = dropout ? localMask & dropoutMask[idx%U] : localMask;
                            out = dropout ? out*localMask*scale : out*localMask;
                            myMask[i*strideF + j*hiddenSize + h] = static_cast<uint8_t>(localMask);
                        }
                        mySum[i*strideF + j*hiddenSize + h] = out;
                    }
                    else if (packOutput == false and u + j < maxGLen)
                        mySum[i*strideF + j*hiddenSize + h] = -1;
                }
            }
            else if (packOutput == false and t + i < maxFLen){
                // Again need to write finite data to don't-care region
                #pragma unroll
                for (int j = 0; j < tileG; ++j){
                    if (u + j < maxGLen)
                        mySum[i*strideF + j*hiddenSize + h] = -1;
                }
            }
        }
    }
    else if (packOutput == false and t < maxFLen and u < maxGLen and hOffset+h < hiddenSize){
        // Only need to ensure the finity in normal mode
        #pragma unroll
        for (int i = 0; i < tileF; ++i){
            if (t + i < maxFLen){
                #pragma unroll
                for (int j = 0; j < tileG; ++j){
                    if (u + j < maxGLen)
                        mySum[i*strideF + j*hiddenSize + h] = -1;
                }
            }
        }
    }
}

/*
Bwd operation (reduction) on one input tensor. Since the operation performed for the two input
tensors are exactly the same, only one kernel is needed, and the different indexing offsets
and strides are handled by OffsetCalBwd.

When packing is enabled in the fwd op, unpacking is needed to restore the gradients in a 
non-packed form.

When ReLU and/or dropout are performed in the fwd pass, this operation becomes a masked operation,
and mask contains the mask information.
*/
template <typename scalar_t, typename acc_t, class OffsetCal, bool masked>
__device__ void transducer_joint_single_backward(
    const scalar_t *grad,
    const uint8_t *mask,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    bool packOutput,
    bool bwdFasterDim,  // whether bwd on the faster moving dimension (u)
    float scale,
    scalar_t *inGrad,
    int yBlockOffset=0) {


    const int batch = blockIdx.z;
    // For the second input tensor, this offset need to be subtracted because the first yBlockOffset
    // sets of thread blocks are for the first input tensor.
    const int x = blockIdx.y-yBlockOffset;
    const int hOffset = blockIdx.x*C10_WARP_SIZE;
    const int wid = threadIdx.y;
    const int lid = threadIdx.x;
    const int numWarp = blockDim.y;
    extern __shared__ char smem8[];
    auto smem = reinterpret_cast<acc_t*>(smem8);

    OffsetCal offsetCal(batch, batchOffset, fLen, gLen, maxFLen, maxGLen, hiddenSize, packOutput, 
                        bwdFasterDim);
    const auto maxXLen = offsetCal.getMaxXLen();
    const auto myXLen = offsetCal.getMyXLen();
    const auto myYLen = offsetCal.getMyYLen();
    scalar_t *myInGrad = inGrad + batch*maxXLen*hiddenSize + x*hiddenSize + hOffset;
    
    if (x < myXLen){
        
        const auto myBatchOffset = offsetCal.getBatchOffset();
        const auto strideX = offsetCal.getStrideX();
        const auto strideY = offsetCal.getStrideY();
        const scalar_t *myGrad = grad + myBatchOffset + x*strideX + hOffset;
        const uint8_t *myMask = masked ? mask + myBatchOffset + x*strideX + hOffset : nullptr;
        
        // Each warp reduces numYPerWarp "y" first
        acc_t warpSum = 0;
        auto numYPerWarp = (myYLen+numWarp-1)/numWarp;
        #pragma unroll
        for (int warpY = 0; warpY < numYPerWarp; ++warpY){
            auto y = wid*numYPerWarp + warpY;
            if (y < myYLen and (hOffset+lid) < hiddenSize)
                if (masked)
                    warpSum += static_cast<acc_t>(myGrad[y*strideY + lid]) * myMask[y*strideY + lid] * scale;
                else    
                    warpSum += myGrad[y*strideY + lid];
        }

        // transpose partial sum in SMEM and reduce further using warpReduce
        smem[lid*numWarp + wid] = warpSum;
        __syncthreads();
        auto sum = smem[wid*C10_WARP_SIZE + lid];
        sum = warpReduce(sum, numWarp);

        // a a b b c c d d
        // a a b b c c d d
        // a a b b c c d d
        // a a b b c c d d
        // example of 4 warps (a, b, c, d) with 8 threads per warp
        // Each warp need 8 / 4 = 2 threads to write the results.
        if (hOffset+wid*C10_WARP_SIZE/numWarp+lid/numWarp < hiddenSize){
            if (lid % numWarp == 0){
                myInGrad[wid*C10_WARP_SIZE/numWarp + lid/numWarp] = sum;
            }
        }
    }
    else if (wid == 0 and hOffset + lid < hiddenSize){
        // Need to ensure the grad is zero for don't care region
        myInGrad[lid] = 0;
    }
}

/*
Actual bwd (reduction) kernel get launched.
Call transducer_joint_single_backward twice on two input tensors. 
The two bwd ops are launched together, the first op uses blockIdx.y < maxFLen, and the second op 
uses the rest.
When ReLU and/or dropout are performed in the fwd pass, this operation becomes a masked operation,
and mask contains the mask information.
*/
template <typename scalar_t, typename acc_t, class OffsetCal, bool masked>
__global__ void transducer_joint_combined_backward(
    const scalar_t *grad,
    const uint8_t *mask,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    bool packOutput,
    float scale,
    scalar_t *fGrad,
    scalar_t *gGrad) {
    if (blockIdx.y < maxFLen){
        transducer_joint_single_backward<scalar_t, acc_t, OffsetCal, masked>(
            grad,
            mask,
            fLen,
            gLen,
            batchOffset,
            maxFLen,
            maxGLen,
            hiddenSize,
            packOutput,
            false,
            scale,
            fGrad);
    }
    else{
        transducer_joint_single_backward<scalar_t, acc_t, OffsetCal, masked>(
            grad,
            mask,
            fLen,
            gLen,
            batchOffset,
            maxFLen,
            maxGLen,
            hiddenSize,
            packOutput,
            true,
            scale,
            gGrad,
            maxFLen);
    }  
}

/*
Vectorized version of transducer_joint_single_backward
Doing exact same operation as transducer_joint_single_backward except the load and store are
vectorized.
When packing is enabled in the fwd op, unpacking is needed to restore the gradients in a 
non-packed form.
When ReLU and/or dropout are performed in the fwd pass, this operation becomes a masked operation,
and mask contains the mask information.
*/
template <typename scalar_t, typename acc_t, typename vec_t, int V, class OffsetCal, bool masked>
__device__ void transducer_joint_single_vec_backward(
    const scalar_t *grad,
    const uint8_t *mask,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    bool packOutput,
    bool bwdFasterDim,
    float scale,
    scalar_t *inGrad,
    int yBlockOffset=0){

    const int batch = blockIdx.z;
    const int x = blockIdx.y - yBlockOffset;
    const int hOffset = blockIdx.x*C10_WARP_SIZE*V;
    const int wid = threadIdx.y;
    const int lid = threadIdx.x;
    const int numWarp = blockDim.y;

    // Figure out the vectorization type for mask
    using mvec_t = mvec_type<V>;

    OffsetCal offsetCal(batch, batchOffset, fLen, gLen, maxFLen, maxGLen, hiddenSize, packOutput, 
                        bwdFasterDim);
    const auto maxXLen = offsetCal.getMaxXLen();
    const auto myXLen = offsetCal.getMyXLen();
    const auto myYLen = offsetCal.getMyYLen();
    scalar_t *myInGrad = inGrad + batch*maxXLen*hiddenSize + x*hiddenSize + hOffset;
    extern __shared__ char smem8[];
    auto smem = reinterpret_cast<acc_t*>(smem8);

    acc_t warpSum[V];
    scalar_t inBuffer[V];
    uint8_t maskBuffer[V];
    scalar_t outBuffer[V];
    auto myInGradVec = reinterpret_cast<vec_t*>(myInGrad);
    auto outBufferVec = reinterpret_cast<vec_t*>(outBuffer);

    if (x < myXLen){
        const auto myBatchOffset = offsetCal.getBatchOffset();
        const auto strideX = offsetCal.getStrideX();
        const auto strideY = offsetCal.getStrideY();
        const scalar_t *myGrad = grad + myBatchOffset + x*strideX + hOffset;
        const uint8_t *myMask = masked ? mask + myBatchOffset + x*strideX + hOffset
                                            :nullptr;

        for (int i = 0; i < V; ++i)
            warpSum[i] = 0;

        // Each warp reduces numYPerWarp "y" first
        auto numYPerWarp = (myYLen+numWarp-1)/numWarp;
        for (int warpY = 0; warpY < numYPerWarp; ++warpY){
            auto y = wid*numYPerWarp + warpY;
            auto myGradVec = reinterpret_cast<vec_t const *>(myGrad + y*strideY);
            auto myMaskVec = masked ? reinterpret_cast<mvec_t const *>(myMask + y*strideY)
                                        : nullptr;
            auto inBufferVec = reinterpret_cast<vec_t*>(inBuffer);
            auto maskBufferVec = reinterpret_cast<mvec_t*>(maskBuffer);
            if (hOffset + lid*V < hiddenSize and y < myYLen){
                *inBufferVec = myGradVec[lid];  // vectorized load
                if (masked){
                    *maskBufferVec = myMaskVec[lid];
                    #pragma unroll
                    for (int i = 0; i < V; ++i)
                        warpSum[i] += static_cast<acc_t>(inBuffer[i]) * maskBuffer[i] * scale;
                }
                else{
                    #pragma unroll
                    for (int i = 0; i < V; ++i)
                        warpSum[i] += inBuffer[i];
                }
            }
        }
        
        // transpose partial sum in SMEM and reduce further using warpReduce
        for (int i = 0; i < V; ++i){
            smem[lid*numWarp + wid] = warpSum[i];
            __syncthreads();
            auto sum = smem[wid*C10_WARP_SIZE + lid];

            if (hOffset+(wid*C10_WARP_SIZE/numWarp)*V < hiddenSize){
                sum = warpReduce(sum, numWarp);
                if (lid % numWarp == 0){
                    outBuffer[i] = sum;
                }
            }
            __syncthreads();
        }

        // a a b b c c d d
        // a a b b c c d d
        // a a b b c c d d
        // a a b b c c d d
        // example of 4 warps (a, b, c, d) with 8 threads per warp
        // Each warp need 8 / 4 = 2 threads to write the results.
        if (lid % numWarp == 0 and hOffset+(wid*C10_WARP_SIZE/numWarp + lid/numWarp)*V < hiddenSize)
            myInGradVec[wid*C10_WARP_SIZE/numWarp + lid/numWarp] = *outBufferVec;     
    }
    else if (wid == 0 and hOffset + lid*V < hiddenSize){
        // Need to ensure the grad is zero for don't care region
        myInGradVec[lid] = 0;
    }
}

/*
Vecotrized version of transducer_joint_combined_backward
Call transducer_joint_single_vec_backward twice on two input tensors. 
The two bwd ops are launched together, the first op uses blockIdx.y < maxFLen, and the second op 
uses the rest.
When ReLU and/or dropout are performed in the fwd pass, this operation becomes a masked operation,
and mask contains the mask information.
*/
template <typename scalar_t, typename acc_t, typename vec_t, int V, class OffsetCal, bool masked>
__global__ void transducer_joint_combined_vec_backward(
    const scalar_t *grad,
    const uint8_t *mask,
    const int *fLen,
    const int *gLen,
    const int64_t *batchOffset,
    int64_t maxFLen,
    int64_t maxGLen,
    int64_t hiddenSize,
    bool packOutput,
    float scale,
    scalar_t *fGrad,
    scalar_t *gGrad) {
    if (blockIdx.y < maxFLen){
        transducer_joint_single_vec_backward<scalar_t, acc_t, vec_t, V, OffsetCal, masked>(
            grad,
            mask,
            fLen,
            gLen,
            batchOffset,
            maxFLen,
            maxGLen,
            hiddenSize,
            packOutput,
            false,
            scale,
            fGrad);
    }
    else{
        transducer_joint_single_vec_backward<scalar_t, acc_t, vec_t, V, OffsetCal, masked>(
            grad,
            mask,
            fLen,
            gLen,
            batchOffset,
            maxFLen,
            maxGLen,
            hiddenSize,
            packOutput,
            true,
            scale,
            gGrad,
            maxFLen);
    }  
}




std::vector<torch::Tensor> transducer_joint_cuda_forward(
    torch::Tensor f,
    torch::Tensor g,
    torch::Tensor fLen,
    torch::Tensor gLen,
    torch::Tensor batchOffset,
    int64_t packedBatch,
    int opt,
    bool packOutput,
    bool relu,
    bool dropout,
    float dropoutProb,
    int tileSize){

    
    auto tensorOpt = f.options();
    auto dtype = f.scalar_type();
    const auto batchSize = f.size(0);
    const auto maxFLen = f.size(1);
    const auto maxGLen = g.size(1);
    const auto hiddenSize = f.size(2);
    bool masked = dropout or relu;
    
    int64_t *batchOffsetPtr = nullptr;
    torch::Tensor sum, mask;
    auto maskOpt = tensorOpt.dtype(torch::kUInt8);
    if (!packOutput){
        sum = torch::empty({batchSize, maxFLen, maxGLen, hiddenSize}, tensorOpt);
        batchOffsetPtr = nullptr;
        if (masked)
            mask = torch::empty({batchSize, maxFLen, maxGLen, hiddenSize}, maskOpt);
    }
    else{
        sum = torch::empty({packedBatch, hiddenSize}, tensorOpt);    
        batchOffsetPtr = batchOffset.data_ptr<int64_t>();
        if (masked)
            mask = torch::empty({packedBatch, hiddenSize}, maskOpt);
    }
    uint8_t *maskPtr = masked ? mask.data_ptr<uint8_t>() : nullptr;

    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    TORCH_CHECK(opt == 0 or opt == 1, "Got an invalid optimization level ", opt);
    // Simple heuristics
    const int numThread = std::min(128, (static_cast<int>(hiddenSize)+C10_WARP_SIZE-1)
                                        / C10_WARP_SIZE * C10_WARP_SIZE);
    
    if (opt == 0){
        // vanilla kernel
        const int threads = numThread;
        const dim3 blocks(maxGLen, maxFLen, batchSize);

        AT_DISPATCH_FLOATING_TYPES_AND_HALF(dtype, "transducer_joint_forward", ([&] {
            transducer_joint_forward<scalar_t, OffsetCalFwd>
            <<<blocks, threads, 0, stream>>>(
                f.data_ptr<scalar_t>(), 
                g.data_ptr<scalar_t>(), 
                fLen.data_ptr<int>(), 
                gLen.data_ptr<int>(), 
                batchOffsetPtr,
                maxFLen,
                maxGLen,
                hiddenSize,
                packOutput,
                sum.data_ptr<scalar_t>());
        }));  
    }
    if (opt == 1){
        // tiled version. For simplicity, assume tileF == tileG, even though the kernel can 
        // support more general cases.
        const int threads = numThread;
        const int hiddenPerBlock = numThread;
        const int hiddenBlock = (hiddenSize + hiddenPerBlock - 1) / hiddenPerBlock;
        const dim3 blocks(  (maxGLen+tileSize-1)/tileSize * hiddenBlock, 
                            (maxFLen+tileSize-1)/tileSize, 
                            batchSize);

        TORCH_CHECK(tileSize == 1 or tileSize == 2 or tileSize == 4, 
                "Expected tileSize to be in [1, 2, 4], but got ", tileSize);

        at::PhiloxCudaState rng_engine_inputs;
        if (masked){
            // set up PRG when the input is masked. rng_engine_inputs will be used as a space filler 
            // for non-masked calls.
            // Therefore no need to initialize.
            c10::optional<at::Generator> gen_;
            auto gen = at::get_generator_or_default<at::CUDAGeneratorImpl>(gen_, 
                                                    at::cuda::detail::getDefaultCUDAGenerator());
            // counterOffset records how many cuRAND calls each thread makes. For a tiled kernel, 
            // each thread processes tileF * tileG output elements. 
            int64_t counterOffset = tileSize * tileSize;
            {
                std::lock_guard<std::mutex> lock(gen->mutex_);
                rng_engine_inputs = gen->philox_cuda_state(counterOffset);
            }
        }

        AT_DISPATCH_FLOATING_TYPES_AND_HALF(dtype, "transducer_joint_forward", ([&] {
            void(*kernel)(const scalar_t*, const scalar_t*, const int*, const int*, const int64_t*, 
                            int64_t, int64_t, int64_t, int64_t, bool, bool, bool, float, 
                            at::PhiloxCudaState, scalar_t*, uint8_t*);
            if (masked){
                switch (tileSize){
                    case 2:
                        kernel = &transducer_joint_tiled_forward<scalar_t, 2, 2, 4, OffsetCalFwd, 
                                                                    true>;
                        break;
                    case 4:
                        kernel = &transducer_joint_tiled_forward<scalar_t, 4, 4, 4, OffsetCalFwd, 
                                                                    true>;
                        break;
                }
            }
            else{
                switch (tileSize){
                    case 1:
                        kernel = &transducer_joint_tiled_forward<scalar_t, 1, 1, 4, OffsetCalFwd, 
                                                                    false>;
                        break;
                    case 2:
                        kernel = &transducer_joint_tiled_forward<scalar_t, 2, 2, 4, OffsetCalFwd, 
                                                                    false>;
                        break;
                    case 4:
                        kernel = &transducer_joint_tiled_forward<scalar_t, 4, 4, 4, OffsetCalFwd, 
                                                                    false>;
                        break;
                }
            }
            
            kernel<<<blocks, threads, 0, stream>>>(
                f.data_ptr<scalar_t>(),
                g.data_ptr<scalar_t>(),
                fLen.data_ptr<int>(),
                gLen.data_ptr<int>(),
                batchOffsetPtr,
                maxFLen,
                maxGLen,
                hiddenSize,
                hiddenPerBlock,
                packOutput,
                relu,
                dropout,
                1.0f - dropoutProb,
                rng_engine_inputs,
                sum.data_ptr<scalar_t>(),
                maskPtr);
        }));  
    }
 
    C10_CUDA_CHECK(hipGetLastError());
    if (masked) 
        return {sum, mask};
    else
        return {sum};
}

std::vector<torch::Tensor> transducer_joint_cuda_backward(
    std::vector<torch::Tensor> in,
    torch::Tensor fLen,
    torch::Tensor gLen,
    torch::Tensor batchOffset,
    int maxFLen,
    int maxGLen,
    bool packOutput,
    float scale){

    auto grad = in[0];
    bool masked = (in.size() == 2);
    uint8_t *maskPtr = masked ? in[1].data_ptr<uint8_t>() : nullptr;

    auto tensorOpt = grad.options();
    auto dtype = grad.scalar_type();
    const int batchSize = fLen.size(0);
    const int hiddenSize = grad.size(-1);

    const auto deviceProperties = at::cuda::getCurrentDeviceProperties();
    const int maxNumWarp = deviceProperties->maxThreadsPerBlock / C10_WARP_SIZE;

    torch::Tensor fGrad = torch::empty({batchSize, maxFLen, hiddenSize}, tensorOpt);
    torch::Tensor gGrad = torch::empty({batchSize, maxGLen, hiddenSize}, tensorOpt);

    int64_t *batchOffsetPtr = (!packOutput) ? nullptr : batchOffset.data_ptr<int64_t>(); 

    // The number "y" I would like each thread to work on
    const int workPerThread = 32;   
    // Since the bwd for f and g have the same thread block size, we need to use the max of the two.
    int numWarp = largestPowerOfTwo((std::max(maxFLen, maxGLen) + workPerThread-1) / workPerThread);
    // Would like to have at least 2 warps 
    numWarp = std::max(2, numWarp);
    // cap on the maximum number of warps allowed
    numWarp = std::min(maxNumWarp, numWarp); 

    // Need smem for transposing the partial sum. The partial sum is in a matrix of the shape
    // numWarp x warpSize
    const int smemSize = numWarp * C10_WARP_SIZE;
    const dim3 threads(C10_WARP_SIZE, numWarp, 1);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(dtype, "transducer_joint_cuda_backward_kernel", ([&] {
        auto gradPtr = grad.data_ptr<scalar_t>();
        auto fLenPtr = fLen.data_ptr<int>();
        auto gLenPtr = gLen.data_ptr<int>(); 
        auto fGradPtr = fGrad.data_ptr<scalar_t>();
        auto gGradPtr = gGrad.data_ptr<scalar_t>();

        // resolve the acc_t type
        using acc_t = at::acc_type<scalar_t, true>;
        using vec_t = uint64_t;

        constexpr int vectFactor = sizeof(vec_t) / sizeof(scalar_t);
        constexpr int vecAlignment = std::alignment_of<vec_t>::value;

        // if all input and output tensors meet the alignment requirement
        bool memAlign = (reinterpret_cast<uint64_t>(gradPtr) % vecAlignment == 0) 
                        and (reinterpret_cast<uint64_t>(fGradPtr) % vecAlignment == 0) 
                        and (reinterpret_cast<uint64_t>(gGradPtr) % vecAlignment == 0);

        if (vectFactor > 1 and hiddenSize%vectFactor == 0 and memAlign){
            // If vectorization helps and the alignment requirement is met, use the vectorized 
            // kernel. For simplicity, hiddenSize needs to be a multiple vecFactor.
            const dim3 blocks(  (hiddenSize+C10_WARP_SIZE*vectFactor-1)/(C10_WARP_SIZE*vectFactor), 
                                maxFLen+maxGLen, 
                                batchSize);
            if (masked){
                transducer_joint_combined_vec_backward
                    <scalar_t, acc_t, vec_t, vectFactor, OffsetCalBwd, true>
                    <<<blocks, threads, smemSize*sizeof(acc_t)>>>(
                    gradPtr,
                    maskPtr,
                    fLenPtr, 
                    gLenPtr, 
                    batchOffsetPtr, 
                    maxFLen,
                    maxGLen,
                    hiddenSize,
                    packOutput,
                    scale,
                    fGradPtr,
                    gGradPtr);
            }
            else{
                transducer_joint_combined_vec_backward
                <scalar_t, acc_t, vec_t, vectFactor, OffsetCalBwd, false>
                <<<blocks, threads, smemSize*sizeof(acc_t)>>>(
                    gradPtr,
                    maskPtr,
                    fLenPtr, 
                    gLenPtr, 
                    batchOffsetPtr, 
                    maxFLen,
                    maxGLen,
                    hiddenSize,
                    packOutput,
                    scale,
                    fGradPtr,
                    gGradPtr);    
            }
        }
        else{
            const dim3 blocks((hiddenSize+C10_WARP_SIZE-1)/C10_WARP_SIZE, 
                                maxFLen + maxGLen, batchSize);
            if (masked){
                transducer_joint_combined_backward<scalar_t, acc_t, OffsetCalBwd, true>
                <<<blocks, threads, smemSize*sizeof(acc_t)>>>(
                    gradPtr,
                    maskPtr,
                    fLenPtr, 
                    gLenPtr, 
                    batchOffsetPtr, 
                    maxFLen,
                    maxGLen,
                    hiddenSize,
                    packOutput,
                    scale,
                    fGradPtr,
                    gGradPtr);
            }
            else{
                transducer_joint_combined_backward<scalar_t, acc_t, OffsetCalBwd, false>
                <<<blocks, threads, smemSize*sizeof(acc_t)>>>(
                    gradPtr,
                    maskPtr,
                    fLenPtr, 
                    gLenPtr, 
                    batchOffsetPtr, 
                    maxFLen,
                    maxGLen,
                    hiddenSize,
                    packOutput,
                    scale,
                    fGradPtr,
                    gGradPtr);
            }
        }
    }));   

    return {fGrad, gGrad};
}
