#include "hip/hip_runtime.h"
#include "gn.hpp"

#include <cstdio>
#include <mutex>
#include <stdexcept>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>

#include "gn_utils.hpp"
#include "gn_dispatch_hw_c.hpp"


#define DISPATCH_NUM_GROUPS_AND_SILU(num_groups, silu, NUM_GROUPS, SILU, ...) [&] { \
    if (num_groups == 16 && silu == true) { constexpr int NUM_GROUPS = 16; constexpr bool SILU = true; return __VA_ARGS__(); } \
    if (num_groups == 32 && silu == false) { constexpr int NUM_GROUPS = 32; constexpr bool SILU = false; return __VA_ARGS__(); } \
    throw std::invalid_argument("DISPATCH_NUM_GROUPS_AND_SILU " + std::to_string(num_groups) + " " + std::to_string(silu)); \
    }()

namespace group_norm_v2 {

template<typename T, int HW, int C, int G, bool SILU>
void gn_cuda_single_shape(GN_CUDA_HOST_PARAMS(T));

template<typename T, int HW, int C, int G, bool SILU>
void gn_bwd_cuda_single_shape(GN_BWD_CUDA_HOST_PARAMS(T));

template<typename T>
void gn_cuda(GN_CUDA_HOST_PARAMS(T)) {
    DISPATCH_HW_C(hw, num_groups * channels_per_group, HW, C, [&] {
        DISPATCH_NUM_GROUPS_AND_SILU(num_groups, silu, G, SILU, [&] {
            return gn_cuda_single_shape<T, HW, C, G, SILU>(GN_CUDA_HOST_ARGS);
        });
    });
}

template<typename T>
void gn_bwd_cuda(GN_BWD_CUDA_HOST_PARAMS(T)) {
    DISPATCH_HW_C(hw, num_groups * channels_per_group, HW, C, [&] {
        DISPATCH_NUM_GROUPS_AND_SILU(num_groups, silu, G, SILU, [&] {
            return gn_bwd_cuda_single_shape<T, HW, C, G, SILU>(GN_BWD_CUDA_HOST_ARGS);
        });
    });
}

template void gn_cuda(GN_CUDA_HOST_PARAMS(half));
template void gn_cuda(GN_CUDA_HOST_PARAMS(__hip_bfloat16));

template void gn_bwd_cuda(GN_BWD_CUDA_HOST_PARAMS(half));
template void gn_bwd_cuda(GN_BWD_CUDA_HOST_PARAMS(__hip_bfloat16));

}  // namespace group_norm_v2
