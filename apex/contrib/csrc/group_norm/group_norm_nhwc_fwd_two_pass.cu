#include "hip/hip_runtime.h"
/***************************************************************************************************
 * Copyright (c) 2011-2023, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without modification, are not permit-
 * ted.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND ANY EXPRESS OR 
 * IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND 
 * FITNESS FOR A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE 
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, 
 * BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; 
 * OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, 
 * STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE 
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 **************************************************************************************************/
#include "group_norm_nhwc.h"
#include "macros.h"
#include "traits.h"
#include <assert.h>
#include <hipcub/hipcub.hpp>

////////////////////////////////////////////////////////////////////////////////////////////////////
//
// F O R W A R D
//
////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename Traits_, int THREADS_PER_BLOCK >
__global__ void group_norm_nhwc_fwd_sum_kernel(Group_norm_nhwc_fwd_params params) {

  // The IO traits.
  using Traits = Traits_;
  // The IO type
  using IOType = typename Traits::IOType;
  // The IO doubled type
  using IOType2 = typename Traits::IOType2;

  // The object in charge of doing the sums for the different blocks.
  typedef hipcub::BlockScan<Group_sums, THREADS_PER_BLOCK> Block_scan;

  // Allocate shared memory for Block_scan.
  __shared__ typename Block_scan::TempStorage temp_storage;
  // Allocate shared memory for the groups. We could reduce the amount of shared memory reserved.
  __shared__ float2 smem[THREADS_PER_BLOCK];

  // The instance in the batch.
  int ni = blockIdx.z;
  // The channel loaded by that thread (2 channels per thread for F16x2).
  int ci = blockIdx.x * params.channels_per_block + threadIdx.x * 2;

  // The first activation loaded by that block.
  int hw_begin = blockIdx.y * params.acts_per_block;
  // The last activation loaded by that block.
  int hw_end = min(hw_begin + params.acts_per_block, params.hw);

  // The sums.
  float sum = 0.f, sum_sq = 0.f;

  // Iterate over the activations to compute the sums.
  for( int hwi = hw_begin; hwi < hw_end; ++hwi ) {

    // The offset.
    int64_t offset = (int64_t) ni*params.hwc + hwi*params.c + ci;

    // Fetch two channels per thread.
    IOType2 v2 = Traits::zero();
    if( ci < params.c ) {
      v2  = *reinterpret_cast<const IOType2*>(&reinterpret_cast<const IOType*>(params.x )[offset]);
    }

    // Extract the two values.
    float2 f2 = Traits::unpack(v2);

    // Update the sum.
    sum += f2.x + f2.y;
    // Update the sum of squares.
    sum_sq += f2.x * f2.x + f2.y * f2.y;
  }

  // The group that thread works on and the channel in the group (modulus).
  int gj = threadIdx.x * 2 / params.channels_per_group;
  int cj = threadIdx.x * 2 - params.channels_per_group * gj;

  // The data for the summations.
  Group_sums inp {cj == 0 ? 1 : 0, sum, sum_sq};

  // Do the segmented scan.
  Group_sums out;
  Block_scan(temp_storage).InclusiveScan(inp, out, Group_sums_op());

  // Store the results for the groups in shared memory (to produce coalesced stores later).
  if( cj == params.channels_per_group - 2 /* 2 channels per thread */ ) {
    smem[gj] = make_float2(out.sum, out.sum_sq);
  }

  // Make sure the data is in shared memory.
  __syncthreads();

  // The global group index.
  int gk = blockIdx.x * params.groups_per_block + threadIdx.x;

  // Threads that have nothing left to do, exit.
  if( threadIdx.x >= params.groups_per_block || gk >= params.groups ) {
    return;
  }

  // The first threads (those storing to global memory, load the values).
  float2 sums = smem[threadIdx.x];

  // Store to global memory.
  atomicAdd(&params.zeroed_red_buffer[(2*ni+0)*params.groups + gk], sums.x);
  atomicAdd(&params.zeroed_red_buffer[(2*ni+1)*params.groups + gk], sums.y);
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void group_norm_nhwc_fwd_two_passes_setup(Group_norm_nhwc_fwd_params &params,
                                          size_t &zeroed_red_buffer_elts) {

  // The pre-computed dimensions.
  params.hw  = params.h * params.w;
  params.hwc = params.c * params.hw;

  // The number of channels per group.
  params.channels_per_group = params.c / params.groups;
  // The inverse to compute the mean/variance.
  params.inv_hwc_per_group = 1.f / (float) (params.hw * params.channels_per_group);

  // Define the number of blocks per activation map. That's a simple heuristic.
  int blocks_per_act_slice = 0;
         if( params.c >= 1280 ) { 
    blocks_per_act_slice = 128 / params.n;
  } else if( params.c >= 640 ) {
    blocks_per_act_slice = 256 / params.n;
  } else {
    blocks_per_act_slice = 512 / params.n;
  }

  // Make sure we launch blocks per activation is no less than activations
  blocks_per_act_slice = min(blocks_per_act_slice, div_up(params.hw, params.n));

  // Define how many activations are computed per block.
  params.acts_per_block = div_up(params.hw, blocks_per_act_slice);
  // The number of channels per block.
  params.channels_per_block = 320;
  // Special case to deal with 30 channels per group.
  if( params.channels_per_block % params.channels_per_group != 0 ) {
    params.channels_per_block = 240;
  }

  // Special case to deal with 70 channels per group.
  if( params.c == 2240 ) {
    params.channels_per_block = 280;
  }

  if (params.c % params.channels_per_block != 0) {
    if (params.c % 512 == 0 && params.c != 1536 && params.c != 3072 && params.c % 448 != 0) {
      params.channels_per_block = 512;
    } else if (params.c % 42 == 0) {
      params.channels_per_block = 336;
    } else if (params.c % 384 == 0) {
      params.channels_per_block = 384;
    } else if (params.c % 256 == 0 && params.c % 448 != 0 && params.c % 392 != 0) {
      params.channels_per_block = 256;
    } else if (params.c % 128 == 0 && params.c % 448 != 0 && params.c % 392 != 0) {
      params.channels_per_block = 128;
    } else if (params.c % 448 == 0 && params.c % 392 != 0) {
      params.channels_per_block = 448;
    } else if (params.c % 392 == 0) {
      params.channels_per_block = 392;
    }
  }

  // The number of groups per block.
  params.groups_per_block = params.channels_per_block / params.channels_per_group;

  // Make sure the number of channels is a multiple of the number of channels per block.
  assert(params.c % params.channels_per_block == 0);
  // Make sure a group does not span multiple blocks.
  assert(params.channels_per_block % params.channels_per_group == 0);

  // The number of elements in the reduction buffer (for the sums and sums of squared). 
  zeroed_red_buffer_elts = params.n * params.groups * 2;
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void group_norm_nhwc_fwd_two_passes_sum(const Group_norm_nhwc_fwd_params &params, 
                                        hipStream_t stream) {

  // The dimension of the grid.
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params.c / params.channels_per_block;
  // The number of blocks to compute all the activations in a given instance.
  grid.y = div_up(params.hw, params.acts_per_block);
  // The number of instances.
  grid.z = params.n;

  // Launch the kernel.
  if (params.precision == PrecisionMode::FP16) {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_sum_kernel, Fp16)
  } else if (params.precision == PrecisionMode::BF16) {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_sum_kernel, Bf16)
  } else {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_sum_kernel, Fp32)
  }

  // Make sure it launched ok.
  CHECK_CUDA(hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////////////////////////

template< typename Traits_, int THREADS_PER_BLOCK >
__global__ void group_norm_nhwc_fwd_scale_kernel(Group_norm_nhwc_fwd_params params) {

  // The IO traits.
  using Traits = Traits_;
  // The IO type
  using IOType = typename Traits::IOType;
  // The IO doubled type
  using IOType2 = typename Traits::IOType2;

  // The instance in the batch.
  int ni = blockIdx.z;
  // The channel loaded by that thread (2 channels per thread for F16x2).
  int ci = blockIdx.x * params.channels_per_block + threadIdx.x * 2;
  // The group that thread works on and the channel in the group (modulus).
  int gi = ci / params.channels_per_group;

  // Load the sum and sum of squares for the group.
  float sum = 0.f, sum_sq = 0.f;
  if( gi < params.groups ) {
    sum    = params.zeroed_red_buffer[(2*ni+0)*params.groups + gi];
    sum_sq = params.zeroed_red_buffer[(2*ni+1)*params.groups + gi];
  }

  // Load gamma/beta.
  float2 gamma_f2, beta_f2;
  if( ci < params.c ) {
    gamma_f2 = *reinterpret_cast<const float2*>(&params.gamma[ci]);
    beta_f2  = *reinterpret_cast<const float2*>(&params.beta [ci]);
  }

  // Compute the mean.
  float mean = sum * params.inv_hwc_per_group;
  // Compute the variance.
  float var = sum_sq * params.inv_hwc_per_group - (mean * mean);
  // Compute the inverse of the stddev.
  float inv_stddev = var <= 0.f ? 1.f : rsqrtf(var + params.epsilon);

  // The first activation loaded by that block.
  int hw_begin = blockIdx.y * params.acts_per_block;
  // The last activation loaded by that block.
  int hw_end = min(hw_begin + params.acts_per_block, params.hw);

  // Iterate over the activations to compute the sums.
  for( int hwi = hw_begin; hwi < hw_end; ++hwi ) {

    // The src/dst offset.
    int64_t offset = (int64_t) ni*params.hwc + hwi*params.c + ci;

    // Fetch two channels per thread.
    IOType2 v2 = Traits::zero();
    if( ci < params.c ) {
      v2  = *reinterpret_cast<const IOType2*>(&reinterpret_cast<const IOType*>(params.x )[offset]);
    }

    // Extract the two values.
    float2 f2 = Traits::unpack(v2);

    // Normalize the channels.
    f2.x = (f2.x - mean) * inv_stddev;
    f2.y = (f2.y - mean) * inv_stddev;

    // Scale by gamma and add beta.
    f2.x = gamma_f2.x * f2.x + beta_f2.x;
    f2.y = gamma_f2.y * f2.y + beta_f2.y;

    // Apply Swish if needed.
    if( params.with_swish ) {
      f2.x = f2.x * sigmoid(f2.x);
      f2.y = f2.y * sigmoid(f2.y);
    }

    // Store the scaled values.
    if( ci < params.c ) {
      *reinterpret_cast<IOType2*>(&reinterpret_cast<IOType*>(params.y)[offset]) = Traits::pack(f2);
    }
  }

  // Write the sums if needed.
  if( params.sums != nullptr && gi < params.groups ) {
    float2 sums;
    sums.x = sum    * params.inv_hwc_per_group;
    sums.y = sum_sq * params.inv_hwc_per_group;
    params.sums[ni*params.groups + gi] = sums;
  }
}

////////////////////////////////////////////////////////////////////////////////////////////////////

void group_norm_nhwc_fwd_two_passes_scale(const Group_norm_nhwc_fwd_params &params, 
                                          hipStream_t stream) {

  // The dimension of the grid.
  dim3 grid;

  // The number of blocks to compute all the channels.
  grid.x = params.c / params.channels_per_block;
  // The number of blocks to compute all the activations in a given instance.
  grid.y = div_up(params.hw, params.acts_per_block);
  // The number of instances.
  grid.z = params.n;

  // Launch the kernel.
  if (params.precision == PrecisionMode::FP16) {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_scale_kernel, Fp16)
  } else if (params.precision == PrecisionMode::BF16) {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_scale_kernel, Bf16)
  } else {
    CALL_TWO_PASS_KERNEL(group_norm_nhwc_fwd_scale_kernel, Fp32)
  }

  // Make sure it launched ok.
  CHECK_CUDA(hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////////////////////////

