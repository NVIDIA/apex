#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
//#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>

#include "dropout.cuh"
#include "softmax.cuh"
#include "strided_batched_gemm.cuh"

namespace multihead_attn {
namespace self_bias {
namespace rocblas_gemmex {

std::vector<torch::Tensor>
fwd_cuda(bool use_time_mask, bool is_training, int heads,
         torch::Tensor const &inputs, torch::Tensor const &input_weights,
         torch::Tensor const &output_weights, torch::Tensor const &input_biases,
         torch::Tensor const &output_biases, const uint8_t *pad_mask,
         float dropout_prob) {
  const int embed_dim = inputs.size(2);
  const int sequences = inputs.size(1);
  const int q_seq_len = inputs.size(0);
  const int k_seq_len = q_seq_len;
  const int batches = sequences * q_seq_len;
  const int head_dim = embed_dim / heads;
  const int output_lin_dim = 3 * embed_dim;
  const int attn_batches = heads * sequences;
  const int lead_dim = attn_batches * 3 * head_dim;
  const int batch_stride = 3 * head_dim;
  const int dropout_elems = attn_batches * q_seq_len * k_seq_len;
  const float alpha = 1.0;
  const float beta_zero = 0.0;
  const float beta_one = 1.0;
  const float scale = 1.0 / sqrt(static_cast<float>(head_dim));

  // There is no reason to use more than one stream as every kernel is
  // sequentially dependent
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  // 3 Intermediate Results + Output (Note: dropout intermediates are generated
  // by ATen library code)
  auto act_options = inputs.options().requires_grad(false);
  auto mask_options = act_options.dtype(torch::kUInt8);

  torch::Tensor input_lin_results =
      torch::empty({q_seq_len, sequences, output_lin_dim}, act_options);
  torch::Tensor softmax_results =
      torch::empty({attn_batches, q_seq_len, k_seq_len}, act_options);
  torch::Tensor dropout_results =
      torch::empty({attn_batches, q_seq_len, k_seq_len}, act_options);
  torch::Tensor dropout_mask =
      torch::empty({attn_batches, q_seq_len, k_seq_len}, mask_options);
  torch::Tensor matmul2_results =
      torch::empty({q_seq_len, attn_batches, head_dim}, act_options);
  torch::Tensor outputs = torch::empty_like(inputs, act_options);

  // Input Linear Results Pointers to Q, K, and V of interviewed activations
  void *q_lin_results_ptr = static_cast<void *>(input_lin_results.data_ptr());
  void *k_lin_results_ptr = static_cast<void *>(
      static_cast<half *>(input_lin_results.data_ptr()) + head_dim);
  void *v_lin_results_ptr = static_cast<void *>(
      static_cast<half *>(input_lin_results.data_ptr()) + 2 * head_dim);

  // Softmax Intermediate Result Ptr (used by Matmul1 -> Softmax)
  void *softmax_results_ptr = static_cast<void *>(softmax_results.data_ptr());

  char a_layout_t{'t'};
  char a_layout_n{'n'};
  char b_layout_n{'n'};

  rocblas_int flags = 0;

  //TORCH_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));

  // Input Linear Fwd
  input_lin_results.copy_(input_biases);
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_T), 
                             hipOperationToRocOperation(HIPBLAS_OP_N),
                             output_lin_dim, 
                             batches, 
                             embed_dim,
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(input_weights.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<const void*>(inputs.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim, 
                             static_cast<const void*>(&beta_one),
                             q_lin_results_ptr,
                             rocblas_datatype_f16_r, 
                             output_lin_dim,
                             q_lin_results_ptr,       
                             rocblas_datatype_f16_r,  
                             output_lin_dim,          
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));

  // MatMul1 of Dot-Product Attention Plus scaling by 1/Sqrt(head size)
  gemm_switch_fp32accum(     a_layout_t, 
                             b_layout_n, 
                             k_seq_len,
                             q_seq_len,
                             head_dim,
                             scale, 
                             static_cast<const half*>(k_lin_results_ptr), 
                             lead_dim, 
                             batch_stride, 
                             static_cast<const half*>(q_lin_results_ptr),
                             lead_dim, 
                             batch_stride, 
                             beta_zero, 
                             static_cast<half*>(softmax_results_ptr), 
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             static_cast<half*>(softmax_results_ptr), 
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             attn_batches,
                             flags);
  
  // Padded Softmax
  bool softmax_success = false;
  if (pad_mask == nullptr) {
    softmax_success = dispatch_softmax<half, half, float>(
        reinterpret_cast<half *>(softmax_results_ptr),
        reinterpret_cast<const half *>(softmax_results_ptr), k_seq_len,
        k_seq_len, attn_batches * q_seq_len);
  } else {
    if (use_time_mask) {
      softmax_success = dispatch_time_masked_softmax<half, half, float>(
          reinterpret_cast<half *>(softmax_results_ptr),
          reinterpret_cast<const half *>(softmax_results_ptr), pad_mask,
          k_seq_len, k_seq_len, attn_batches * q_seq_len, q_seq_len);
    } else {
      softmax_success = dispatch_masked_softmax<half, half, float>(
          reinterpret_cast<half *>(softmax_results_ptr),
          reinterpret_cast<const half *>(softmax_results_ptr), pad_mask,
          k_seq_len, k_seq_len, attn_batches * q_seq_len,
          attn_batches * q_seq_len / sequences);
    }
  }

  if (is_training) {
    // use at:: function so that C++ version generates the same random mask as
    // python version
    auto dropout_tuple =
        at::_fused_dropout(softmax_results, 1.0f - dropout_prob);
    dropout_results = std::get<0>(dropout_tuple);
    dropout_mask = std::get<1>(dropout_tuple);
  }

  // Matmul2
  gemm_switch_fp32accum(     a_layout_n, 
                             b_layout_n, 
                             head_dim, 
                             q_seq_len, 
                             k_seq_len, 
                             alpha, 
                             static_cast<const half*>(v_lin_results_ptr), 
                             lead_dim, 
                             batch_stride, 
                             (is_training) ? static_cast<const half*>(dropout_results.data_ptr()) : static_cast<const half*>(softmax_results.data_ptr()) , 
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             beta_zero, 
                             static_cast<half*>(matmul2_results.data_ptr()), 
                             head_dim*attn_batches, 
                             head_dim, 
                             static_cast<half*>(matmul2_results.data_ptr()), 
                             head_dim*attn_batches, 
                             head_dim, 
                             attn_batches,
                             flags);

  outputs.copy_(output_biases);

  // Output Linear
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_T), 
                             hipOperationToRocOperation(HIPBLAS_OP_N),
                             embed_dim, 
                             batches, 
                             embed_dim,
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(output_weights.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<const void*>(matmul2_results.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim, 
                             static_cast<const void*>(&beta_one),
                             static_cast<void*>(outputs.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<void*>(outputs.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));
  //TORCH_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));

  return {input_lin_results, softmax_results, dropout_results,
          dropout_mask,      matmul2_results, outputs};
}

std::vector<torch::Tensor> bwd_cuda(
    int heads, torch::Tensor const &output_grads,
    torch::Tensor const &matmul2_results, torch::Tensor const &dropout_results,
    torch::Tensor const &softmax_results,
    torch::Tensor const &input_lin_results, torch::Tensor const &inputs,
    torch::Tensor const &input_weights, torch::Tensor const &output_weights,
    torch::Tensor const &dropout_mask, float dropout_prob) {
  const int embed_dim = inputs.size(2);
  const int sequences = inputs.size(1);
  const int q_seq_len = inputs.size(0);
  const int k_seq_len = q_seq_len;
  const int batches = sequences * q_seq_len;
  const int head_dim = embed_dim / heads;
  const int output_lin_dim = 3 * embed_dim;
  const int attn_batches = heads * sequences;
  const int lead_dim = attn_batches * 3 * head_dim;
  const int batch_stride = 3 * head_dim;
  const int dropout_elems = attn_batches * q_seq_len * k_seq_len;
  const float alpha = 1.0;
  const float beta = 0.0;
  const float scale = 1.0 / sqrt(static_cast<float>(head_dim));

  // TODO: Streams can be used in Backprop but I haven't added more than one
  // in my first attempt to create the code
  hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();
  hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
  hipblasSetStream(handle, stream);

  // Output Tensor Allocations
  torch::Tensor input_grads = torch::empty_like(inputs);
  torch::Tensor input_weight_grads = torch::empty_like(input_weights);
  torch::Tensor output_weight_grads = torch::empty_like(output_weights);
  // Intermediate Tensor Allocations
  at::Tensor output_lin_grads = torch::empty_like(matmul2_results);
  at::Tensor matmul2_grads = torch::empty_like(dropout_results);
  at::Tensor input_lin_output_grads = torch::empty_like(input_lin_results);

  auto q_lin_results_ptr = static_cast<half *>(input_lin_results.data_ptr());
  auto k_lin_results_ptr =
      static_cast<half *>(input_lin_results.data_ptr()) + head_dim;
  auto v_lin_results_ptr =
      static_cast<half *>(input_lin_results.data_ptr()) + 2 * head_dim;

  auto q_lin_grads_ptr = static_cast<half *>(input_lin_output_grads.data_ptr());
  auto k_lin_grads_ptr =
      static_cast<half *>(input_lin_output_grads.data_ptr()) + head_dim;
  auto v_lin_grads_ptr =
      static_cast<half *>(input_lin_output_grads.data_ptr()) + 2 * head_dim;

  char a_layout_n{'n'};
  char a_layout_t{'t'};
  char b_layout_n{'n'};
  char b_layout_t{'t'};

  rocblas_int flags = 0;

  //TORCH_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_TENSOR_OP_MATH));
  #ifdef __HIP_PLATFORM_HCC__
    #define PYTORCH_ROCBLAS_VERSION_DECIMAL (ROCBLAS_VERSION_MAJOR * 100 + ROCBLAS_VERSION_MINOR)
    #define USE_GEMM_FLAGS_FP16_ALT_IMPL (PYTORCH_ROCBLAS_VERSION_DECIMAL >= 242)
    #if USE_GEMM_FLAGS_FP16_ALT_IMPL
      #ifdef BACKWARD_PASS_GUARD
        flags = at::BACKWARD_PASS_GUARD_CLASS::is_backward_pass() ? rocblas_gemm_flags_fp16_alt_impl : 0;
      #endif
    #endif
  #endif

  // Output Linear Dgrad
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_N), 
                             hipOperationToRocOperation(HIPBLAS_OP_N),
                             embed_dim, 
                             batches, 
                             embed_dim,
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(output_weights.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<const void*>(output_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim, 
                             static_cast<const void*>(&beta),
                             static_cast<void*>(output_lin_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<void*>(output_lin_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));

  // Output Linear Wgrad
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_N), 
                             hipOperationToRocOperation(HIPBLAS_OP_T),
                             embed_dim, 
                             embed_dim,
                             batches, 
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(matmul2_results.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<const void*>(output_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim, 
                             static_cast<const void*>(&beta),
                             static_cast<void*>(output_weight_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<void*>(output_weight_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));

  auto  output_bias_grads = output_grads.view({-1, embed_dim}) .sum(0, false);
  // MatMul2 Dgrad1
  gemm_switch_fp32accum(     a_layout_t, 
                             b_layout_n, 
                             k_seq_len,
                             q_seq_len,
                             head_dim,
                             alpha, 
                             static_cast<const half*>(v_lin_results_ptr),
                             lead_dim, 
                             batch_stride,
                             static_cast<const half*>(output_lin_grads.data_ptr()),
                             head_dim*attn_batches, 
                             head_dim, 
                             beta, 
                             static_cast<half*>(matmul2_grads.data_ptr()),
                             k_seq_len, 
                             k_seq_len*q_seq_len,
                             static_cast<half*>(matmul2_grads.data_ptr()),
                             k_seq_len, 
                             k_seq_len*q_seq_len,
                             attn_batches,
                             flags);

  // Matmul2 Dgrad2
  gemm_switch_fp32accum(     a_layout_n, 
                             b_layout_t, 
                             head_dim, 
                             k_seq_len, 
                             q_seq_len, 
                             alpha, 
                             static_cast<const half*>(output_lin_grads.data_ptr()),
                             head_dim*attn_batches, 
                             head_dim, 
                             static_cast<const half*>(dropout_results.data_ptr()),
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             beta, 
                             v_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride, 
                             v_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride, 
                             attn_batches,
                             flags);

  // Apply Dropout Mask and Scale by Dropout Probability 
  // Softmax Grad
  dispatch_masked_scale_softmax_backward_stream<half, half, float, false>(
      static_cast<half *>(matmul2_grads.data_ptr()),
      static_cast<half *>(matmul2_grads.data_ptr()),
      reinterpret_cast<half const *>(softmax_results.data_ptr()),
      static_cast<uint8_t const *>(dropout_mask.data_ptr()),
      1.0 / (1.0 - dropout_prob), k_seq_len, k_seq_len,
      attn_batches * q_seq_len, stream);

  // Matmul1 Dgrad1
  gemm_switch_fp32accum(     a_layout_n, 
                             b_layout_n, 
                             head_dim, 
                             q_seq_len, 
                             k_seq_len, 
                             scale, 
                             k_lin_results_ptr, 
                             lead_dim, 
                             batch_stride, 
                             static_cast<half*>(matmul2_grads.data_ptr()),
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             beta,
                             q_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride,  
                             q_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride, 
                             attn_batches,
                             flags);

  // Matmul1 Dgrad2
  gemm_switch_fp32accum(     a_layout_n, 
                             b_layout_t, 
                             head_dim, 
                             k_seq_len, 
                             q_seq_len, 
                             scale, 
                             q_lin_results_ptr, 
                             lead_dim, 
                             batch_stride, 
                             static_cast<half*>(matmul2_grads.data_ptr()),
                             k_seq_len, 
                             k_seq_len*q_seq_len, 
                             beta, 
                             k_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride,
                             k_lin_grads_ptr, 
                             lead_dim, 
                             batch_stride, 
                             attn_batches,
                             flags);
  // Input Linear Dgrad  
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_N), 
                             hipOperationToRocOperation(HIPBLAS_OP_N),
                             embed_dim,
                             batches, 
                             output_lin_dim,
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(input_weights.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<const void*>(input_lin_output_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             output_lin_dim, 
                             static_cast<const void*>(&beta),
                             static_cast<void*>(input_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<void*>(input_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));

  // Input Linear Wgrad  
  TORCH_CUDABLAS_CHECK(rocBLASStatusToHIPStatus(rocblas_gemm_ex((rocblas_handle) handle,
                             hipOperationToRocOperation(HIPBLAS_OP_N), 
                             hipOperationToRocOperation(HIPBLAS_OP_T),
                             embed_dim, 
                             output_lin_dim,
                             batches, 
                             static_cast<const void*>(&alpha),
                             static_cast<const void*>(inputs.data_ptr()),
                             rocblas_datatype_f16_r,
                             embed_dim,
                             static_cast<const void*>(q_lin_grads_ptr),
                             rocblas_datatype_f16_r,
                             output_lin_dim,
                             static_cast<const void*>(&beta),
                             static_cast<void*>(input_weight_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             static_cast<void*>(input_weight_grads.data_ptr()),
                             rocblas_datatype_f16_r, 
                             embed_dim,
                             rocblas_datatype_f32_r,
                             rocblas_gemm_algo_standard /*algo*/,
                             0 /*solution_index*/,
                             flags)));

  auto  input_bias_grads = input_lin_output_grads.view({-1, output_lin_dim}).sum(0, false);
  //TORCH_CUDABLAS_CHECK(hipblasSetMathMode(handle, HIPBLAS_DEFAULT_MATH));

  return {input_grads, input_weight_grads, output_weight_grads,
          input_bias_grads, output_bias_grads};
}

} // end namespace rocblas_gemmex
} // end namespace self
} // end namespace multihead_attn
