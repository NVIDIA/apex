#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "fmha.h"
#include "fmha_fprop_kernel_1xN.h"

using Kernel_traits = FMHA_kernel_traits<384, 64, 16, 1, 4, 0x18u>;

template<bool Is_training>
__global__ 
void fmha_fprop_fp16_384_64_sm80_kernel(Fused_multihead_attention_fprop_params params,
                                           const int num_full_heads,
                                           const int num_main_groups,
                                           const int main_group_size,
                                           const int main_steps,
                                           const int rest_steps) {

    fmha::device_1xN<Kernel_traits, Is_training>(
        params, num_full_heads, num_main_groups, main_group_size, main_steps, rest_steps);
}

void run_fmha_fp16_384_64_sm80(Launch_params<Fused_multihead_attention_fprop_params> &launch_params, const bool configure) {

    auto kernel = launch_params.is_training ? &fmha_fprop_fp16_384_64_sm80_kernel<true> : &fmha_fprop_fp16_384_64_sm80_kernel<false>;

    constexpr int smem_size = fmha::get_dynamic_smem_size<Kernel_traits>();

    if( smem_size >= 48 * 1024 ) {
        FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }

    const int sm_count = launch_params.props->multiProcessorCount;
    int ctas_per_sm;
    FMHA_CHECK_CUDA(hipOccupancyMaxActiveBlocksPerMultiprocessor(&ctas_per_sm, kernel, Kernel_traits::THREADS, smem_size));
    int total_ctas = sm_count * ctas_per_sm;

    if(configure) {
        const int heads_total = launch_params.params.b * launch_params.params.h;
        std::tie(launch_params.num_full_heads,
                 launch_params.num_main_groups, 
                 launch_params.heads_last_wave, 
                 launch_params.main_steps, 
                 launch_params.rest_steps, 
                 launch_params.elts_per_thread) = fmha::work_dist<Kernel_traits>(total_ctas, heads_total);
        return;
    }

    dim3 grid(total_ctas);
    kernel<<<grid, Kernel_traits::THREADS, smem_size, launch_params.stream>>>(
        launch_params.params,
        launch_params.num_full_heads, 
        launch_params.num_main_groups, 
        launch_params.heads_last_wave, 
        launch_params.main_steps, 
        launch_params.rest_steps);

    FMHA_CHECK_CUDA(hipPeekAtLastError());

}

