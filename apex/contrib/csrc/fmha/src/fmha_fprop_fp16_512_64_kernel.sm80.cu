#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2021, NVIDIA CORPORATION.  All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 * 
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

#include "fmha.h"
#include "fmha_fprop_kernel_1xN.h"

using Kernel_traits = FMHA_kernel_traits< 512, 64, 16, 1, 8, 0x08u>;

extern "C" __global__ void fmha_fprop_fp16_512_64_sm80_train_kernel(Fused_multihead_attention_fprop_params params) {
    fmha::device_1xN<Kernel_traits, true>(params);
}

extern "C" __global__ void fmha_fprop_fp16_512_64_sm80_predict_kernel(Fused_multihead_attention_fprop_params params) {
    fmha::device_1xN<Kernel_traits, false>(params);
}

void run_fmha_fp16_512_64_sm80(const Fused_multihead_attention_fprop_params &params, bool is_training, hipStream_t stream) {

    auto kernel = is_training ? &fmha_fprop_fp16_512_64_sm80_train_kernel : &fmha_fprop_fp16_512_64_sm80_predict_kernel;

    constexpr int smem_size_softmax = Kernel_traits::Cta_tile_p::M * Kernel_traits::Cta_tile_p::WARPS_N * sizeof(float);
    constexpr int smem_size_q = Kernel_traits::Smem_tile_q::BYTES_PER_TILE;
    constexpr int smem_size_v = Kernel_traits::Smem_tile_v::BYTES_PER_TILE;
    constexpr int smem_size_o = Kernel_traits::Smem_tile_o::BYTES_PER_TILE;

    constexpr int smem_size = smem_size_q + std::max(smem_size_v, smem_size_o + smem_size_softmax);
    if( smem_size >= 48 * 1024 ) {
        FMHA_CHECK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
    }
    dim3 grid(params.h, params.b);
    kernel<<<grid, Kernel_traits::THREADS, smem_size, stream>>>(params);
}
