#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "type_shim.h"
#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template<typename T>
__device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

template <typename FROM_T, typename TO_T> 
__device__ void convert(const FROM_T vi, TO_T& vo)
{
    vo = static_cast<TO_T>(vi);
}

template <>
__device__ void convert(const float vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = vi;
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, float& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = static_cast<float>(t.as_half);
}

template <>
__device__ void convert(const at::Half vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = static_cast<float>(vi);
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, at::Half& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = t.as_half;
}

typedef enum{
  MOMENT_MODE_0   =0, // L2 regularization mode
  MOMENT_MODE_1   =1  // Decoupled weight decay mode
} adamMode_t;

template<typename T, typename GRAD_T, typename MATH_T>
struct DistOptLAMBStage1Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<5>& tl,
    const MATH_T* per_tensor_beta1,
    const MATH_T* per_tensor_beta2,
    const MATH_T* per_tensor_beta3,
    const int* per_tensor_bias_correction,
    const int* step,
    const MATH_T* per_tensor_epsilon,
    adamMode_t mode,
    const MATH_T* per_tensor_decay,
    const MATH_T* global_scale,
    const MATH_T* global_grad_norm,
    const float max_grad_norm)
  {
    // I'd like this kernel to propagate infs/nans.
    if (*noop_gmem == 1)
        return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    float combined_scale = *global_scale;
    if (max_grad_norm > 0) {
        combined_scale = max_grad_norm / (*global_grad_norm / *global_scale + 1e-6);
	combined_scale = *global_scale / std::min((float) 1.0, combined_scale);
    }

    MATH_T beta1 = per_tensor_beta1[tensor_num];
    MATH_T beta2 = per_tensor_beta2[tensor_num];
    MATH_T beta3 = 1 - beta1;
    MATH_T beta1_correction, beta2_correction;
    if (per_tensor_bias_correction[tensor_num] == 1) {
        beta1_correction = 1 - pow(beta1, *step);
        beta2_correction = 1 - pow(beta2, *step);
    } else {
        beta1_correction = (MATH_T) 1.0;
        beta2_correction = (MATH_T) 1.0;
    }
    MATH_T epsilon = per_tensor_epsilon[tensor_num];
    MATH_T decay = per_tensor_decay[tensor_num];

    GRAD_T* g = (GRAD_T*)tl.addresses[0][tensor_loc];
    g += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    T* m = (T*)tl.addresses[2][tensor_loc];
    m += chunk_idx*chunk_size;

    T* v = (T*)tl.addresses[3][tensor_loc];
    v += chunk_idx*chunk_size;

    MATH_T* u = (MATH_T*)tl.addresses[4][tensor_loc];
    u += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    MATH_T r_g[ILP];
    MATH_T r_p[ILP];
    MATH_T r_m[ILP];
    MATH_T r_v[ILP];
    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 &&
       chunk_size % ILP == 0 &&
       is_aligned(g) &&
       is_aligned(p) &&
       is_aligned(m) &&
       is_aligned(v))
    {
      GRAD_T l_g[ILP];
      T l_p[ILP];
      T l_m[ILP];
      T l_v[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(l_g, g, 0, i_start);
        if (decay != 0)
          load_store(l_p, p, 0, i_start);
        load_store(l_m, m, 0, i_start);
        load_store(l_v, v, 0, i_start);
        // unpack
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_g[ii] = l_g[ii];
          if (decay == 0) {
            r_p[ii] = MATH_T(0);
          }
          else {
            r_p[ii] = l_p[ii];
          }
          r_m[ii] = l_m[ii];
          r_v[ii] = l_v[ii];
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / combined_scale;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / combined_scale;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          l_m[ii] = r_m[ii];
          l_v[ii] = r_v[ii];
        }
        // store
        load_store(u, r_p, i_start, 0);
        load_store(m, l_m, i_start, 0);
        load_store(v, l_v, i_start, 0);
      }
    }
    else
    {
      // see note in multi_tensor_scale_kernel.cu
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_g[ILP];
        MATH_T r_p[ILP];
        MATH_T r_m[ILP];
        MATH_T r_v[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_g[ii] = g[i];
            // special ?optimization? for lamb stage 1
            if (decay == 0) {
              r_p[ii] = MATH_T(0);
            }
            else {
              r_p[ii] = p[i];
            }
            r_m[ii] = m[i];
            r_v[ii] = v[i];
          } else {
            r_g[ii] = MATH_T(0);
            r_p[ii] = MATH_T(0);
            r_m[ii] = MATH_T(0);
            r_v[ii] = MATH_T(0);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          if (mode == MOMENT_MODE_0) {
            MATH_T scaled_grad = r_g[ii] / combined_scale;
            // L2 on scaled grad
            scaled_grad = scaled_grad + decay*r_p[ii];
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = next_m_unbiased / denom;
          }
          else {
            MATH_T scaled_grad = r_g[ii] / combined_scale;
            r_m[ii] = r_m[ii] * beta1 + beta3 * scaled_grad;
            r_v[ii] = r_v[ii] * beta2 + (1-beta2) * scaled_grad * scaled_grad;
            MATH_T next_m_unbiased = r_m[ii] / beta1_correction;
            MATH_T next_v_unbiased = r_v[ii] / beta2_correction;
            MATH_T denom = sqrtf(next_v_unbiased) + epsilon;
            r_p[ii] = (next_m_unbiased/denom) + (decay*r_p[ii]);
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            u[i] = r_p[ii];
            m[i] = r_m[ii];
            v[i] = r_v[ii];
          }
        }
      }
    }
  }
};

// Step 2 reads in 'update' value and per-tensor param_norm and update_norm.
// It computes new parameter value.
template<typename T, typename GRAD_T, typename MATH_T>
struct DistOptLAMBStage2Functor
{
   __device__ __forceinline__ void operator()(
    int chunk_size,
    volatile int* noop_gmem,
    TensorListMetadata<3>& tl,
    const MATH_T* per_tensor_param_norm,
    const MATH_T* per_tensor_update_norm,
    const long* update_norm_offset,
    const MATH_T* learning_rate,
    const MATH_T* per_tensor_decay,
    const MATH_T* global_grad_norm,
    bool use_nvlamb)
  {
    // I'd like this kernel to propagate infs/nans.
    if (*noop_gmem == 1)
        return;

    int tensor_loc = tl.block_to_tensor[blockIdx.x];
    int tensor_num = tl.start_tensor_this_launch + tensor_loc;
    int chunk_idx = tl.block_to_chunk[blockIdx.x];
    int n = tl.sizes[tensor_loc];

    MATH_T decay = per_tensor_decay[tensor_num];

    MATH_T ratio = *learning_rate;
    // nvlamb: apply adaptive learning rate to all parameters
    // otherwise, only apply to those with non-zero weight decay
    if (use_nvlamb || (decay != (MATH_T) 0.0))
    {
      MATH_T param_norm = per_tensor_param_norm[tensor_num];
      MATH_T update_norm = per_tensor_update_norm[update_norm_offset[tensor_num]];
      ratio = (update_norm != 0.0 && param_norm != 0.0) ? (*learning_rate) * (param_norm / update_norm) : (*learning_rate);
    }

    MATH_T* update = (MATH_T*)tl.addresses[0][tensor_loc];
    update += chunk_idx*chunk_size;

    T* p = (T*)tl.addresses[1][tensor_loc];
    p += chunk_idx*chunk_size;

    GRAD_T* p_copy = (GRAD_T*)tl.addresses[2][tensor_loc];
    p_copy += chunk_idx*chunk_size;

    n -= chunk_idx*chunk_size;

    // to make things simple, we put aligned case in a different code path
    if(n % ILP == 0 &&
       chunk_size % ILP == 0 &&
       is_aligned(p) &&
       is_aligned(update))
    {
      T r_p[ILP];
      MATH_T r_update[ILP];
      GRAD_T r_p_copy[ILP];
      for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
      {
        // load
        load_store(r_p, p, 0, i_start);
        load_store(r_update, update, 0, i_start);
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
	  r_p[ii] = static_cast<MATH_T>(r_p[ii]) - (ratio * r_update[ii]);
          convert(r_p[ii], r_p_copy[ii]);
        }
        load_store(p, r_p, i_start, 0);
        load_store(p_copy, r_p_copy, i_start, 0);
      }
    }
    else
    {
      for(int i_start = 0;
          i_start < n && i_start < chunk_size;
          i_start += blockDim.x*ILP)
      {
        MATH_T r_p[ILP];
        MATH_T r_update[ILP];
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            r_p[ii] = p[i];
            r_update[ii] = update[i];
          }
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          r_p[ii] = r_p[ii] - (ratio * r_update[ii]);
        }
#pragma unroll
        for(int ii = 0; ii < ILP; ii++)
        {
          int i = i_start + threadIdx.x + ii*blockDim.x;
          if(i < n && i < chunk_size)
          {
            p[i] = r_p[ii];
            convert(r_p[ii], p_copy[i]);
          }
        }
      }
    }
  }
};

void multi_tensor_lamb_compute_update_term_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor per_tensor_beta1,
  at::Tensor per_tensor_beta2,
  at::Tensor per_tensor_beta3,
  at::Tensor per_tensor_bias_correction,
  at::Tensor step,
  at::Tensor per_tensor_epsilon,
  const int mode,
  at::Tensor per_tensor_decay,
  at::Tensor global_scale,
  at::Tensor global_grad_norm,
  const float max_grad_norm)
{
  using namespace at;

  DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 0, "lamb_stage_1",
    DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 1, "lamb_stage_1",
      DISPATCH_FLOAT_AND_HALF(tensor_lists[4][0].scalar_type(), 2, "lamb_stage_1",
        multi_tensor_apply<5>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          tensor_lists,
          DistOptLAMBStage1Functor<scalar_t_0, scalar_t_1, scalar_t_2>(),
          per_tensor_beta1.DATA_PTR<scalar_t_2>(),
          per_tensor_beta2.DATA_PTR<scalar_t_2>(),
          per_tensor_beta3.DATA_PTR<scalar_t_2>(),
          per_tensor_bias_correction.DATA_PTR<int>(),
          step.DATA_PTR<int>(),
          per_tensor_epsilon.DATA_PTR<scalar_t_2>(),
          (adamMode_t) mode,
          per_tensor_decay.DATA_PTR<scalar_t_2>(),
          global_scale.DATA_PTR<scalar_t_2>(),
	  global_grad_norm.DATA_PTR<scalar_t_2>(),
	  max_grad_norm); )))

  AT_CUDA_CHECK(hipGetLastError());
}

void multi_tensor_lamb_update_weights_cuda(
  int chunk_size,
  at::Tensor noop_flag,
  std::vector<std::vector<at::Tensor>> tensor_lists,
  at::Tensor per_tensor_param_norm,
  at::Tensor per_tensor_update_norm,
  at::Tensor update_norm_offset,
  at::Tensor learning_rate,
  at::Tensor per_tensor_decay,
  at::Tensor global_grad_norm,
  bool use_nvlamb)
{
  using namespace at;

  DISPATCH_FLOAT_AND_HALF(tensor_lists[1][0].scalar_type(), 0, "lamb_stage_2",
    DISPATCH_FLOAT_HALF_AND_BYTE(tensor_lists[2][0].scalar_type(), 1, "lamb_stage_2",
      DISPATCH_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(), 2, "lamb_stage_2",
        multi_tensor_apply<3>(
          BLOCK_SIZE,
          chunk_size,
          noop_flag,
          tensor_lists,
          DistOptLAMBStage2Functor<scalar_t_0, scalar_t_1, scalar_t_2>(),
          per_tensor_param_norm.DATA_PTR<scalar_t_2>(),
          per_tensor_update_norm.DATA_PTR<scalar_t_2>(),
          update_norm_offset.DATA_PTR<long>(),
	  learning_rate.DATA_PTR<scalar_t_2>(),
          per_tensor_decay.DATA_PTR<scalar_t_2>(),
	  global_grad_norm.DATA_PTR<scalar_t_2>(),
          use_nvlamb); )))

  AT_CUDA_CHECK(hipGetLastError());
}
