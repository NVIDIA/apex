#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cmath>

#include "ATen/ATen.h"
#include "ATen/cuda/HIPContext.h"
#include "ATen/cuda/detail/IndexUtils.cuh"
#include "ATen/TensorUtils.h"
// #include "ATen/Type.h"
#include "ATen/AccumulateType.h"

#include "multi_tensor_apply.cuh"

#define BLOCK_SIZE 512
#define ILP 4

template<typename T>
__device__ __forceinline__ bool is_aligned(T* p){
  return ((uint64_t)p) % (ILP*sizeof(T)) == 0;
}

template<typename T>
__device__ __forceinline__ void load_store(T* dst, T* src, int dst_offset, int src_offset){
  typedef typename std::aligned_storage<ILP*sizeof(T), ILP*alignof(T)>::type LT;
  ((LT*)dst)[dst_offset] = ((LT*)src)[src_offset];
}

#include "type_shim.h"

typedef enum{
    ADAM_MODE_0   =0, // eps under square root
    ADAM_MODE_1   =1  // eps outside square root
} adamMode_t;

template <typename T, typename GRAD_T>
__global__ void adam_cuda_kernel(
        T* __restrict__ p,
        GRAD_T* __restrict__ p_copy, // For mixed precision training, pass NULL if not needed
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
        //Assuming 2D grids and 2D blocks
        const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
        const int threadsPerBlock = blockDim.x * blockDim.y;
        const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
        const int i = (blockId * threadsPerBlock + threadIdInBlock);
        const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

        for (int j = i; j < tsize; j+=totThreads) {
                T scaled_grad = g[j]/grad_scale;
                m[j] = b1*m[j] + (1-b1)*scaled_grad;
                v[j] = b2*v[j] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(v[j] + eps);
                else // Mode 1
                    denom = sqrtf(v[j]) + eps;
                float update = (m[j]/denom) + (decay*p[j]);
                p[j] = p[j] - (step_size*update);
                if (p_copy != NULL) p_copy[j] = (GRAD_T) p[j];
        }
}

template <int DEPTH, typename T, typename GRAD_T>
struct AdamFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* noop_gmem,
        TensorListMetadata<DEPTH>& tl,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        adamMode_t mode,
        const float decay)
    {
        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* p = (T *)tl.addresses[0][tensor_loc];
        p += chunk_idx*chunk_size;
        T* m = (T *)tl.addresses[1][tensor_loc];
        m += chunk_idx*chunk_size;
        T* v = (T *)tl.addresses[2][tensor_loc];
        v += chunk_idx*chunk_size;
        GRAD_T* g = (GRAD_T *)tl.addresses[3][tensor_loc];
        g += chunk_idx*chunk_size;
        GRAD_T* p_copy = NULL;
        if (DEPTH == 5) {
            p_copy = (GRAD_T *)tl.addresses[4][tensor_loc];
            p_copy += chunk_idx*chunk_size;
        }

        n -= chunk_idx*chunk_size;

        T incoming_p[ILP];
        T incoming_m[ILP];
        T incoming_v[ILP];
        T incoming_g[ILP];

        // to make things simple, we put aligned case in a different code path
        if(n % ILP == 0 &&
           chunk_size % ILP == 0 &&
           is_aligned(p) &&
           is_aligned(m) &&
           is_aligned(v) &&
           is_aligned(g) &&
           is_aligned(p_copy))
        {
          for(int i_start = threadIdx.x; i_start*ILP < n && i_start*ILP < chunk_size; i_start += blockDim.x)
          {
            // load
            GRAD_T tmp_g[ILP];
            load_store(incoming_p, p, 0, i_start);
            load_store(incoming_m, m, 0, i_start);
            load_store(incoming_v, v, 0, i_start);
            load_store(tmp_g, g, 0, i_start);
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
              incoming_g[ii] = static_cast<T>(tmp_g[ii]);
              T scaled_grad = incoming_g[ii]/grad_scale;
              incoming_m[ii] = b1*incoming_m[ii] + (1-b1)*scaled_grad;
              incoming_v[ii] = b2*incoming_v[ii] + (1-b2)*scaled_grad*scaled_grad;
              float denom;
              if (mode == ADAM_MODE_0)
                denom = sqrtf(incoming_v[ii] + eps);
              else // Mode 1
                denom = sqrtf(incoming_v[ii]) + eps;
              float update = (incoming_m[ii]/denom) + (decay*incoming_p[ii]);
              incoming_p[ii] = incoming_p[ii] - (step_size*update);
              if (DEPTH == 5)  tmp_g[ii] = static_cast<GRAD_T>(incoming_p[ii]);
            }
            load_store(p, incoming_p, i_start, 0);
            load_store(m, incoming_m, i_start, 0);
            load_store(v, incoming_v, i_start, 0);
            if (DEPTH == 5) load_store(p_copy, tmp_g, i_start, 0);
          }
        }
        else
        {
          for(int i_start = 0;
              i_start < n && i_start < chunk_size;
              i_start += blockDim.x*ILP) {

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
              incoming_p[ii] = 0;
              incoming_m[ii] = 0;
              incoming_v[ii] = 0;
              incoming_g[ii] = 0;

              int i = i_start + threadIdx.x + ii*blockDim.x;
              if (i < n && i < chunk_size) {
                incoming_p[ii] = p[i];
                incoming_m[ii] = m[i];
                incoming_v[ii] = v[i];
                incoming_g[ii] = static_cast<T>(g[i]);
              }
            }

            // note for clarification to future michael:
            // From a pure memory dependency perspective, there's likely no point unrolling
            // the write loop, since writes just fire off once their LDGs arrive.
            // Put another way, the STGs are dependent on the LDGs, but not on each other.
            // There is still compute ILP benefit from unrolling the loop though.
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
              int j = i_start + threadIdx.x + ii*blockDim.x;

              if(j < n && j < chunk_size) {
                T scaled_grad = incoming_g[ii]/grad_scale;
                m[j] = b1*incoming_m[ii] + (1-b1)*scaled_grad;
                v[j] = b2*incoming_v[ii] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                  denom = sqrtf(v[j] + eps);
                else // Mode 1
                  denom = sqrtf(v[j]) + eps;
                float update = (m[j]/denom) + (decay*incoming_p[ii]);
                p[j] = incoming_p[ii] - (step_size*update);
                if (DEPTH == 5)  p_copy[j] = (GRAD_T) p[j];
              }
            }
          }
        }
    }
};

void fused_adam_cuda(
        at::Tensor & p,
        at::Tensor & p_copy,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//        using namespace at;

        //Get tensor size
        int tsize = p.numel();
        //Determine #threads and #blocks
        const int threadsPerBlock = 512;
        const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
        TORCH_CHECK(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
        //Constants
        float step_size = 0;
        if (bias_correction == 1) {
            const float bias_correction1 = 1 - std::pow(beta1, step);
            const float bias_correction2 = 1 - std::pow(beta2, step);
            step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
        }
        else {
            step_size = lr;
        }
        hipStream_t stream = at::cuda::getCurrentCUDAStream();

        if (g.scalar_type() == at::ScalarType::Half) {
//all other values should be fp32 for half gradients
            TORCH_CHECK(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dispatch is done on the gradient type
            using namespace at; // prevents "toString is undefined" errors
            DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                adam_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<accscalar_t>(),
                        p_copy.numel() ? p_copy.DATA_PTR<scalar_t_0>() : NULL,
                        m.DATA_PTR<accscalar_t>(),
                        v.DATA_PTR<accscalar_t>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
                );
      } else {
            using namespace at;
            DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                adam_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                        p.DATA_PTR<scalar_t_0>(),
                        NULL, //don't output p_copy for fp32, it's wasted write
                        m.DATA_PTR<scalar_t_0>(),
                        v.DATA_PTR<scalar_t_0>(),
                        g.DATA_PTR<scalar_t_0>(),
                        beta1,
                        beta2,
                        eps,
                        grad_scale,
                        step_size,
                        tsize,
                        (adamMode_t) mode,
                        decay);
            );
      }
      C10_CUDA_CHECK(hipGetLastError());

}

void fused_adam_cuda_mt(
    int chunk_size,
    at::Tensor noop_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists, // p, m, v, g, p_copy
    float lr,
    float beta1,
    float beta2,
    float eps,
    float grad_scale,
    int step,
    int mode,
    int bias_correction,
    float decay) {

    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    TORCH_CHECK(tl_sz == 4 || tl_sz == 5, "expected tensor lists of size 4 or 5");

    if (tensor_lists[3][0].scalar_type() == at::ScalarType::Half) {
//alher values should be fp32 for half gradients
        TORCH_CHECK(tensor_lists[0][0].scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
//dich is done on the gradient type
        if (tl_sz == 5) {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                multi_tensor_apply<5>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamFunctor<5, accscalar_t, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
            );
        } else {
            DISPATCH_FLOAT_AND_HALF(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamFunctor<4, accscalar_t, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
            );
        }
    } else {
        if (tl_sz == 5) {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                multi_tensor_apply<5>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamFunctor<5, scalar_t_0, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
            );
        } else {
            DISPATCH_DOUBLE_AND_FLOAT(tensor_lists[3][0].scalar_type(), 0, "adam_cuda_mt_kernel",
                multi_tensor_apply<4>(
                    BLOCK_SIZE,
                    chunk_size,
                    noop_flag,
                    tensor_lists,
                    AdamFunctor<4, scalar_t_0, scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    (adamMode_t) mode,
                    decay);
            );
        }
    }
    C10_CUDA_CHECK(hipGetLastError());
}

template <typename FROM_T, typename TO_T> 
__device__ void convert(const FROM_T vi, TO_T& vo)
{
    vo = static_cast<TO_T>(vi);
}

template <>
__device__ void convert(const float vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = vi;
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, float& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = static_cast<float>(t.as_half);
}

template <>
__device__ void convert(const at::Half vi, uint8_t& vo)
{
    union S
    {
	float as_float;
	int as_int;
    };
    S s;
    s.as_float = static_cast<float>(vi);
    s.as_int = s.as_int & 0xFF800000;
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_half = static_cast<at::Half>(vi + s.as_float / 8.0f);
    vo = t.as_byte[1];
}

template <>
__device__ void convert(const uint8_t vi, at::Half& vo)
{
    union T
    {
        at::Half as_half;
	uint8_t as_byte[2];
    };
    T t;
    t.as_byte[0] = 0;
    t.as_byte[1] = vi;
    vo = t.as_half;
}

template <typename GRAD_T>
__global__ void strided_check_finite_cuda_kernel(
        volatile int* noop_gmem,
        GRAD_T* __restrict__ p_copy,
        const size_t tsize,
        int stride,
        int clear_overflow_first)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock) * stride;
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock*stride;

    if (clear_overflow_first) {
        if (i == 0) {
            *noop_gmem = 0;
        }
        __syncthreads();
    }

    for (int j = i; j < tsize; j+=totThreads) {
        GRAD_T pi = p_copy[j];
        if (!isfinite(pi)) {
            *noop_gmem = 1;
        }
    }
}
template <>
__global__ void strided_check_finite_cuda_kernel(
        volatile int* noop_gmem,
        uint8_t* __restrict__ p_copy,
        const size_t tsize,
        int stride,
        int clear_overflow_first)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock) * stride;
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock*stride;

    if (clear_overflow_first) {
        if (i == 0) {
            *noop_gmem = 0;
        }
        __syncthreads();
    }

    for (int j = i; j < tsize; j+=totThreads) {
        at::Half pi;
        convert(p_copy[j], pi);
        if (!isfinite(pi)) {
            *noop_gmem = 1;
        }
    }
}

template <typename FROM_T, typename TO_T> 
__global__ void maybe_cast_kernel(
        volatile int* overflow_flag,
        const FROM_T* p_in,
        TO_T* p_out,
        const size_t tsize)
{
    if (overflow_flag && *overflow_flag != 0) return;

    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    FROM_T pi[ILP];
    TO_T po[ILP];

    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            pi[ii] = 0;

            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                pi[ii] = p_in[j];
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            convert(pi[ii], po[ii]);
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                p_out[j] = po[ii];
            }
        }
    }
}

template <typename T, typename GRAD_T, typename REDU_T>
__global__ void reversible_adam_cuda_kernel(
        T* __restrict__ p,
        REDU_T* __restrict__ p_copy, // For mixed precision training, pass NULL if not needed
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    bool overflow = false;
    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                mi[ii] = b1*mi[ii] + (1-b1)*scaled_grad;
                vi[ii] = b2*vi[ii] + (1-b2)*scaled_grad*scaled_grad;
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                float update = (mi[ii]/denom) + (decay*pi[ii]);
                pi[ii] = pi[ii] - (step_size*update);
            } else {
                overflow = true;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i + totThreads*ii;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
                if (p_copy != NULL) {
                    convert(pi[ii], p_copy[j]);
                }
            }
        }
    }

    if (p_copy != NULL) {
        __syncthreads();
        if (overflow) {
            convert(float(INFINITY), p_copy[0]);
        }
    }
}

template <typename T, typename GRAD_T>
__global__ void maybe_adam_undo_cuda_kernel(
        volatile int* overflow_flag,
        T* __restrict__ p,
        T* __restrict__ m,
        T* __restrict__ v,
        const GRAD_T * __restrict__ g,
        const float b1,
        const float b2,
        const float eps,
        const float grad_scale,
        const float step_size,
        const size_t tsize,
        adamMode_t mode,
        const float decay)
{
    // NB! Skip undo kernel when overflow flag is NOT set
    if (overflow_flag && *overflow_flag == 0) return;

    //Assuming 2D grids and 2D blocks
    const int blockId = gridDim.x * blockIdx.y + blockIdx.x;
    const int threadsPerBlock = blockDim.x * blockDim.y;
    const int threadIdInBlock = threadIdx.y * blockDim.x + threadIdx.x;
    const int i = (blockId * threadsPerBlock + threadIdInBlock);
    const int totThreads = gridDim.x*gridDim.y*threadsPerBlock;

    T mi[ILP];
    T vi[ILP];
    T pi[ILP];
    T gi[ILP];

    for(int j_start = 0;  j_start < tsize;  j_start+=totThreads*ILP) {
#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            mi[ii] = T(0);
            vi[ii] = T(0);
            pi[ii] = T(0);
            gi[ii] = GRAD_T(0);

            int j = j_start + i*ILP;
            if (j < tsize) {
                pi[ii] = p[j];
                mi[ii] = m[j];
                vi[ii] = v[j];
                gi[ii] = static_cast<T>(g[j]);
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            T scaled_grad = gi[ii]/grad_scale;
            if (isfinite(scaled_grad)) {
                float denom;
                if (mode == ADAM_MODE_0)
                    denom = sqrtf(vi[ii] + eps);
                else // Mode 1
                    denom = sqrtf(vi[ii]) + eps;
                pi[ii] = (pi[ii] + step_size*(mi[ii]/denom)) / (1.0f - step_size*decay);
                mi[ii] = (mi[ii] - (1-b1)*scaled_grad) / b1;
                vi[ii] = (vi[ii] - (1-b2)*scaled_grad*scaled_grad) / b2;
                // Make sure round off errors don't create (small) negative value.
                // This can happen if we have to revert the very first step.
                vi[ii] = vi[ii] >= 0.0f ? vi[ii] : 0.0f;
            }
        }

#pragma unroll
        for(int ii = 0; ii < ILP; ii++) {
            int j = j_start + i*ILP;
            if (j < tsize) {
                m[j] = mi[ii];
                v[j] = vi[ii];
                p[j] = pi[ii];
            }
        }
    }
}

template <int DEPTH, typename FROM_T, typename TO_T>
struct MaybeCastFunctor
{
    __device__ __forceinline__ void operator()(
        int chunk_size,
        volatile int* overflow_flag,
        TensorListMetadata<DEPTH>& tl)
    {
        if (overflow_flag && *overflow_flag != 0) return;

        int tensor_loc = tl.block_to_tensor[blockIdx.x];
        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        FROM_T* p_in = (FROM_T *)tl.addresses[0][tensor_loc];
        p_in += chunk_idx*chunk_size;
        TO_T* p_out = (TO_T *)tl.addresses[1][tensor_loc];
        p_out += chunk_idx*chunk_size;

        n -= chunk_idx*chunk_size;
        int dim = chunk_size < n ? chunk_size : n;

	FROM_T pi[ILP];
        TO_T po[ILP];

        for(int j_start = 0;  j_start < dim;  j_start+=blockDim.x*ILP) {
#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                pi[ii] = FROM_T(0);
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    pi[ii] = p_in[j];
                }
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                convert(pi[ii], po[ii]);
            }

#pragma unroll
            for(int ii = 0; ii < ILP; ii++) {
                int j = j_start + threadIdx.x + ii*blockDim.x;
                if (j < dim) {
                    p_out[j] = po[ii];
                }
            }
        }
    }
};

void fused_strided_check_finite(
	at::Tensor & overflow_flag,
        at::Tensor & p_copy,
        int stride,
	int clear_overflow_first)
{
	//Get tensor size
	int tsize = p_copy.numel();
	int niter = (tsize + stride - 1) / stride;

	//Determine #threads and #blocks
	const int threadsPerBlock = 512;
	//In order to avoid race condition, blocks must be 1 when clear_overflow_first flag is set.
	const dim3 blocks(clear_overflow_first ? 1 : (niter+threadsPerBlock-1)/threadsPerBlock);
	TORCH_CHECK(at::cuda::detail::canUse32BitIndexMath(p_copy), "parameter tensor is too large to be indexed with int32");

	hipStream_t stream = at::cuda::getCurrentCUDAStream();
        using namespace at; // prevents "toString is undefined" errors
        DISPATCH_FLOAT_HALF_AND_BYTE(p_copy.scalar_type(), 0, "check_finite_cuda_kernel",
                strided_check_finite_cuda_kernel<scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.DATA_PTR<int>(),
                    p_copy.DATA_PTR<scalar_t_0>(),
                    tsize,
                    stride,
                    clear_overflow_first);
                );
	C10_CUDA_CHECK(hipGetLastError());
}

void fused_reversible_adam_cuda(
        at::Tensor & p,
        at::Tensor & p_copy,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
//      using namespace at;

      //Get tensor size
      int tsize = p.numel();
      //Determine #threads and #blocks
      const int threadsPerBlock = 512;
      const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
      TORCH_CHECK(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
      //Constants
      float step_size = 0;
      if (bias_correction == 1) {
          const float bias_correction1 = 1 - std::pow(beta1, step);
          const float bias_correction2 = 1 - std::pow(beta2, step);
          step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
      }
      else {
          step_size = lr;
      }
      hipStream_t stream = at::cuda::getCurrentCUDAStream();

      if (g.scalar_type() == at::ScalarType::Half) {
          //all other values should be fp32 for half gradients
          TORCH_CHECK(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
          //dispatch is done on the gradient type
          using namespace at; // prevents "toString is undefined" errors
          if (p_copy.numel() == 0 || p_copy.scalar_type() == g.scalar_type()) {
              DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                      using accscalar_t = at::acc_type<scalar_t_0, true>;
                      reversible_adam_cuda_kernel<accscalar_t, scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                          p.DATA_PTR<accscalar_t>(),
                          p_copy.numel() ? p_copy.DATA_PTR<scalar_t_0>() : NULL,
                          m.DATA_PTR<accscalar_t>(),
                          v.DATA_PTR<accscalar_t>(),
                          g.DATA_PTR<scalar_t_0>(),
                          beta1,
                          beta2,
                          eps,
                          grad_scale,
                          step_size,
                          tsize,
                          (adamMode_t) mode,
                          decay);
                      );
          } else {
              TORCH_CHECK(p_copy.scalar_type() == at::ScalarType::Byte, "expected parameter to be of byte type");
              DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_e5m2_kernel",
                      using accscalar_t = at::acc_type<scalar_t_0, true>;
                      reversible_adam_cuda_kernel<accscalar_t, scalar_t_0, uint8_t><<<blocks,threadsPerBlock, 0, stream>>>(
                          p.DATA_PTR<accscalar_t>(),
                          p_copy.DATA_PTR<uint8_t>(),
                          m.DATA_PTR<accscalar_t>(),
                          v.DATA_PTR<accscalar_t>(),
                          g.DATA_PTR<scalar_t_0>(),
                          beta1,
                          beta2,
                          eps,
                          grad_scale,
                          step_size,
                          tsize,
                          (adamMode_t) mode,
                          decay);
                      );
          }
      } else {
          using namespace at;
          DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                  reversible_adam_cuda_kernel<scalar_t_0, scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                      p.DATA_PTR<scalar_t_0>(),
                      NULL, //don't output p_copy for fp32, it's wasted write
                      m.DATA_PTR<scalar_t_0>(),
                      v.DATA_PTR<scalar_t_0>(),
                      g.DATA_PTR<scalar_t_0>(),
                      beta1,
                      beta2,
                      eps,
                      grad_scale,
                      step_size,
                      tsize,
                      (adamMode_t) mode,
                      decay);
                  );
      }
      C10_CUDA_CHECK(hipGetLastError());
}

void maybe_cast_cuda(
        at::Tensor & overflow_flag,
        at::Tensor & p_in,
        at::Tensor & p_out)
{
      //Get tensor size
      int tsize = p_in.numel();
      TORCH_CHECK(tsize == p_out.numel(), "p_in.numel() must equal p_out.numel()");
      //Determine #threads and #blocks
      const int threadsPerBlock = 512;
      const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
      TORCH_CHECK(at::cuda::detail::canUse32BitIndexMath(p_in), "parameter tensor is too large to be indexed with int32");
      //Constants
      hipStream_t stream = at::cuda::getCurrentCUDAStream();
      DISPATCH_FLOAT_HALF_AND_BYTE(p_in.scalar_type(), 0, "maybe_cast_cuda"
              DISPATCH_FLOAT_HALF_AND_BYTE(p_out.scalar_type(), 1, "maybe_cast_cuda",
                  maybe_cast_kernel<scalar_t_0,scalar_t_1><<<blocks,threadsPerBlock, 0, stream>>>(
                      overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                      p_in.DATA_PTR<scalar_t_0>(),
                      p_out.DATA_PTR<scalar_t_1>(),
                      tsize); ))
      C10_CUDA_CHECK(hipGetLastError());
}

void maybe_cast_cuda_mt(
    int chunk_size,
    at::Tensor overflow_flag,
    std::vector<std::vector<at::Tensor>> tensor_lists) // p_in, p_out
{
    //Constants
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    size_t tl_sz = tensor_lists.size();
    TORCH_CHECK(tl_sz == 2, "expected tensor lists of size 2");

    DISPATCH_FLOAT_HALF_AND_BYTE(tensor_lists[0][0].scalar_type(), 0, "maybe_cast_cuda_mt_kernel",
            DISPATCH_FLOAT_HALF_AND_BYTE(tensor_lists[1][0].scalar_type(), 1, "maybe_cast_cuda_mt_kernel",
                multi_tensor_apply<2>(
                    BLOCK_SIZE,
                    chunk_size,
                    overflow_flag,
                    tensor_lists,
                    MaybeCastFunctor<2, scalar_t_0, scalar_t_1>()); ))
    C10_CUDA_CHECK(hipGetLastError());
}

void fused_maybe_adam_undo_cuda(
        at::Tensor & overflow_flag,
        at::Tensor & p,
        at::Tensor & m,
        at::Tensor & v,
        at::Tensor & g,
        float lr,
        float beta1,
        float beta2,
        float eps,
        float grad_scale,
        int step,
        int mode,
        int bias_correction,
        float decay)
{
    //Get tensor size
    int tsize = p.numel();
    //Determine #threads and #blocks
    const int threadsPerBlock = 512;
    const dim3 blocks((tsize+threadsPerBlock-1)/threadsPerBlock);
    TORCH_CHECK(at::cuda::detail::canUse32BitIndexMath(p), "parameter tensor is too large to be indexed with int32");
    //Constants
    float step_size = 0;
    if (bias_correction == 1) {
        const float bias_correction1 = 1 - std::pow(beta1, step);
        const float bias_correction2 = 1 - std::pow(beta2, step);
        step_size = lr * std::sqrt(bias_correction2)/bias_correction1;
    }
    else {
        step_size = lr;
    }
    hipStream_t stream = at::cuda::getCurrentCUDAStream();

    if (g.scalar_type() == at::ScalarType::Half) {
        //all other values should be fp32 for half gradients
        TORCH_CHECK(p.scalar_type() == at::ScalarType::Float, "expected parameter to be of float type");
        //dispatch is done on the gradient type
        using namespace at; // prevents "toString is undefined" errors
        DISPATCH_FLOAT_AND_HALF(g.scalar_type(), 0, "adam_cuda_kernel",
                using accscalar_t = at::acc_type<scalar_t_0, true>;
                maybe_adam_undo_cuda_kernel<accscalar_t, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                    p.DATA_PTR<accscalar_t>(),
                    m.DATA_PTR<accscalar_t>(),
                    v.DATA_PTR<accscalar_t>(),
                    g.DATA_PTR<scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    tsize,
                    (adamMode_t) mode,
                    decay);
                );
    } else {
        using namespace at;
        DISPATCH_DOUBLE_AND_FLOAT(g.scalar_type(), 0, "adam_cuda_kernel",
                maybe_adam_undo_cuda_kernel<scalar_t_0, scalar_t_0><<<blocks,threadsPerBlock, 0, stream>>>(
                    overflow_flag.numel() ? overflow_flag.DATA_PTR<int>() : NULL,
                    p.DATA_PTR<scalar_t_0>(),
                    m.DATA_PTR<scalar_t_0>(),
                    v.DATA_PTR<scalar_t_0>(),
                    g.DATA_PTR<scalar_t_0>(),
                    beta1,
                    beta2,
                    eps,
                    grad_scale,
                    step_size,
                    tsize,
                    (adamMode_t) mode,
                    decay);
                );
    }
    C10_CUDA_CHECK(hipGetLastError());
}
