#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <c10/cuda/CUDACachingAllocator.h>
#include <ATen/cuda/HIPContext.h>
#include <list>
#include <cstdio>
#include <ctime>
#include <cassert>
#include "nccl.h"

/*
 * This file implements a crude but effective mechanism for copying data between tenors owned by different ranks
 * on the same machine using hipMemcpyAsync peer-to-peer transfers.
 */

namespace {

__global__ void AddDelay_kernel(const int delay, int* counter) {
    if (blockIdx.x == 0 && threadIdx.x == 0) {
        // waste time while doing something compiler can't predict, thus preventing it from optimizing away this code.
        int new_counter = 0;
        double elapsed = 0;
        clock_t start = clock();
        do {
            clock_t now = clock();
            elapsed = (double)(now - start)*1e9 / CLOCKS_PER_SEC;
            ++new_counter;
        } while (elapsed < (double)delay);
        *counter = new_counter;
    }
}

class NcclCommWrapper
{
    private:
        ncclComm_t comm;
        int rank, world_size;

        ncclDataType_t get_nccl_type(at::Tensor input)
        {
            switch (input.scalar_type())
            {
                case at::ScalarType::Half:
                    return ncclFloat16;
                case at::ScalarType::Float:
                    return ncclFloat32;
                case at::ScalarType::Double:
                    return ncclFloat64;
                case at::ScalarType::Byte:
                    return ncclUint8;
                case at::ScalarType::Char:
                    return ncclInt8;
                case at::ScalarType::Int:
                    return ncclInt32;
                case at::ScalarType::Long:
                    return ncclInt64;
                case at::ScalarType::BFloat16:
                    return ncclBfloat16;
                default:
                    assert(false);
            }
        }

    public:
        NcclCommWrapper()
        {
            memset(&comm, 0, sizeof(ncclComm_t));
            rank = 0;
            world_size = 0;
        }
        NcclCommWrapper(ncclUniqueId id, int my_rank, int num_ranks)
        {
            ncclCommInitRank(&comm, num_ranks, id, my_rank);
            rank = my_rank;
            world_size = num_ranks;
        }

        ~NcclCommWrapper()
        {
            printf("ncclCommDestroy()\n");
            ncclCommDestroy(comm);
        }

	void left_right_halo_exchange_inplace(int left_rank, int right_rank, at::Tensor left_output_halo, at::Tensor right_output_halo, at::Tensor left_input_halo, at::Tensor right_input_halo)
	{
            auto stream = at::cuda::getCurrentCUDAStream();
            ncclGroupStart();
            ncclDataType_t ncclType = get_nccl_type(left_output_halo);
	    bool left_zero = (left_rank < 0);
	    bool right_zero = (right_rank < 0);
            size_t left_n = torch::numel(left_output_halo);
            size_t right_n = torch::numel(right_output_halo);
	    assert(left_n > 0 && left_n == right_n);
	    if (left_zero) {
		left_input_halo.zero_();
	    } else {
                AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Bool, at::ScalarType::BFloat16, at::ScalarType::Half, left_output_halo.scalar_type(), "left_halo_exch", [&]() {
                    // send left (to my_rank - 1)
                    ncclSend(left_output_halo.data_ptr<scalar_t>(), left_n, ncclType, left_rank, comm, stream);
                    // receive left (from my_rank - 1)
                    ncclRecv(left_input_halo.data_ptr<scalar_t>(), right_n, ncclType, left_rank, comm, stream);
                });
            }
            if (right_zero) {
		right_input_halo.zero_();
	    } else {
                AT_DISPATCH_ALL_TYPES_AND3(at::ScalarType::Bool, at::ScalarType::BFloat16, at::ScalarType::Half, right_output_halo.scalar_type(), "right_halo_exch", [&]() {
                    // send right (to my_rank + 1 )
                    ncclSend(right_output_halo.data_ptr<scalar_t>(), right_n, ncclType, right_rank, comm, stream);
                    // receive right (from my_rank + 1)
                    ncclRecv(right_input_halo.data_ptr<scalar_t>(), left_n, ncclType, right_rank, comm, stream);
                });
            }
            ncclGroupEnd();
	}

        std::vector<at::Tensor> left_right_halo_exchange(int left_rank, int right_rank, at::Tensor left_output_halo, at::Tensor right_output_halo)
        {
            // after halo exchange:
            // left_output_halo of rank+1 ends up in right_input_halo of rank
            // right_output_halo of rank-1 ends up in left_input_halo of rank
            auto right_input_halo = torch::empty_like(left_output_halo);
            auto left_input_halo = torch::empty_like(right_output_halo);
	    left_right_halo_exchange_inplace(left_rank, right_rank, left_output_halo, right_output_halo, left_input_halo, right_input_halo);
	    return {left_input_halo, right_input_halo};
        }
};

class ManagedObjects
{
    public:
	ManagedObjects()
	{
	}
	~ManagedObjects()
	{
	    for (auto it = _nccl_comms.begin(); it != _nccl_comms.end();  ++it)
	    {
		delete *it;
	    }
	}

	int add_comm(NcclCommWrapper* comm)
	{
	    int handle = _nccl_comms.size();
	    _nccl_comms.push_back(comm);
	    return handle;
	}

	NcclCommWrapper& get_comm(int handle)
	{
            assert(handle >= 0 && handle < _nccl_comms.size());
	    return *_nccl_comms[handle];
	}

    private:
	std::vector<NcclCommWrapper*> _nccl_comms;
};
class ManagedObjects mo;

} // end anonymous namespace

namespace apex { namespace contrib { namespace nccl_p2p {

at::Tensor get_unique_nccl_id(int n)
{
    ncclUniqueId id;
    ncclGetUniqueId(&id);
    auto id_tensor = torch::empty({n,(int)sizeof(ncclUniqueId)}, torch::dtype(torch::kUInt8).device(torch::kCPU).requires_grad(false));
    auto id_ptr = id_tensor.data_ptr<uint8_t>();
    size_t offset = 0;
    for (int i = 0;  i < n;  ++i)
    {
        ncclUniqueId id;
        ncclGetUniqueId(&id);
        memcpy(id_ptr+offset, &id, sizeof(ncclUniqueId));
        offset += sizeof(ncclUniqueId);
    }
    return id_tensor;
}

int init_nccl_comm(at::Tensor unique_nccl_id, int my_rank, int num_ranks)
{
    ncclUniqueId id;
    auto unique_nccl_id_ptr = unique_nccl_id.data_ptr<uint8_t>();
    memcpy(&id, unique_nccl_id_ptr, sizeof(ncclUniqueId));
    NcclCommWrapper* comm = new NcclCommWrapper(id, my_rank, num_ranks);
    int handle = mo.add_comm(comm);
    comm = 0L;
    return handle;
}

void left_right_halo_exchange_inplace(int handle, int left_rank, int right_rank, at::Tensor left_output_halo, at::Tensor right_output_halo, at::Tensor left_input_halo, at::Tensor right_input_halo)
{
    class NcclCommWrapper& communicator = mo.get_comm(handle);
    return communicator.left_right_halo_exchange_inplace(left_rank, right_rank, left_output_halo, right_output_halo, left_input_halo, right_input_halo);
}

std::vector<at::Tensor> left_right_halo_exchange(int handle, int left_rank, int right_rank, at::Tensor left_output_halo, at::Tensor right_output_halo)
{
    class NcclCommWrapper& communicator = mo.get_comm(handle);
    return communicator.left_right_halo_exchange(left_rank, right_rank, left_output_halo, right_output_halo);
}

void add_delay(int delay)
{
    auto stream = at::cuda::getCurrentCUDAStream();
    auto t = torch::empty({1}, torch::dtype(torch::kInt32).device(torch::kCUDA));
    AddDelay_kernel<<<1,1,0,stream>>>(delay, t.data_ptr<int>());
}

}}}
